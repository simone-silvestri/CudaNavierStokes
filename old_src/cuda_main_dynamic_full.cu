#include "hip/hip_runtime.h"
#include "globals.h"
#include "cuda_functions.h"
#include "cuda_globals.h"
#include "cuda_main.h"

__global__ void runDevice(myprec *kin, myprec *enst, myprec *time) {

	dtC = d_dt;

	/* allocating temporary arrays and streams */
	void (*RHSDeviceDir[3])(myprec*, myprec*, myprec*, myprec*, myprec*, myprec*, myprec*, myprec*, myprec*, myprec*, myprec*, myprec*, myprec*, myprec*, myprec**, myprec *);
	void (*calcStresDir[3])(myprec*, myprec*, myprec*, myprec**);
	calcStresDir[0] = calcStressX;
	calcStresDir[1] = calcStressY;
	calcStresDir[2] = calcStressZ;

	RHSDeviceDir[0] = RHSDeviceSharedFlxX;
#if lPencilY == 1
	RHSDeviceDir[1] = RHSDeviceFullYL;
#else
	RHSDeviceDir[1] = RHSDeviceY;
#endif
#if lPencilZ == 1
	RHSDeviceDir[2] = RHSDeviceFullZL;
#else
	RHSDeviceDir[2] = RHSDeviceZ;
#endif

	__syncthreads();

	hipStream_t s[3];
    for (int i=0; i<3; i++) {
    	checkCudaDev( hipStreamCreateWithFlags(&s[i], hipStreamNonBlocking) );
    }

    initSolver();

    for (int istep = 0; istep < nsteps; istep++) {

    	calcState<<<grid0,block0>>>(d_r,d_u,d_v,d_w,d_e,d_h,d_t,d_p,d_m,d_l);
    	hipDeviceSynchronize();
//    	if (istep%100==0) {
//    		calcTimeStep<<<grid0,block0>>>(dttemp,d_r,d_u,d_v,d_w,d_e,d_m);
//    		for (int it=0; it<block0.x*block0.y; it++)
//    			dtC = MIN(dtC,dttemp[it]);
//    	}

    	dt2 = dtC/2.;


    	/* rk step 1 */
    	hipDeviceSynchronize();
    	for (int d = 0; d < 3; d++)
    		calcStresDir[d]<<<d_grid[d],d_block[d],0,s[d]>>>(d_u,d_v,d_w,gij);
    	hipDeviceSynchronize();

    	/* Only if you want a bulk solution changing in time!!! */
    	if(istep%250==0) {
    		calcIntegrals<<<1,1>>>(d_r,d_u,d_v,d_w,gij,&kin[istep],&enst[istep]);
    	}
    	if(istep > 0) 	time[istep] = time[istep-1] + dtC;
    	/* This will take some time to execute so take it away if not needed!! */


    	calcDil<<<grid0,block0>>>(gij,d_dil);
    	for (int d = 0; d < 3; d++)
    		RHSDeviceDir[d]<<<d_grid[d],d_block[d],0,s[d]>>>(d_rhsr1[d],d_rhsu1[d],d_rhsv1[d],d_rhsw1[d],d_rhse1[d],d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,gij,d_dil);
    	hipDeviceSynchronize();
    	eulerSum<<<grid0,block0>>>(d_tr,d_r,d_rhsr1[0],d_rhsr1[1],d_rhsr1[2],&dt2);
    	eulerSum<<<grid0,block0>>>(d_tu,d_u,d_rhsu1[0],d_rhsu1[1],d_rhsu1[2],&dt2);
    	eulerSum<<<grid0,block0>>>(d_tv,d_v,d_rhsv1[0],d_rhsv1[1],d_rhsv1[2],&dt2);
    	eulerSum<<<grid0,block0>>>(d_tw,d_w,d_rhsw1[0],d_rhsw1[1],d_rhsw1[2],&dt2);
    	eulerSum<<<grid0,block0>>>(d_te,d_e,d_rhse1[0],d_rhse1[1],d_rhse1[2],&dt2);
    	hipDeviceSynchronize();

    	//rk step 2
    	calcState<<<grid0,block0>>>(d_tr,d_tu,d_tv,d_tw,d_te,d_h,d_t,d_p,d_m,d_l);
    	for (int d = 0; d < 3; d++)
    		calcStresDir[d]<<<d_grid[d],d_block[d],0,s[d]>>>(d_tu,d_tv,d_tw,gij);
    	hipDeviceSynchronize();
    	calcDil<<<grid0,block0>>>(gij,d_dil);
    	hipDeviceSynchronize();
    	for (int d = 0; d < 3; d++)
    		RHSDeviceDir[d]<<<d_grid[d],d_block[d],0,s[d]>>>(d_rhsr2[d],d_rhsu2[d],d_rhsv2[d],d_rhsw2[d],d_rhse2[d],d_tr,d_tu,d_tv,d_tw,d_h,d_t,d_p,d_m,d_l,gij,d_dil);
    	hipDeviceSynchronize();
    	eulerSum<<<grid0,block0>>>(d_tr,d_r,d_rhsr2[0],d_rhsr2[1],d_rhsr2[2],&dt2);
    	eulerSum<<<grid0,block0>>>(d_tu,d_u,d_rhsu2[0],d_rhsu2[1],d_rhsu2[2],&dt2);
    	eulerSum<<<grid0,block0>>>(d_tv,d_v,d_rhsv2[0],d_rhsv2[1],d_rhsv2[2],&dt2);
    	eulerSum<<<grid0,block0>>>(d_tw,d_w,d_rhsw2[0],d_rhsw2[1],d_rhsw2[2],&dt2);
    	eulerSum<<<grid0,block0>>>(d_te,d_e,d_rhse2[0],d_rhse2[1],d_rhse2[2],&dt2);
    	hipDeviceSynchronize();


    	//rk step 3
    	calcState<<<grid0,block0>>>(d_tr,d_tu,d_tv,d_tw,d_te,d_h,d_t,d_p,d_m,d_l);
    	for (int d = 0; d < 3; d++)
    		calcStresDir[d]<<<d_grid[d],d_block[d],0,s[d]>>>(d_tu,d_tv,d_tw,gij);
    	hipDeviceSynchronize();
    	calcDil<<<grid0,block0>>>(gij,d_dil);
    	hipDeviceSynchronize();
    	for (int d = 0; d < 3; d++)
    		RHSDeviceDir[d]<<<d_grid[d],d_block[d],0,s[d]>>>(d_rhsr3[d],d_rhsu3[d],d_rhsv3[d],d_rhsw3[d],d_rhse3[d],d_tr,d_tu,d_tv,d_tw,d_h,d_t,d_p,d_m,d_l,gij,d_dil);
    	hipDeviceSynchronize();
    	eulerSum<<<grid0,block0>>>(d_tr,d_r,d_rhsr3[0],d_rhsr3[1],d_rhsr3[2],&dtC);
    	eulerSum<<<grid0,block0>>>(d_tu,d_u,d_rhsu3[0],d_rhsu3[1],d_rhsu3[2],&dtC);
    	eulerSum<<<grid0,block0>>>(d_tv,d_v,d_rhsv3[0],d_rhsv3[1],d_rhsv3[2],&dtC);
    	eulerSum<<<grid0,block0>>>(d_tw,d_w,d_rhsw3[0],d_rhsw3[1],d_rhsw3[2],&dtC);
    	eulerSum<<<grid0,block0>>>(d_te,d_e,d_rhse3[0],d_rhse3[1],d_rhse3[2],&dtC);
    	hipDeviceSynchronize();

    	//rk step 4
    	calcState<<<grid0,block0>>>(d_tr,d_tu,d_tv,d_tw,d_te,d_h,d_t,d_p,d_m,d_l);
    	hipDeviceSynchronize();
    	for (int d = 0; d < 3; d++)
    		calcStresDir[d]<<<d_grid[d],d_block[d],0,s[d]>>>(d_tu,d_tv,d_tw,gij);
    	hipDeviceSynchronize();
    	calcDil<<<grid0,block0>>>(gij,d_dil);
    	for (int d = 0; d < 3; d++)
    		RHSDeviceDir[d]<<<d_grid[d],d_block[d],0,s[d]>>>(d_rhsr4[d],d_rhsu4[d],d_rhsv4[d],d_rhsw4[d],d_rhse4[d],d_tr,d_tu,d_tv,d_tw,d_h,d_t,d_p,d_m,d_l,gij,d_dil);
    	hipDeviceSynchronize();
    	rk4final<<<grid0,block0>>>(d_r  ,d_rhsr1[0],d_rhsr2[0],d_rhsr3[0],d_rhsr4[0],
											 d_rhsr1[1],d_rhsr2[1],d_rhsr3[1],d_rhsr4[1],
											 d_rhsr1[2],d_rhsr2[2],d_rhsr3[2],d_rhsr4[2],&dtC);
    	rk4final<<<grid0,block0>>>(d_u  ,d_rhsu1[0],d_rhsu2[0],d_rhsu3[0],d_rhsu4[0],
											 d_rhsu1[1],d_rhsu2[1],d_rhsu3[1],d_rhsu4[1],
											 d_rhsu1[2],d_rhsu2[2],d_rhsu3[2],d_rhsu4[2],&dtC);
    	rk4final<<<grid0,block0>>>(d_v  ,d_rhsv1[0],d_rhsv2[0],d_rhsv3[0],d_rhsv4[0],
											 d_rhsv1[1],d_rhsv2[1],d_rhsv3[1],d_rhsv4[1],
											 d_rhsv1[2],d_rhsv2[2],d_rhsv3[2],d_rhsv4[2],&dtC);
    	rk4final<<<grid0,block0>>>(d_w  ,d_rhsw1[0],d_rhsw2[0],d_rhsw3[0],d_rhsw4[0],
											 d_rhsw1[1],d_rhsw2[1],d_rhsw3[1],d_rhsw4[1],
											 d_rhsw1[2],d_rhsw2[2],d_rhsw3[2],d_rhsw4[2],&dtC);
    	rk4final<<<grid0,block0>>>(d_e  ,d_rhse1[0],d_rhse2[0],d_rhse3[0],d_rhse4[0],
											 d_rhse1[1],d_rhse2[1],d_rhse3[1],d_rhse4[1],
											 d_rhse1[2],d_rhse2[2],d_rhse3[2],d_rhse4[2],&dtC);
    	hipDeviceSynchronize();

	}

    __syncthreads();

	for (int i=0; i<3; i++) {
		checkCudaDev( hipStreamDestroy(s[i]) );
	}
    clearSolver();

}

__global__ void eulerSum(myprec *a, myprec *b, myprec *cx, myprec *cy, myprec *cz, myprec *dt) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidX();
	a[id.g] = b[id.g] + ( cx[id.g] + cy[id.g] + cz[id.g] )*(*dt);
}

__global__ void rk4final(myprec *a, myprec *bx, myprec *cx, myprec *dx, myprec *ex,	myprec *by, myprec *cy, myprec *dy, myprec *ey,	myprec *bz, myprec *cz, myprec *dz, myprec *ez, myprec *dt) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidX();
	a[id.g] = a[id.g] + (*dt)*( bx[id.g] + 2*cx[id.g] + 2*dx[id.g] + ex[id.g] +
								by[id.g] + 2*cy[id.g] + 2*dy[id.g] + ey[id.g] +
								bz[id.g] + 2*cz[id.g] + 2*dz[id.g] + ez[id.g])/6.;
}

__global__ void calcState(myprec *rho, myprec *uvel, myprec *vvel, myprec *wvel, myprec *ret, myprec *ht, myprec *tem, myprec *pre, myprec *mu, myprec *lam) {

	int threadsPerBlock  = blockDim.x * blockDim.y;
	int threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
	int blockNumInGrid   = blockIdx.x  + gridDim.x  * blockIdx.y;

	int gt = blockNumInGrid * threadsPerBlock + threadNumInBlock;

    myprec cvInv = (gamma - 1.0)/Rgas;

    myprec invrho = 1.0/rho[gt];

    myprec en = ret[gt]*invrho - 0.5*(uvel[gt]*uvel[gt] + vvel[gt]*vvel[gt] + wvel[gt]*wvel[gt]);
    tem[gt]   = cvInv*en;
    pre[gt]   = rho[gt]*Rgas*tem[gt];
    ht[gt]    = (ret[gt] + pre[gt])*invrho;

    myprec suth = pow(tem[gt],0.75);
    mu[gt]      = suth/Re;
    lam[gt]     = suth/Re/Pr/Ec;
    __syncthreads();

}

__global__ void calcTimeStep(myprec *temporary, myprec *rho, myprec *uvel, myprec *vvel, myprec *wvel, myprec *ret, myprec *mu) {

	__shared__ myprec temp[mx];

	int threadsPerBlock  = blockDim.x * blockDim.y;
	int threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
	int blockNumInGrid   = blockIdx.x  + gridDim.x  * blockIdx.y;

	int gt = blockNumInGrid * threadsPerBlock + threadNumInBlock;

    myprec dtConvInv = 0.0;
    myprec dtViscInv = 0.0;


    myprec ien = ret[gt]/rho[gt] - 0.5*(uvel[gt]*uvel[gt] + vvel[gt]*vvel[gt] + wvel[gt]*wvel[gt]);
    myprec sos = pow(gamma*(gamma-1)*ien,0.5);

    dtConvInv =  MAX( (abs(uvel[gt]) + sos)*d_dx, MAX( (abs(vvel[gt]) + sos)*d_dy, (abs(wvel[gt]) + sos)*d_dz) );
    dtViscInv =  MAX( mu[gt]*d_d2x, MAX( mu[gt]*d_d2y, mu[gt]*d_d2z) );

    temp[threadNumInBlock] = CFL/MAX(dtConvInv, dtViscInv);
    __syncthreads();

    if(threadNumInBlock == 0) {
    	temporary[blockNumInGrid] = 1000.0;
    	for (int it = 0; it < blockDim.x*blockDim.y; it++)
    		temporary[blockNumInGrid] = MIN(temporary[blockNumInGrid],temp[it] );
    }

    __syncthreads();
}

__device__ void initSolver() {

    for (int i=0; i<3; i++) {
    	checkCudaDev( hipMalloc((void**)&d_rhsr1[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhsr2[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhsr3[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhsr4[i],mx*my*mz*sizeof(myprec)) );
		checkCudaDev( hipMalloc((void**)&d_rhsu1[i],mx*my*mz*sizeof(myprec)) );
		checkCudaDev( hipMalloc((void**)&d_rhsu2[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhsu3[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhsu4[i],mx*my*mz*sizeof(myprec)) );
		checkCudaDev( hipMalloc((void**)&d_rhsv1[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhsv2[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhsv3[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhsv4[i],mx*my*mz*sizeof(myprec)) );
		checkCudaDev( hipMalloc((void**)&d_rhsw1[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhsw2[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhsw3[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhsw4[i],mx*my*mz*sizeof(myprec)) );
		checkCudaDev( hipMalloc((void**)&d_rhse1[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhse2[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhse3[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhse4[i],mx*my*mz*sizeof(myprec)) );
    }
	checkCudaDev( hipMalloc((void**)&dttemp,block0.x*block0.y*sizeof(myprec)) );
    for (int i=0; i<9; i++)
    	checkCudaDev( hipMalloc((void**)&gij[i],mx*my*mz*sizeof(myprec)) );

}

__device__ void clearSolver() {

	for (int i=0; i<3; i++) {
		checkCudaDev( hipFree(d_rhsr1[i]) );
		checkCudaDev( hipFree(d_rhsr2[i]) );
		checkCudaDev( hipFree(d_rhsr3[i]) );
		checkCudaDev( hipFree(d_rhsr4[i]) );
		checkCudaDev( hipFree(d_rhsu1[i]) );
		checkCudaDev( hipFree(d_rhsu2[i]) );
		checkCudaDev( hipFree(d_rhsu3[i]) );
		checkCudaDev( hipFree(d_rhsu4[i]) );
		checkCudaDev( hipFree(d_rhsv1[i]) );
		checkCudaDev( hipFree(d_rhsv2[i]) );
		checkCudaDev( hipFree(d_rhsv3[i]) );
		checkCudaDev( hipFree(d_rhsv4[i]) );
		checkCudaDev( hipFree(d_rhsw1[i]) );
		checkCudaDev( hipFree(d_rhsw2[i]) );
		checkCudaDev( hipFree(d_rhsw3[i]) );
		checkCudaDev( hipFree(d_rhsw4[i]) );
		checkCudaDev( hipFree(d_rhse1[i]) );
		checkCudaDev( hipFree(d_rhse2[i]) );
		checkCudaDev( hipFree(d_rhse3[i]) );
		checkCudaDev( hipFree(d_rhse4[i]) );
	}
	checkCudaDev( hipFree(dttemp) );
    for (int i=0; i<9; i++)
    	checkCudaDev( hipFree(gij[i]) );

}
