#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include "globals.h"
#include "cuda_functions.h"
#include "cuda_math.h"
#include "boundary.h"


/*
 *  The L-versions of the RHS have to be ran with
 *  - the L-version of the derivatives
 *  i.e.: derDev1xL instead of derDev1x
 *  - the L-version of the grid
 *  i.e.: h_gridL[0] instead of h_grid[0]
 */

/* The whole RHS in the X direction is calculated in RHSDeviceSharedFlxX_old thanks to the beneficial memory layout that allows to use small pencils */
/* For the Y and Z direction, fluxes require a small pencil discretization while the rest of the RHS can be calculated on large pencils which speed
 * up significantly the computation. Therefore 5 streams are used
 * stream 0 -> complete X RHS (in RHSDeviceSharedFlxX_old) (small pencil grid)
 * stream 1 -> viscous terms and pressure terms in Y (in RHSDeviceFullYL) (large pencil grid)
 * stream 2 -> viscous terms and pressure terms in Z (in RHSDeviceFullZL) (large pencil grid)
 * stream 3 -> advective fluxes in Y direction (in FLXDeviceY) (small pencil transposed grid)
 * stream 4 -> advective fluxes in Z direction (in FLXDeviceZ) (small pencil transposed grid)*/

__global__ void RHSDeviceSharedFlxX(myprec *rX, myprec *uX, myprec *vX, myprec *wX, myprec *eX,
		myprec *r,  myprec *u,  myprec *v,  myprec *w,  myprec *h ,
		myprec *t,  myprec *p,  myprec *mu, myprec *lam,
		myprec *dil, myprec *dpdz) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidX();

	int si = id.i + stencilSize;       // local i for shared memory access + halo offset
	int sj = id.tiy;                   // local j for shared memory access

	myprec rXtmp=0;
	myprec uXtmp=0;
	myprec vXtmp=0;
	myprec wXtmp=0;
	myprec eXtmp=0;

	myprec wrk1=0;
	myprec wrk2=0;

	__shared__ myprec s_r[sPencils][mx+stencilSize*2];
	__shared__ myprec s_u[sPencils][mx+stencilSize*2];
	__shared__ myprec s_v[sPencils][mx+stencilSize*2];
	__shared__ myprec s_w[sPencils][mx+stencilSize*2];
	__shared__ myprec s_h[sPencils][mx+stencilSize*2];
	__shared__ myprec s_t[sPencils][mx+stencilSize*2];
	__shared__ myprec s_p[sPencils][mx+stencilSize*2];
	__shared__ myprec s_m[sPencils][mx+stencilSize*2];
	__shared__ myprec s_l[sPencils][mx+stencilSize*2];
#if !periodicX
	__shared__ myprec s_s0[sPencils][mx+stencilSize*2];
	__shared__ myprec s_s4[sPencils][mx+stencilSize*2];
	__shared__ myprec s_s8[sPencils][mx+stencilSize*2];
#endif
	__shared__ myprec s_dil[sPencils][mx+stencilSize*2];

	s_r[sj][si] = r[id.g];
	s_u[sj][si] = u[id.g];
	s_v[sj][si] = v[id.g];
	s_w[sj][si] = w[id.g];
	s_h[sj][si] = h[id.g];
	s_t[sj][si] = t[id.g];
	s_p[sj][si] = p[id.g];
	s_m[sj][si] = mu[id.g];
	s_l[sj][si] = lam[id.g];
#if !periodicX
	s_s0[sj][si]= sij[0][id.g];
	s_s4[sj][si]= sij[4][id.g];
	s_s8[sj][si]= sij[8][id.g];
#endif
	s_dil[sj][si] = dil[id.g];
	__syncthreads();

	// fill in periodic images in shared memory array
	if (id.i < stencilSize) {
#if periodicX
		perBCx(s_r[sj],si); perBCx(s_u[sj],si);
		perBCx(s_v[sj],si); perBCx(s_w[sj],si);
		perBCx(s_h[sj],si); perBCx(s_t[sj],si);
		perBCx(s_p[sj],si); perBCx(s_m[sj],si);
		perBCx(s_l[sj],si);
#else
		wallBCxMir(s_p[sj],si);
		wallBCxVel(s_u[sj],si); wallBCxVel(s_v[sj],si); wallBCxVel(s_w[sj],si);
		wallBCxExt(s_t[sj],si,1.0,1.0);
		stateBoundPT(s_r[sj], s_t[sj], s_u[sj], s_v[sj], s_w[sj], s_h[sj], s_p[sj], s_m[sj], s_l[sj], si);
		wallBCxMir(s_s0[sj],si); wallBCxVel(s_s4[sj],si);  wallBCxVel(s_s8[sj],si);
#endif
	}

	__syncthreads();

	//initialize momentum RHS with stresses so that they can be added for both viscous terms and viscous heating without having to load additional terms
	uXtmp = ( 2 * sij[0][id.g] - 2./3.*s_dil[sj][si] );
	vXtmp = (     sij[1][id.g] + sij[3][id.g]  );
	wXtmp = (     sij[2][id.g] + sij[6][id.g]  );

	//adding the viscous dissipation part duidx*mu*six
	eXtmp = s_m[sj][si]*(uXtmp*sij[0][id.g] + vXtmp*sij[1][id.g] + wXtmp*sij[2][id.g]);

	//Adding here the terms d (mu) dx * sxj; (lambda in case of h in rhse);
	derDevSharedV1x(&wrk2,s_m[sj],si); //wrk2 = d (mu) dx
    uXtmp *= wrk2;
	vXtmp *= wrk2;
	wXtmp *= wrk2;

	// viscous fluxes derivative mu*d^2ui dx^2
	derDevSharedV2x(&wrk1,s_u[sj],si);
	uXtmp = uXtmp + wrk1*s_m[sj][si];
	derDevSharedV2x(&wrk1,s_v[sj],si);
	vXtmp = vXtmp + wrk1*s_m[sj][si];
	derDevSharedV2x(&wrk1,s_w[sj],si);
	wXtmp = wXtmp + wrk1*s_m[sj][si];

	//adding the viscous dissipation part ui*(mu * d2duidx2 + dmudx * six)
	eXtmp = eXtmp + s_u[sj][si]*uXtmp + s_v[sj][si]*vXtmp + s_w[sj][si]*wXtmp;

	//adding the molecular conduction part (d2 temp dx2*lambda + dlambda dx * d temp dx)
	derDevSharedV2x(&wrk1,s_t[sj],si);
	eXtmp = eXtmp + wrk1*s_l[sj][si];
	derDevSharedV1x(&wrk2,s_l[sj],si); //wrk2 = d (lam) dx
	derDevSharedV1x(&wrk1,s_t[sj],si); //wrk1 = d (t) dx
	eXtmp = eXtmp + wrk1*wrk2;

	//Adding here the terms - d (ru phi) dx;

	fluxQuadSharedx(&wrk1,s_r[sj],s_u[sj],si);
	rXtmp = wrk1;
	__syncthreads();
	fluxCubeSharedx(&wrk1,s_r[sj],s_u[sj],s_u[sj],si);
	uXtmp = uXtmp + wrk1;
	__syncthreads();
	fluxCubeSharedx(&wrk1,s_r[sj],s_u[sj],s_v[sj],si);
	vXtmp = vXtmp + wrk1;
	__syncthreads();
	fluxCubeSharedx(&wrk1,s_r[sj],s_u[sj],s_w[sj],si);
	wXtmp = wXtmp + wrk1;
	__syncthreads();
	fluxCubeSharedx(&wrk1,s_r[sj],s_u[sj],s_h[sj],si);
	eXtmp = eXtmp + wrk1;
	__syncthreads();

	// pressure and dilation derivatives
	if (id.i < stencilSize) {
#if periodicX
		perBCx(s_dil[sj],si);
#else
		wallBCxDil(s_dil[sj],s_s0[sj],s_s4[sj],s_s8[sj],si);
#endif
	}
	__syncthreads();

	derDevSharedV1x(&wrk2,s_dil[sj],si);
	derDevShared1x(&wrk1 ,s_p[sj],si);
	uXtmp = uXtmp + s_m[sj][si]*wrk2/3.0     - wrk1 ;
	eXtmp = eXtmp + s_m[sj][si]*wrk2/3.0*s_u[sj][si];

	rX[id.g] = rXtmp;
	uX[id.g] = uXtmp;
	vX[id.g] = vXtmp;
	wX[id.g] = wXtmp;
	eX[id.g] = eXtmp ;
}

__global__ void RHSDeviceSharedFlxY(myprec *rY, myprec *uY, myprec *vY, myprec *wY, myprec *eY,
		myprec *r,  myprec *u,  myprec *v,  myprec *w,  myprec *h ,
		myprec *t,  myprec *p,  myprec *mu, myprec *lam,
		myprec *dil, myprec *dpdz) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidYFlx();

	int si = id.j + stencilSize;       // local i for shared memory access + halo offset
	int sj = id.tiy;                   // local j for shared memory access

	myprec rYtmp=0;
	myprec uYtmp=0;
	myprec vYtmp=0;
	myprec wYtmp=0;
	myprec eYtmp=0;

	myprec wrk1=0;
	myprec wrk2=0;

	__shared__ myprec s_r[sPencils][my+stencilSize*2];
	__shared__ myprec s_u[sPencils][my+stencilSize*2];
	__shared__ myprec s_v[sPencils][my+stencilSize*2];
	__shared__ myprec s_w[sPencils][my+stencilSize*2];
	__shared__ myprec s_h[sPencils][my+stencilSize*2];
	__shared__ myprec s_t[sPencils][my+stencilSize*2];
	__shared__ myprec s_p[sPencils][my+stencilSize*2];
	__shared__ myprec s_m[sPencils][my+stencilSize*2];
	__shared__ myprec s_l[sPencils][my+stencilSize*2];
	__shared__ myprec s_s3[sPencils][my+stencilSize*2];
	__shared__ myprec s_s4[sPencils][my+stencilSize*2];
	__shared__ myprec s_s5[sPencils][my+stencilSize*2];
	__shared__ myprec s_dil[sPencils][my+stencilSize*2];

	s_r[sj][si] = r[id.g];
	s_u[sj][si] = u[id.g];
	s_v[sj][si] = v[id.g];
	s_w[sj][si] = w[id.g];
	s_h[sj][si] = h[id.g];
	s_t[sj][si] = t[id.g];
	s_p[sj][si] = p[id.g];
	s_m[sj][si] = mu[id.g];
	s_l[sj][si] = lam[id.g];
	s_dil[sj][si] = dil[id.g];
	s_s3[sj][si] = sij[3][id.g];
	s_s4[sj][si] = sij[4][id.g];
	s_s5[sj][si] = sij[5][id.g];
	__syncthreads();

	// fill in periodic images in shared memory array
	if (id.j < stencilSize) {
		perBCy(s_r[sj],si); perBCy(s_u[sj],si);
		perBCy(s_v[sj],si); perBCy(s_w[sj],si);
		perBCy(s_h[sj],si); perBCy(s_t[sj],si);
		perBCy(s_p[sj],si); perBCy(s_m[sj],si);
		perBCy(s_l[sj],si);
	}
	__syncthreads();

	//initialize momentum RHS with stresses so that they can be added for both viscous terms and viscous heating without having to load additional terms
	uYtmp = (     s_s3[sj][si] + sij[1][id.g]        ) ;
	vYtmp = ( 2 * s_s4[sj][si] - 2./3.*s_dil[sj][si] ) ;
	wYtmp = (     s_s5[sj][si] + sij[7][id.g]        ) ;

	//adding the viscous dissipation part duidy*mu*siy
	eYtmp = s_m[sj][si]*(uYtmp*s_s3[sj][si] + vYtmp*s_s4[sj][si] + wYtmp*s_s5[sj][si]);

	//Adding here the terms d (mu) dy * siy;
	derDevSharedV1y(&wrk2,s_m[sj],si); //wrk2 = d (mu) dx
	uYtmp *= wrk2;
	vYtmp *= wrk2;
	wYtmp *= wrk2;

	// viscous fluxes derivative mu*d^2dui dy^2
	derDevSharedV2y(&wrk1,s_u[sj],si);
	uYtmp = uYtmp + wrk1*s_m[sj][si];
	derDevSharedV2y(&wrk1,s_v[sj],si);
	vYtmp = vYtmp + wrk1*s_m[sj][si];
	derDevSharedV2y(&wrk1,s_w[sj],si);
	wYtmp = wYtmp + wrk1*s_m[sj][si];

	//adding the viscous dissipation part ui*(mu * d2duidy2 + dmudy * siy)
	eYtmp = eYtmp + s_u[sj][si]*uYtmp + s_v[sj][si]*vYtmp + s_w[sj][si]*wYtmp;

	derDevSharedV2y(&wrk1,s_t[sj],si);
	eYtmp = eYtmp + wrk1*s_l[sj][si];
	derDevSharedV1y(&wrk2,s_l[sj],si); //wrk2 = d (lam) dx
	derDevSharedV1y(&wrk1,s_t[sj],si); //wrk1 = d (t) dx
	eYtmp = eYtmp + wrk1*wrk2;


	// split advection terms

	//Adding here the terms - d (ru phi) dy;

	fluxQuadSharedy(&wrk1,s_r[sj],s_v[sj],si);
	rYtmp = wrk1;
	__syncthreads();
	fluxCubeSharedy(&wrk1,s_r[sj],s_v[sj],s_u[sj],si);
	uYtmp = uYtmp + wrk1;
	__syncthreads();
	fluxCubeSharedy(&wrk1,s_r[sj],s_v[sj],s_v[sj],si);
	vYtmp = vYtmp + wrk1;
	__syncthreads();
	fluxCubeSharedy(&wrk1,s_r[sj],s_v[sj],s_w[sj],si);
	wYtmp = wYtmp + wrk1;
	__syncthreads();
	fluxCubeSharedy(&wrk1,s_r[sj],s_v[sj],s_h[sj],si);
	eYtmp = eYtmp + wrk1;
	__syncthreads();

	// pressure and dilation derivatives
	if (id.j < stencilSize) {
		perBCy(s_dil[sj],si);
	}
	__syncthreads();
	derDevSharedV1y(&wrk2,s_dil[sj],si);
	derDevShared1y(&wrk1,s_p[sj],si);
	vYtmp = vYtmp + s_m[sj][si]*wrk2/3.0     - wrk1 ;
	eYtmp = eYtmp + s_m[sj][si]*wrk2/3.0*s_v[sj][si];
#if useStreams
	rY[id.g] = rYtmp;
	uY[id.g] = uYtmp;
	vY[id.g] = vYtmp;
	wY[id.g] = wYtmp;
	eY[id.g] = eYtmp;
#else
	rY[id.g] += rYtmp;
	uY[id.g] += uYtmp;
	vY[id.g] += vYtmp;
	wY[id.g] += wYtmp;
	eY[id.g] += eYtmp;
#endif
}

__global__ void RHSDeviceSharedFlxZ(myprec *rZ, myprec *uZ, myprec *vZ, myprec *wZ, myprec *eZ,
		myprec *r,  myprec *u,  myprec *v,  myprec *w,  myprec *h ,
		myprec *t,  myprec *p,  myprec *mu, myprec *lam,
		myprec *dil, myprec *dpdz) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidZFlx();

	int si = id.k + stencilSize;       // local i for shared memory access + halo offset
	int sj = id.tiy;                   // local j for shared memory access

	myprec rZtmp=0;
	myprec uZtmp=0;
	myprec vZtmp=0;
	myprec wZtmp=0;
	myprec eZtmp=0;

	myprec wrk1=0;
	myprec wrk2=0;

	__shared__ myprec s_r[sPencils][mz+stencilSize*2];
	__shared__ myprec s_u[sPencils][mz+stencilSize*2];
	__shared__ myprec s_v[sPencils][mz+stencilSize*2];
	__shared__ myprec s_w[sPencils][mz+stencilSize*2];
	__shared__ myprec s_h[sPencils][mz+stencilSize*2];
	__shared__ myprec s_t[sPencils][mz+stencilSize*2];
	__shared__ myprec s_p[sPencils][mz+stencilSize*2];
	__shared__ myprec s_m[sPencils][mz+stencilSize*2];
	__shared__ myprec s_l[sPencils][mz+stencilSize*2];
	__shared__ myprec s_s6[sPencils][mz+stencilSize*2];
	__shared__ myprec s_s7[sPencils][mz+stencilSize*2];
	__shared__ myprec s_s8[sPencils][mz+stencilSize*2];
	__shared__ myprec s_dil[sPencils][mz+stencilSize*2];

	s_r[sj][si] = r[id.g];
	s_u[sj][si] = u[id.g];
	s_v[sj][si] = v[id.g];
	s_w[sj][si] = w[id.g];
	s_h[sj][si] = h[id.g];
	s_t[sj][si] = t[id.g];
	s_p[sj][si] = p[id.g];
	s_m[sj][si] = mu[id.g];
	s_l[sj][si] = lam[id.g];
	s_s6[sj][si] = sij[6][id.g];
	s_s7[sj][si] = sij[7][id.g];
	s_s8[sj][si] = sij[8][id.g];
	s_dil[sj][si] = dil[id.g];
	__syncthreads();

	// fill in periodic images in shared memory array
	if (id.k < stencilSize) {
		perBCz(s_r[sj],si); perBCz(s_u[sj],si);
		perBCz(s_v[sj],si); perBCz(s_w[sj],si);
		perBCz(s_h[sj],si); perBCz(s_t[sj],si);
		perBCz(s_p[sj],si); perBCz(s_m[sj],si);
		perBCz(s_l[sj],si);
	}

	__syncthreads();

	//initialize momentum RHS with stresses so that they can be added for both viscous terms and viscous heating without having to load additional terms
	uZtmp = (    s_s6[sj][si] + sij[2][id.g]        );
	vZtmp = (    s_s7[sj][si] + sij[5][id.g]        );
	wZtmp = (2 * s_s8[sj][si] - 2./3.*s_dil[sj][si] );

	//adding the viscous dissipation part duidz*mu*siz
	eZtmp = s_m[sj][si]*(uZtmp*s_s6[sj][si] + vZtmp*s_s7[sj][si] + wZtmp*s_s8[sj][si]);

	//Adding here the terms d (mu) dz * szj;
	derDevSharedV1z(&wrk2,s_m[sj],si); //wrk2 = d (mu) dz
    uZtmp *= wrk2;
	vZtmp *= wrk2;
	wZtmp *= wrk2;

	// viscous fluxes derivative
	derDevSharedV2z(&wrk1,s_u[sj],si);
	uZtmp = uZtmp + wrk1*s_m[sj][si];
	derDevSharedV2z(&wrk1,s_v[sj],si);
	vZtmp = vZtmp + wrk1*s_m[sj][si];
	derDevSharedV2z(&wrk1,s_w[sj],si);
	wZtmp = wZtmp + wrk1*s_m[sj][si];

	//adding the viscous dissipation part ui*(mu * d2duidz2 + dmudz * siz)
	derDevSharedV2z(&wrk1,s_t[sj],si);
	eZtmp = eZtmp + s_u[sj][si]*uZtmp + s_v[sj][si]*vZtmp + s_w[sj][si]*wZtmp + wrk1*s_l[sj][si];

	derDevSharedV1z(&wrk2,s_l[sj],si); //wrk2 = d (lam) dz
	derDevSharedV1z(&wrk1,s_t[sj],si); //wrk1 = d (t) dx
	eZtmp = eZtmp + wrk1*wrk2;

	//Adding here the terms - d (ru phi) dz;

	fluxQuadSharedz(&wrk1,s_r[sj],s_w[sj],si);
	rZtmp = wrk1;
	__syncthreads();
	fluxCubeSharedz(&wrk1,s_r[sj],s_w[sj],s_u[sj],si);
	uZtmp = uZtmp + wrk1;
	__syncthreads();
	fluxCubeSharedz(&wrk1,s_r[sj],s_w[sj],s_v[sj],si);
	vZtmp = vZtmp + wrk1;
	__syncthreads();
	fluxCubeSharedz(&wrk1,s_r[sj],s_w[sj],s_w[sj],si);
	wZtmp = wZtmp + wrk1;
	__syncthreads();
	fluxCubeSharedz(&wrk1,s_r[sj],s_w[sj],s_h[sj],si);
	eZtmp = eZtmp + wrk1;
	__syncthreads();

	// pressure and dilation derivatives
	__syncthreads();
	if (id.k < stencilSize) {
		perBCz(s_dil[sj],si);
	}
	__syncthreads();
	derDevSharedV1z(&wrk2,s_dil[sj],si);
	derDevShared1z(&wrk1,s_p[sj],si);
	wZtmp = wZtmp + s_m[sj][si]*wrk2/3.0     - wrk1 ;
	eZtmp = eZtmp + s_m[sj][si]*wrk2/3.0*s_w[sj][si];

#if useStreams
	rZ[id.g] = rZtmp;
	uZ[id.g] = uZtmp;
	vZ[id.g] = vZtmp;
	wZ[id.g] = wZtmp + *dpdz;
	eZ[id.g] = eZtmp + *dpdz*s_w[sj][si] ;
#else
	rZ[id.g] += rZtmp;
	uZ[id.g] += uZtmp;
	vZ[id.g] += vZtmp;
	wZ[id.g] += wZtmp + *dpdz;
	eZ[id.g] += eZtmp + *dpdz*s_w[sj][si] ;
#endif
	__syncthreads();
}
