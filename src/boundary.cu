#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "cuda_functions.h"
#include "cuda_globals.h"

__global__ void calcBCxStress(myprec *sij[9]) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidXBound();
}
