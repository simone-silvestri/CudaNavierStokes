#include "hip/hip_runtime.h"

/**
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */

#include "globals.h"
#include "cuda_functions.h"
#include "cuda_math.h"
#include "boundary.h"


/*
 *  The L-versions of the RHS have to be ran with
 *  - the L-version of the derivatives
 *  i.e.: derDev1xL instead of derDev1x
 *  - the L-version of the grid
 *  i.e.: h_gridL[0] instead of h_grid[0]
 */

/* The whole RHS in the X direction is calculated in RHSDeviceSharedFlxX_old thanks to the beneficial memory layout that allows to use small pencils */
/* For the Y and Z direction, fluxes require a small pencil discretization while the rest of the RHS can be calculated on large pencils which speed
 * up significantly the computation. Therefore 5 streams are used
 * stream 0 -> complete X RHS (in RHSDeviceSharedFlxX_old) (small pencil grid)
 * stream 1 -> viscous terms and pressure terms in Y (in RHSDeviceFullYL) (large pencil grid)
 * stream 2 -> viscous terms and pressure terms in Z (in RHSDeviceFullZL) (large pencil grid)
 * stream 3 -> advective fluxes in Y direction (in FLXDeviceY) (small pencil transposed grid)
 * stream 4 -> advective fluxes in Z direction (in FLXDeviceZ) (small pencil transposed grid)*/

__global__ void RHSDeviceSharedFlxX(myprec *rX, myprec *uX, myprec *vX, myprec *wX, myprec *eX,
		myprec *r,  myprec *u,  myprec *v,  myprec *w,  myprec *h ,
		myprec *t,  myprec *p,  myprec *mu, myprec *lam,
		myprec *sij[9], myprec *dil) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidX();

	int si = id.i + stencilSize;       // local i for shared memory access + halo offset
	int sj = id.tiy;                   // local j for shared memory access

	myprec rXtmp=0;
	myprec uXtmp=0;
	myprec vXtmp=0;
	myprec wXtmp=0;
	myprec eXtmp=0;

	myprec wrk1=0;
	myprec wrk2=0;

	__shared__ myprec s_r[sPencils][mx+stencilSize*2];
	__shared__ myprec s_u[sPencils][mx+stencilSize*2];
	__shared__ myprec s_v[sPencils][mx+stencilSize*2];
	__shared__ myprec s_w[sPencils][mx+stencilSize*2];
	__shared__ myprec s_h[sPencils][mx+stencilSize*2];
	__shared__ myprec s_t[sPencils][mx+stencilSize*2];
	__shared__ myprec s_p[sPencils][mx+stencilSize*2];
	__shared__ myprec s_m[sPencils][mx+stencilSize*2];
	__shared__ myprec s_l[sPencils][mx+stencilSize*2];
	__shared__ myprec s_s0[sPencils][mx+stencilSize*2];
#if !periodicX
	__shared__ myprec s_s4[sPencils][mx+stencilSize*2];
	__shared__ myprec s_s8[sPencils][mx+stencilSize*2];
#endif
	__shared__ myprec s_dil[sPencils][mx+stencilSize*2];

	s_r[sj][si] = r[id.g];
	s_u[sj][si] = u[id.g];
	s_v[sj][si] = v[id.g];
	s_w[sj][si] = w[id.g];
	s_h[sj][si] = h[id.g];
	s_t[sj][si] = t[id.g];
	s_p[sj][si] = p[id.g];
	s_m[sj][si] = mu[id.g];
	s_l[sj][si] = lam[id.g];
	s_s0[sj][si]= sij[0][id.g];
#if !periodicX
	s_s4[sj][si]= sij[4][id.g];
	s_s8[sj][si]= sij[8][id.g];
#endif
	s_dil[sj][si] = dil[id.g];
	__syncthreads();

	// fill in periodic images in shared memory array
	if (id.i < stencilSize) {
#if periodicX
		perBCx(s_r[sj],si); perBCx(s_u[sj],si);
		perBCx(s_v[sj],si); perBCx(s_w[sj],si);
		perBCx(s_h[sj],si); perBCx(s_t[sj],si);
		perBCx(s_p[sj],si); perBCx(s_m[sj],si);
		perBCx(s_l[sj],si);
#else
		wallBCxMir(s_r[sj],si);
		wallBCxVel(s_u[sj],si); wallBCxVel(s_v[sj],si); wallBCxVel(s_w[sj],si);
		wallBCxExt(s_t[sj],si,1.0,1.0);
		stateBoundTr(s_r[sj], s_t[sj], s_u[sj], s_v[sj], s_w[sj], s_h[sj], s_p[sj], s_m[sj], s_l[sj], si);
		wallBCxMir(s_s0[sj],si); wallBCxVel(s_s4[sj],si);  wallBCxVel(s_s8[sj],si);
#endif
	}

	__syncthreads();

	//initialize momentum RHS with stresses so that they can be added for both viscous terms and viscous heating without having to load additional terms
	uXtmp = ( 2 * sij[0][id.g] - 2./3.*s_dil[sj][si] );
	vXtmp = (     sij[1][id.g] + sij[3][id.g]  );
	wXtmp = (     sij[2][id.g] + sij[6][id.g]  );

	//adding the viscous dissipation part duidx*mu*six
	eXtmp = s_m[sj][si]*(uXtmp*s_s0[sj][si] + vXtmp*sij[1][id.g] + wXtmp*sij[2][id.g]);

	//Adding here the terms d (mu) dx * sxj; (lambda in case of h in rhse);
	derDevSharedV1x(&wrk2,s_m[sj],si); //wrk2 = d (mu) dx
    uXtmp *= wrk2;
	vXtmp *= wrk2;
	wXtmp *= wrk2;

	// viscous fluxes derivative mu*d^2ui dx^2
	derDevSharedV2x(&wrk1,s_u[sj],si);
	uXtmp = uXtmp + wrk1*s_m[sj][si];
	derDevSharedV2x(&wrk1,s_v[sj],si);
	vXtmp = uXtmp + wrk1*s_m[sj][si];
	derDevSharedV2x(&wrk1,s_w[sj],si);
	wXtmp = uXtmp + wrk1*s_m[sj][si];

	//adding the viscous dissipation part ui*(mu * d2duidx2 + dmudx * six)
	derDevSharedV2x(&wrk1,s_t[sj],si);
	eXtmp = eXtmp + s_u[sj][si]*uXtmp + s_v[sj][si]*vXtmp + s_w[sj][si]*wXtmp + wrk1*s_l[sj][si];

	derDevSharedV1x(&wrk2,s_l[sj],si); //wrk2 = d (lam) dx
	derDevSharedV1x(&wrk1,s_t[sj],si); //wrk1 = d (t) dx
	eXtmp = eXtmp + wrk1*wrk2;

	//Adding here the terms - d (ru phi) dx;

	fluxQuadSharedx(&wrk1,s_r[sj],s_u[sj],si);
	rXtmp = wrk1;
	__syncthreads();
	fluxCubeSharedx(&wrk1,s_r[sj],s_u[sj],s_u[sj],si);
	uXtmp = uXtmp + wrk1;
	__syncthreads();
	fluxCubeSharedx(&wrk1,s_r[sj],s_u[sj],s_v[sj],si);
	vXtmp = vXtmp + wrk1;
	__syncthreads();
	fluxCubeSharedx(&wrk1,s_r[sj],s_u[sj],s_w[sj],si);
	wXtmp = wXtmp + wrk1;
	__syncthreads();
	fluxCubeSharedx(&wrk1,s_r[sj],s_u[sj],s_h[sj],si);
	eXtmp = eXtmp + wrk1;
	__syncthreads();

	// pressure and dilation derivatives
	if (id.i < stencilSize) {
#if periodicX
		perBCx(s_dil[sj],si);
#else
		wallBCxDil(s_dil[sj],s_s0[sj],s_s4[sj],s_s8[sj],si);
#endif
	}
	__syncthreads();

	derDevSharedV1x(&wrk2,s_dil[sj],si);
	derDevShared1x(&wrk1 ,s_p[sj],si);
	uXtmp = uXtmp + s_m[sj][si]*wrk2/3.0     - wrk1 ;
	eXtmp = eXtmp + s_m[sj][si]*wrk2/3.0*s_u[sj][si];

	rX[id.g] = rXtmp;
	uX[id.g] = uXtmp;
	vX[id.g] = vXtmp;
	wX[id.g] = wXtmp;
	eX[id.g] = eXtmp ;
}

__global__ void RHSDeviceSharedFlxY(myprec *rY, myprec *uY, myprec *vY, myprec *wY, myprec *eY,
		myprec *r,  myprec *u,  myprec *v,  myprec *w,  myprec *h ,
		myprec *t,  myprec *p,  myprec *mu, myprec *lam,
		myprec *sij[9], myprec *dil) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidYFlx();

	int si = id.j + stencilSize;       // local i for shared memory access + halo offset
	int sj = id.tiy;                   // local j for shared memory access

	myprec rYtmp=0;
	myprec uYtmp=0;
	myprec vYtmp=0;
	myprec wYtmp=0;
	myprec eYtmp=0;

	myprec wrk1=0;
	myprec wrk2=0;

	__shared__ myprec s_r[sPencils][my+stencilSize*2];
	__shared__ myprec s_u[sPencils][my+stencilSize*2];
	__shared__ myprec s_v[sPencils][my+stencilSize*2];
	__shared__ myprec s_w[sPencils][my+stencilSize*2];
	__shared__ myprec s_h[sPencils][my+stencilSize*2];
	__shared__ myprec s_t[sPencils][my+stencilSize*2];
	__shared__ myprec s_p[sPencils][my+stencilSize*2];
	__shared__ myprec s_m[sPencils][my+stencilSize*2];
	__shared__ myprec s_l[sPencils][my+stencilSize*2];
	__shared__ myprec s_s3[sPencils][mz+stencilSize*2];
	__shared__ myprec s_s4[sPencils][mz+stencilSize*2];
	__shared__ myprec s_s5[sPencils][mz+stencilSize*2];
	__shared__ myprec s_dil[sPencils][my+stencilSize*2];

	s_r[sj][si] = r[id.g];
	s_u[sj][si] = u[id.g];
	s_v[sj][si] = v[id.g];
	s_w[sj][si] = w[id.g];
	s_h[sj][si] = h[id.g];
	s_t[sj][si] = t[id.g];
	s_p[sj][si] = p[id.g];
	s_m[sj][si] = mu[id.g];
	s_l[sj][si] = lam[id.g];
	s_dil[sj][si] = dil[id.g];
	s_s3[sj][si] = sij[3][id.g];
	s_s4[sj][si] = sij[4][id.g];
	s_s5[sj][si] = sij[5][id.g];
	__syncthreads();

	// fill in periodic images in shared memory array
	if (id.j < stencilSize) {
		perBCy(s_r[sj],si); perBCy(s_u[sj],si);
		perBCy(s_v[sj],si); perBCy(s_w[sj],si);
		perBCy(s_h[sj],si); perBCy(s_t[sj],si);
		perBCy(s_p[sj],si); perBCy(s_m[sj],si);
		perBCy(s_l[sj],si);
	}
	__syncthreads();

	//initialize momentum RHS with stresses so that they can be added for both viscous terms and viscous heating without having to load additional terms
	uYtmp =     s_s3[sj][si] + sij[1][id.g];
	vYtmp = 2 * s_s4[sj][si] - 2./3.*s_dil[sj][si];
	wYtmp =     s_s5[sj][si] + sij[7][id.g];

	//adding the viscous dissipation part duidy*mu*siy
	eYtmp = s_m[sj][si]*(uYtmp*s_s3[sj][si] + vYtmp*s_s4[sj][si] + wYtmp*s_s5[sj][si]);

	//Adding here the terms d (mu) dy * siy;
	derDevSharedV1y(&wrk2,s_m[sj],si); //wrk2 = d (mu) dx
	uYtmp *= wrk2;
	vYtmp *= wrk2;
	wYtmp *= wrk2;

	// viscous fluxes derivative mu*d^2dui dy^2
	derDevSharedV2y(&wrk1,s_u[sj],si);
	uYtmp = uYtmp + wrk1*s_m[sj][si];
	derDevSharedV2y(&wrk1,s_v[sj],si);
	vYtmp = vYtmp + wrk1*s_m[sj][si];
	derDevSharedV2y(&wrk1,s_w[sj],si);
	wYtmp = wYtmp + wrk1*s_m[sj][si];

	//adding the viscous dissipation part ui*(mu * d2duidy2 + dmudy * siy)
	derDevSharedV2y(&wrk1,s_t[sj],si);
	eYtmp = eYtmp + s_u[sj][si]*uYtmp + s_v[sj][si]*vYtmp + s_w[sj][si]*wYtmp + wrk1*s_l[sj][si];

	derDevSharedV1y(&wrk2,s_l[sj],si); //wrk2 = d (lam) dx
	derDevSharedV1y(&wrk1,s_t[sj],si); //wrk1 = d (t) dx
	eYtmp = eYtmp + wrk1*wrk2;


	// split advection terms

	//Adding here the terms - d (ru phi) dy;

	fluxQuadSharedy(&wrk1,s_r[sj],s_v[sj],si);
	rYtmp = wrk1;
	__syncthreads();
	fluxCubeSharedy(&wrk1,s_r[sj],s_v[sj],s_u[sj],si);
	uYtmp = uYtmp + wrk1;
	__syncthreads();
	fluxCubeSharedy(&wrk1,s_r[sj],s_v[sj],s_v[sj],si);
	vYtmp = vYtmp + wrk1;
	__syncthreads();
	fluxCubeSharedy(&wrk1,s_r[sj],s_v[sj],s_w[sj],si);
	wYtmp = wYtmp + wrk1;
	__syncthreads();
	fluxCubeSharedy(&wrk1,s_r[sj],s_v[sj],s_h[sj],si);
	eYtmp = eYtmp + wrk1;
	__syncthreads();

	// pressure and dilation derivatives
	if (id.j < stencilSize) {
		perBCy(s_dil[sj],si);
	}
	__syncthreads();
	derDevSharedV1y(&wrk2,s_dil[sj],si);
	derDevShared1y(&wrk1,s_p[sj],si);
	vYtmp = vYtmp + s_m[sj][si]*wrk2/3.0     - wrk1 ;
	eYtmp = eYtmp + s_m[sj][si]*wrk2/3.0*s_v[sj][si];

	rY[id.g] = rYtmp;
	uY[id.g] = uYtmp;
	vY[id.g] = vYtmp;
	wY[id.g] = wYtmp;
	eY[id.g] = eYtmp;
}

__global__ void RHSDeviceSharedFlxZ(myprec *rZ, myprec *uZ, myprec *vZ, myprec *wZ, myprec *eZ,
		myprec *r,  myprec *u,  myprec *v,  myprec *w,  myprec *h ,
		myprec *t,  myprec *p,  myprec *mu, myprec *lam,
		myprec *sij[9], myprec *dil) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidZFlx();

	int si = id.k + stencilSize;       // local i for shared memory access + halo offset
	int sj = id.tiy;                   // local j for shared memory access

	myprec rZtmp=0;
	myprec uZtmp=0;
	myprec vZtmp=0;
	myprec wZtmp=0;
	myprec eZtmp=0;

	myprec wrk1=0;
	myprec wrk2=0;

	__shared__ myprec s_r[sPencils][mz+stencilSize*2];
	__shared__ myprec s_u[sPencils][mz+stencilSize*2];
	__shared__ myprec s_v[sPencils][mz+stencilSize*2];
	__shared__ myprec s_w[sPencils][mz+stencilSize*2];
	__shared__ myprec s_h[sPencils][mz+stencilSize*2];
	__shared__ myprec s_t[sPencils][mz+stencilSize*2];
	__shared__ myprec s_p[sPencils][mz+stencilSize*2];
	__shared__ myprec s_m[sPencils][mz+stencilSize*2];
	__shared__ myprec s_l[sPencils][mz+stencilSize*2];
	__shared__ myprec s_s6[sPencils][mz+stencilSize*2];
	__shared__ myprec s_s7[sPencils][mz+stencilSize*2];
	__shared__ myprec s_s8[sPencils][mz+stencilSize*2];
	__shared__ myprec s_dil[sPencils][mz+stencilSize*2];

	s_r[sj][si] = r[id.g];
	s_u[sj][si] = u[id.g];
	s_v[sj][si] = v[id.g];
	s_w[sj][si] = w[id.g];
	s_h[sj][si] = h[id.g];
	s_t[sj][si] = t[id.g];
	s_p[sj][si] = p[id.g];
	s_m[sj][si] = mu[id.g];
	s_l[sj][si] = lam[id.g];
	s_s6[sj][si] = sij[6][id.g];
	s_s7[sj][si] = sij[7][id.g];
	s_s8[sj][si] = sij[8][id.g];
	s_dil[sj][si] = dil[id.g];
	__syncthreads();

	// fill in periodic images in shared memory array
	if (id.k < stencilSize) {
		perBCz(s_r[sj],si); perBCz(s_u[sj],si);
		perBCz(s_v[sj],si); perBCz(s_w[sj],si); perBCz(s_t[sj],si);
		perBCz(s_h[sj],si);
		perBCz(s_p[sj],si); perBCz(s_m[sj],si);
		perBCz(s_l[sj],si);
	}

	__syncthreads();

	//initialize momentum RHS with stresses so that they can be added for both viscous terms and viscous heating without having to load additional terms
	uZtmp = (    s_s6[sj][si] + sij[2][id.g]        );
	vZtmp = (    s_s7[sj][si] + sij[5][id.g]        );
	wZtmp = (2 * s_s8[sj][si] - 2./3.*s_dil[sj][si] );

	//adding the viscous dissipation part duidz*mu*siz
	eZtmp = s_m[sj][si]*(uZtmp*s_s6[sj][si] + vZtmp*s_s7[sj][si] + wZtmp*s_s8[sj][si]);

	//Adding here the terms d (mu) dz * szj;
	derDevSharedV1z(&wrk2,s_m[sj],si); //wrk2 = d (mu) dz
    uZtmp *= wrk2;
	vZtmp *= wrk2;
	wZtmp *= wrk2;

	// viscous fluxes derivative
	derDevSharedV2z(&wrk1,s_u[sj],si);
	uZtmp = wrk1*s_m[sj][si];
	derDevSharedV2z(&wrk1,s_v[sj],si);
	vZtmp = wrk1*s_m[sj][si];
	derDevSharedV2z(&wrk1,s_w[sj],si);
	wZtmp = wrk1*s_m[sj][si];

	//adding the viscous dissipation part ui*(mu * d2duidz2 + dmudz * siz)
	derDevSharedV2z(&wrk1,s_t[sj],si);
	eZtmp = eZtmp + s_u[sj][si]*uZtmp + s_v[sj][si]*vZtmp + s_w[sj][si]*wZtmp + wrk1*s_l[sj][si];

	derDevSharedV1z(&wrk2,s_l[sj],si); //wrk2 = d (lam) dz
	derDevSharedV1z(&wrk1,s_t[sj],si); //wrk1 = d (t) dx
	eZtmp = eZtmp + wrk1*wrk2;

	//Adding here the terms - d (ru phi) dz;

	fluxQuadSharedz(&wrk1,s_r[sj],s_w[sj],si);
	rZtmp = wrk1;
	__syncthreads();
	fluxCubeSharedz(&wrk1,s_r[sj],s_w[sj],s_u[sj],si);
	uZtmp = uZtmp + wrk1;
	__syncthreads();
	fluxCubeSharedz(&wrk1,s_r[sj],s_w[sj],s_v[sj],si);
	vZtmp = vZtmp + wrk1;
	__syncthreads();
	fluxCubeSharedz(&wrk1,s_r[sj],s_w[sj],s_w[sj],si);
	wZtmp = wZtmp + wrk1;
	__syncthreads();
	fluxCubeSharedz(&wrk1,s_r[sj],s_w[sj],s_h[sj],si);
	eZtmp = eZtmp + wrk1;
	__syncthreads();

	// pressure and dilation derivatives
	__syncthreads();
	if (id.k < stencilSize) {
		perBCz(s_dil[sj],si);
	}
	__syncthreads();
	derDevSharedV1z(&wrk2,s_dil[sj],si);
	derDevShared1z(&wrk1,s_p[sj],si);
	wZtmp = wZtmp + s_m[sj][si]*wrk2/3.0     - wrk1 ;
	eZtmp = eZtmp + s_m[sj][si]*wrk2/3.0*s_w[sj][si];

	rZ[id.g] = rZtmp;
	uZ[id.g] = uZtmp;
	vZ[id.g] = vZtmp;
	wZ[id.g] = wZtmp;
	eZ[id.g] = eZtmp; // + 1.0*s_w[sj][si] ;
	__syncthreads();
}



__global__ void RHSDeviceSharedFlxX_old(myprec *rX, myprec *uX, myprec *vX, myprec *wX, myprec *eX,
		myprec *r,  myprec *u,  myprec *v,  myprec *w,  myprec *h ,
		myprec *t,  myprec *p,  myprec *mu, myprec *lam,
		myprec *sij[9], myprec *dil) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidX();

	int si = id.i + stencilSize;       // local i for shared memory access + halo offset
	int sj = id.tiy;                   // local j for shared memory access

	myprec rXtmp=0;
	myprec uXtmp=0;
	myprec vXtmp=0;
	myprec wXtmp=0;
	myprec eXtmp=0;

	myprec wrk1=0;
	myprec wrk2=0;

	__shared__ myprec s_r[sPencils][mx+stencilSize*2];
	__shared__ myprec s_u[sPencils][mx+stencilSize*2];
	__shared__ myprec s_v[sPencils][mx+stencilSize*2];
	__shared__ myprec s_w[sPencils][mx+stencilSize*2];
	__shared__ myprec s_h[sPencils][mx+stencilSize*2];
	__shared__ myprec s_t[sPencils][mx+stencilSize*2];
	__shared__ myprec s_p[sPencils][mx+stencilSize*2];
	__shared__ myprec s_m[sPencils][mx+stencilSize*2];
	__shared__ myprec s_l[sPencils][mx+stencilSize*2];
	__shared__ myprec s_wrk[sPencils][mx+stencilSize*2];

	s_r[sj][si] = r[id.g];
	s_u[sj][si] = u[id.g];
	s_v[sj][si] = v[id.g];
	s_w[sj][si] = w[id.g];
	s_h[sj][si] = h[id.g];
	s_t[sj][si] = t[id.g];
	s_p[sj][si] = p[id.g];
	s_m[sj][si] = mu[id.g];
	s_l[sj][si] = lam[id.g];
	__syncthreads();

	// fill in periodic images in shared memory array
	if (id.i < stencilSize) {
#if periodicX
		perBCx(s_r[sj],si); perBCx(s_u[sj],si);
		perBCx(s_v[sj],si); perBCx(s_w[sj],si);
		perBCx(s_h[sj],si); perBCx(s_t[sj],si);
		perBCx(s_p[sj],si); perBCx(s_m[sj],si);
		perBCx(s_l[sj],si);
#else
		wallBCxMir(s_r[sj],si);
		wallBCxVel(s_u[sj],si); wallBCxVel(s_v[sj],si); wallBCxVel(s_w[sj],si);
		wallBCxExt(s_t[sj],si,1.0,1.0);
		stateBoundTr(s_r[sj], s_t[sj], s_u[sj], s_v[sj], s_w[sj], s_h[sj], s_p[sj], s_m[sj], s_l[sj], si);
#endif
	}

	__syncthreads();

	// viscous fluxes derivative
	derDevSharedV2x(&wrk1,s_u[sj],si);
	uXtmp = wrk1*s_m[sj][si];
	derDevSharedV2x(&wrk1,s_v[sj],si);
	vXtmp = wrk1*s_m[sj][si];
	derDevSharedV2x(&wrk1,s_w[sj],si);
	wXtmp = wrk1*s_m[sj][si];
	derDevSharedV2x(&wrk1,s_t[sj],si);
	eXtmp = wrk1*s_l[sj][si];
	__syncthreads();
	derDevSharedV1x(&wrk2,s_l[sj],si); //wrk2 = d (lam) dx
	derDevSharedV1x(&wrk1,s_t[sj],si); //wrk1 = d (t) dx
	eXtmp = eXtmp + wrk1*wrk2;

	//Adding here the terms d (mu) dx * sxj; (lambda in case of h in rhse);

	derDevSharedV1x(&wrk2,s_m[sj],si); //wrk2 = d (mu) dx
	uXtmp = uXtmp + wrk2*sij[0][id.g];
	vXtmp = vXtmp + wrk2*sij[1][id.g];
	wXtmp = wXtmp + wrk2*sij[2][id.g];

	// split advection terms

	//Adding here the terms - d (ru phi) dx;

	fluxQuadSharedx(&wrk1,s_r[sj],s_u[sj],si);
	rXtmp = wrk1;
	__syncthreads();
	fluxCubeSharedx(&wrk1,s_r[sj],s_u[sj],s_u[sj],si);
	uXtmp = uXtmp + wrk1;
	__syncthreads();
	fluxCubeSharedx(&wrk1,s_r[sj],s_u[sj],s_v[sj],si);
	vXtmp = vXtmp + wrk1;
	__syncthreads();
	fluxCubeSharedx(&wrk1,s_r[sj],s_u[sj],s_w[sj],si);
	wXtmp = wXtmp + wrk1;
	__syncthreads();
	fluxCubeSharedx(&wrk1,s_r[sj],s_u[sj],s_h[sj],si);
	eXtmp = eXtmp + wrk1;
	__syncthreads();

	// pressure and dilation derivatives
	s_wrk[sj][si] = dil[id.g];
	__syncthreads();

	if (id.i < stencilSize) {
#if periodicX
		perBCx(s_wrk[sj],si);
#else
		wallBCxMir(s_wrk[sj],si);
#endif
	}
	__syncthreads();

	derDevSharedV1x(&wrk2,s_wrk[sj],si);
	derDevShared1x(&wrk1,s_p[sj],si);
	uXtmp = uXtmp - wrk1 + s_m[sj][si]*wrk2*1.0/3.0;

	//viscous dissipation
	s_wrk[sj][si] = s_m[sj][si]*(
					s_u[sj][si]*(  sij[0][id.g]  ) +
					s_v[sj][si]*(  sij[1][id.g]  ) +
					s_w[sj][si]*(  sij[2][id.g]  )
					);
	__syncthreads();

	if (id.i < stencilSize) {
#if periodicX
		perBCx(s_wrk[sj],si);
#else
		wallBCxMir(s_wrk[sj],si);
#endif
	}
	__syncthreads();

	derDevSharedV1x(&wrk2,s_wrk[sj],si);

	rX[id.g] = rXtmp;
	uX[id.g] = uXtmp;
	vX[id.g] = vXtmp;
	wX[id.g] = wXtmp;
	eX[id.g] = eXtmp + wrk2;
}

__global__ void RHSDeviceSharedFlxY_old(myprec *rY, myprec *uY, myprec *vY, myprec *wY, myprec *eY,
		myprec *r,  myprec *u,  myprec *v,  myprec *w,  myprec *h ,
		myprec *t,  myprec *p,  myprec *mu, myprec *lam,
		myprec *sij[9], myprec *dil) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidYFlx();

	int si = id.j + stencilSize;       // local i for shared memory access + halo offset
	int sj = id.tiy;                   // local j for shared memory access

	myprec rYtmp=0;
	myprec uYtmp=0;
	myprec vYtmp=0;
	myprec wYtmp=0;
	myprec eYtmp=0;

	myprec wrk1=0;
	myprec wrk2=0;

	__shared__ myprec s_r[sPencils][my+stencilSize*2];
	__shared__ myprec s_u[sPencils][my+stencilSize*2];
	__shared__ myprec s_v[sPencils][my+stencilSize*2];
	__shared__ myprec s_w[sPencils][my+stencilSize*2];
	__shared__ myprec s_h[sPencils][my+stencilSize*2];
	__shared__ myprec s_t[sPencils][my+stencilSize*2];
	__shared__ myprec s_p[sPencils][my+stencilSize*2];
	__shared__ myprec s_m[sPencils][my+stencilSize*2];
	__shared__ myprec s_l[sPencils][my+stencilSize*2];
	__shared__ myprec s_s1[sPencils][mz+stencilSize*2];
	__shared__ myprec s_s2[sPencils][mz+stencilSize*2];
	__shared__ myprec s_s3[sPencils][mz+stencilSize*2];
	__shared__ myprec s_wrk[sPencils][my+stencilSize*2];

	s_r[sj][si] = r[id.g];
	s_u[sj][si] = u[id.g];
	s_v[sj][si] = v[id.g];
	s_w[sj][si] = w[id.g];
	s_h[sj][si] = h[id.g];
	s_t[sj][si] = t[id.g];
	s_p[sj][si] = p[id.g];
	s_m[sj][si] = mu[id.g];
	s_s1[sj][si] = sij[3][id.g];
	s_s2[sj][si] = sij[4][id.g];
	s_s3[sj][si] = sij[5][id.g];
	s_l[sj][si] = lam[id.g];
	__syncthreads();

	// fill in periodic images in shared memory array
	if (id.j < stencilSize) {
		perBCy(s_r[sj],si); perBCy(s_u[sj],si);
		perBCy(s_v[sj],si); perBCy(s_w[sj],si);
		perBCy(s_h[sj],si); perBCy(s_t[sj],si);
		perBCy(s_p[sj],si); perBCy(s_m[sj],si);
		perBCy(s_l[sj],si);
	}

	__syncthreads();

	// viscous fluxes derivative
	derDevSharedV2y(&wrk1,s_u[sj],si);
	uYtmp = wrk1*s_m[sj][si];
	derDevSharedV2y(&wrk1,s_v[sj],si);
	vYtmp = wrk1*s_m[sj][si];
	derDevSharedV2y(&wrk1,s_w[sj],si);
	wYtmp = wrk1*s_m[sj][si];
	derDevSharedV2y(&wrk1,s_t[sj],si);
	eYtmp = wrk1*s_l[sj][si];
	__syncthreads();
	derDevSharedV1y(&wrk2,s_l[sj],si); //wrk2 = d (lam) dx
	derDevSharedV1y(&wrk1,s_t[sj],si); //wrk1 = d (t) dx
	eYtmp = eYtmp + wrk1*wrk2;

	//Adding here the terms d (mu) dy * syj; (lambda in case of h in rhse);

	derDevSharedV1y(&wrk2,s_m[sj],si); //wrk2 = d (mu) dx
	uYtmp = uYtmp + wrk2*s_s1[sj][si];
	vYtmp = vYtmp + wrk2*s_s2[sj][si];
	wYtmp = wYtmp + wrk2*s_s3[sj][si];

	// split advection terms

	//Adding here the terms - d (ru phi) dy;

	fluxQuadSharedy(&wrk1,s_r[sj],s_v[sj],si);
	rYtmp = wrk1;
	__syncthreads();
	fluxCubeSharedy(&wrk1,s_r[sj],s_v[sj],s_u[sj],si);
	uYtmp = uYtmp + wrk1;
	__syncthreads();
	fluxCubeSharedy(&wrk1,s_r[sj],s_v[sj],s_v[sj],si);
	vYtmp = vYtmp + wrk1;
	__syncthreads();
	fluxCubeSharedy(&wrk1,s_r[sj],s_v[sj],s_w[sj],si);
	wYtmp = wYtmp + wrk1;
	__syncthreads();
	fluxCubeSharedy(&wrk1,s_r[sj],s_v[sj],s_h[sj],si);
	eYtmp = eYtmp + wrk1;
	__syncthreads();

	// pressure and dilation derivatives
	s_wrk[sj][si] = dil[id.g];
	__syncthreads();
	if (id.j < stencilSize) {
		perBCy(s_wrk[sj],si);
	}
	__syncthreads();
	derDevSharedV1y(&wrk2,s_wrk[sj],si);
	derDevShared1y(&wrk1,s_p[sj],si);
	vYtmp = vYtmp - wrk1 + s_m[sj][si]*wrk2*1.0/3.0;

	//viscous dissipation
	s_wrk[sj][si] = s_m[sj][si]*(
					s_u[sj][si]*(  s_s1[sj][si]  ) +
					s_v[sj][si]*(  s_s2[sj][si]  ) +
					s_w[sj][si]*(  s_s3[sj][si]  )
					);
	__syncthreads();
	if (id.j < stencilSize) {
		perBCy(s_wrk[sj],si);
	}
	__syncthreads();
	derDevSharedV1y(&wrk2,s_wrk[sj],si);

	rY[id.g] = rYtmp;
	uY[id.g] = uYtmp;
	vY[id.g] = vYtmp;
	wY[id.g] = wYtmp;
	eY[id.g] = eYtmp + wrk2;
}

__global__ void RHSDeviceSharedFlxZ_old(myprec *rZ, myprec *uZ, myprec *vZ, myprec *wZ, myprec *eZ,
		myprec *r,  myprec *u,  myprec *v,  myprec *w,  myprec *h ,
		myprec *t,  myprec *p,  myprec *mu, myprec *lam,
		myprec *sij[9], myprec *dil) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidZFlx();

	int si = id.k + stencilSize;       // local i for shared memory access + halo offset
	int sj = id.tiy;                   // local j for shared memory access

	myprec rZtmp=0;
	myprec uZtmp=0;
	myprec vZtmp=0;
	myprec wZtmp=0;
	myprec eZtmp=0;

	myprec wrk1=0;
	myprec wrk2=0;

	__shared__ myprec s_r[sPencils][mz+stencilSize*2];
	__shared__ myprec s_u[sPencils][mz+stencilSize*2];
	__shared__ myprec s_v[sPencils][mz+stencilSize*2];
	__shared__ myprec s_w[sPencils][mz+stencilSize*2];
	__shared__ myprec s_h[sPencils][mz+stencilSize*2];
	__shared__ myprec s_t[sPencils][mz+stencilSize*2];
	__shared__ myprec s_p[sPencils][mz+stencilSize*2];
	__shared__ myprec s_m[sPencils][mz+stencilSize*2];
	__shared__ myprec s_l[sPencils][mz+stencilSize*2];
	__shared__ myprec s_s1[sPencils][mz+stencilSize*2];
	__shared__ myprec s_s2[sPencils][mz+stencilSize*2];
	__shared__ myprec s_s3[sPencils][mz+stencilSize*2];
	__shared__ myprec s_wrk[sPencils][mz+stencilSize*2];

	s_r[sj][si] = r[id.g];
	s_u[sj][si] = u[id.g];
	s_v[sj][si] = v[id.g];
	s_w[sj][si] = w[id.g];
	s_h[sj][si] = h[id.g];
	s_t[sj][si] = t[id.g];
	s_p[sj][si] = p[id.g];
	s_m[sj][si] = mu[id.g];
	s_l[sj][si] = lam[id.g];
	s_s1[sj][si] = sij[6][id.g];
	s_s2[sj][si] = sij[7][id.g];
	s_s3[sj][si] = sij[8][id.g];
	__syncthreads();

	// fill in periodic images in shared memory array
	if (id.k < stencilSize) {
		perBCz(s_r[sj],si); perBCz(s_u[sj],si);
		perBCz(s_v[sj],si); perBCz(s_w[sj],si);
		perBCz(s_t[sj],si); perBCz(s_h[sj],si);
		perBCz(s_p[sj],si); perBCz(s_m[sj],si);
		perBCz(s_l[sj],si);
	}

	__syncthreads();

	// viscous fluxes derivative
	derDevSharedV2z(&wrk1,s_u[sj],si);
	uZtmp = wrk1*s_m[sj][si];
	derDevSharedV2z(&wrk1,s_v[sj],si);
	vZtmp = wrk1*s_m[sj][si];
	derDevSharedV2z(&wrk1,s_w[sj],si);
	wZtmp = wrk1*s_m[sj][si];
	derDevSharedV2z(&wrk1,s_t[sj],si);
	eZtmp = wrk1*s_l[sj][si];
	__syncthreads();
	derDevSharedV1z(&wrk2,s_l[sj],si); //wrk2 = d (lam) dz
	derDevSharedV1z(&wrk1,s_t[sj],si); //wrk1 = d (t) dx
	eZtmp = eZtmp + wrk1*wrk2;

	//Adding here the terms d (mu) dz * szj; (lambda in case of h in rhse);

	derDevSharedV1z(&wrk2,s_m[sj],si); //wrk2 = d (mu) dz
	uZtmp = uZtmp + wrk2*s_s1[sj][si];
	vZtmp = vZtmp + wrk2*s_s2[sj][si];
	wZtmp = wZtmp + wrk2*s_s3[sj][si];

	// split advection terms

	//Adding here the terms - d (ru phi) dz;

	fluxQuadSharedz(&wrk1,s_r[sj],s_w[sj],si);
	rZtmp = wrk1;
	__syncthreads();
	fluxCubeSharedz(&wrk1,s_r[sj],s_w[sj],s_u[sj],si);
	uZtmp = uZtmp + wrk1;
	__syncthreads();
	fluxCubeSharedz(&wrk1,s_r[sj],s_w[sj],s_v[sj],si);
	vZtmp = vZtmp + wrk1;
	__syncthreads();
	fluxCubeSharedz(&wrk1,s_r[sj],s_w[sj],s_w[sj],si);
	wZtmp = wZtmp + wrk1;
	__syncthreads();
	fluxCubeSharedz(&wrk1,s_r[sj],s_w[sj],s_h[sj],si);
	eZtmp = eZtmp + wrk1;
	__syncthreads();

	// pressure and dilation derivatives
	s_wrk[sj][si] = dil[id.g];
	__syncthreads();
	if (id.k < stencilSize) {
		perBCz(s_wrk[sj],si);
	}
	__syncthreads();
	derDevSharedV1z(&wrk2,s_wrk[sj],si);
	derDevShared1z(&wrk1,s_p[sj],si);
	wZtmp = wZtmp - wrk1 + s_m[sj][si]*wrk2*1.0/3.0;

	//viscous dissipation
	s_wrk[sj][si] = s_m[sj][si]*(
					s_u[sj][si]*(  s_s1[sj][si]  ) +
					s_v[sj][si]*(  s_s2[sj][si]  ) +
					s_w[sj][si]*(  s_s3[sj][si]  )
					);
	__syncthreads();
	if (id.k < stencilSize) {
		perBCz(s_wrk[sj],si);
	}
	__syncthreads();
	derDevSharedV1z(&wrk2,s_wrk[sj],si);

	rZ[id.g] = rZtmp;
	uZ[id.g] = uZtmp;
	vZ[id.g] = vZtmp;
	wZ[id.g] = wZtmp;
	eZ[id.g] = eZtmp + wrk2;
}

