#include "hip/hip_runtime.h"
/* To run the debugger!!
 * CUDA_VISIBLE_DEVICES="0" cuda-gdb -tui ns
 *  */

#include "globals.h"
#include "cuda_functions.h"
#include "cuda_globals.h"

__constant__ myprec dcoeffF[stencilSize];
__constant__ myprec dcoeffS[stencilSize+1];
__constant__ myprec dcoeffVF[stencilVisc];
__constant__ myprec dcoeffVS[stencilVisc+1];
__constant__ myprec d_dt, d_dx, d_dy, d_dz, d_d2x, d_d2y, d_d2z;
#if (capability>60)
__constant__ dim3 d_block[5], grid0;
__constant__ dim3 d_grid[5], block0;
#else
__constant__ int d_block[5*2], grid0[2];
__constant__ int d_grid[5*2], block0[2];
#endif

dim3 hgrid, hblock;

// host routine to set constant data

#if (capability>60)
void setDerivativeParameters(dim3 &grid, dim3 &block)
{

  // check to make sure dimensions are integral multiples of sPencils
  if ((mx % sPencils != 0) || (my %sPencils != 0) || (mz % sPencils != 0)) {
    printf("'mx', 'my', and 'mz' must be integral multiples of sPencils\n");
    exit(1);
  }

  myprec h_dt = (myprec) dt;
  myprec h_dx = (myprec) 1.0/(x[1] - x[0]);
  myprec h_dy = (myprec) 1.0/(y[1] - y[0]);
  myprec h_dz = (myprec) 1.0/(z[1] - z[0]);

  myprec h_d2x = h_dx*h_dx;
  myprec h_d2y = h_dy*h_dy;
  myprec h_d2z = h_dz*h_dz;

  myprec *h_coeffF  = new myprec[stencilSize];
  myprec *h_coeffS  = new myprec[stencilSize+1];
  myprec *h_coeffVF = new myprec[stencilSize];
  myprec *h_coeffVS = new myprec[stencilSize+1];

  for (int it=0; it<stencilSize; it++) {
	  h_coeffF[it]  = (myprec) coeffF[it]; }
  for (int it=0; it<stencilVisc; it++) {
	  h_coeffVF[it] = (myprec) coeffVF[it]; }
  for (int it=0; it<stencilSize+1; it++) {
	  h_coeffS[it]  = (myprec) coeffS[it]; }
  for (int it=0; it<stencilVisc+1; it++) {
	  h_coeffVS[it] = (myprec) coeffVS[it]; }

  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(dcoeffF ), h_coeffF ,  stencilSize   *sizeof(myprec), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(dcoeffS ), h_coeffS , (stencilSize+1)*sizeof(myprec), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(dcoeffVF), h_coeffVF,  stencilVisc   *sizeof(myprec), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(dcoeffVS), h_coeffVS, (stencilVisc+1)*sizeof(myprec), 0, hipMemcpyHostToDevice) );


  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_dt  ), &h_dt  ,   sizeof(myprec), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_dx  ), &h_dx  ,   sizeof(myprec), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_dy  ), &h_dy  ,   sizeof(myprec), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_dz  ), &h_dz  ,   sizeof(myprec), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_d2x ), &h_d2x ,   sizeof(myprec), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_d2y ), &h_d2y ,   sizeof(myprec), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_d2z ), &h_d2z ,   sizeof(myprec), 0, hipMemcpyHostToDevice) );

  dim3 *h_grid, *h_block;
  h_grid  = new dim3[5];
  h_block = new dim3[5];


  // X-grid
  h_grid[0]  = dim3(my / sPencils, mz, 1);
  h_block[0] = dim3(mx, sPencils, 1);

  // Y-grid (2) for viscous fluxes and (4) for advective fluxes
  h_grid[1]  = dim3(mx / lPencils, mz, 1);
  h_block[1] = dim3(lPencils, (my * sPencils) / lPencils, 1);

  h_grid[3]  = dim3(mx / sPencils, mz, 1);
  h_block[3] = dim3(my , sPencils, 1); //if not using shared change!!

  // Z-grid (2) for viscous fluxes and (4) for advective fluxes
  h_grid[2]  = dim3(mx / lPencils, my, 1);
  h_block[2] = dim3(lPencils, (mz * sPencils) / lPencils, 1);

  h_grid[4]  = dim3(mx / sPencils, my, 1);
  h_block[4] = dim3(mz , sPencils, 1); //if not using shared change!!


  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_grid  ), h_grid  , 5*sizeof(dim3), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_block ), h_block , 5*sizeof(dim3), 0, hipMemcpyHostToDevice) );

  printf("Grid configuration:\n");
  printf("Grid 0: {%d, %d, %d} blocks. Blocks 0: {%d, %d, %d} threads.\n",h_grid[0].x, h_grid[0].y, h_grid[0].z, h_block[0].x, h_block[0].y, h_block[0].z);
  printf("Grid 1: {%d, %d, %d} blocks. Blocks 1: {%d, %d, %d} threads.\n",h_grid[1].x, h_grid[1].y, h_grid[1].z, h_block[1].x, h_block[1].y, h_block[1].z);
  printf("Grid 2: {%d, %d, %d} blocks. Blocks 2: {%d, %d, %d} threads.\n",h_grid[2].x, h_grid[2].y, h_grid[2].z, h_block[2].x, h_block[2].y, h_block[2].z);
  printf("Grid 3: {%d, %d, %d} blocks. Blocks 1: {%d, %d, %d} threads.\n",h_grid[3].x, h_grid[3].y, h_grid[3].z, h_block[3].x, h_block[3].y, h_block[3].z);
  printf("Grid 4: {%d, %d, %d} blocks. Blocks 2: {%d, %d, %d} threads.\n",h_grid[4].x, h_grid[4].y, h_grid[4].z, h_block[4].x, h_block[4].y, h_block[4].z);
  printf("\n");

  hgrid  = dim3(my / sPencils, mz, 1);
  hblock = dim3(mx, sPencils, 1);

  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(grid0  ), &hgrid  , sizeof(dim3), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(block0 ), &hblock , sizeof(dim3), 0, hipMemcpyHostToDevice) );

  grid  = 1;
  block = 1;

  delete [] h_coeffF;
  delete [] h_coeffS;
  delete [] h_coeffVF;
  delete [] h_coeffVS;
  delete [] h_grid;
  delete [] h_block;

}
#else
void setDerivativeParameters(dim3 &grid, dim3 &block)
{

  // check to make sure dimensions are integral multiples of sPencils
  if ((mx % sPencils != 0) || (my %sPencils != 0) || (mz % sPencils != 0)) {
    printf("'mx', 'my', and 'mz' must be integral multiples of sPencils\n");
    exit(1);
  }

  myprec h_dt = (myprec) dt;
  myprec h_dx = (myprec) 1.0/(x[1] - x[0]);
  myprec h_dy = (myprec) 1.0/(y[1] - y[0]);
  myprec h_dz = (myprec) 1.0/(z[1] - z[0]);

  myprec h_d2x = h_dx*h_dx;
  myprec h_d2y = h_dy*h_dy;
  myprec h_d2z = h_dz*h_dz;

  myprec *h_coeffF  = new myprec[stencilSize];
  myprec *h_coeffS  = new myprec[stencilSize+1];
  myprec *h_coeffVF = new myprec[stencilSize];
  myprec *h_coeffVS = new myprec[stencilSize+1];

  for (int it=0; it<stencilSize; it++) {
	  h_coeffF[it]  = (myprec) coeffF[it]; }
  for (int it=0; it<stencilVisc; it++) {
	  h_coeffVF[it] = (myprec) coeffVF[it]; }
  for (int it=0; it<stencilSize+1; it++) {
	  h_coeffS[it]  = (myprec) coeffS[it]; }
  for (int it=0; it<stencilVisc+1; it++) {
	  h_coeffVS[it] = (myprec) coeffVS[it]; }

  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(dcoeffF ), h_coeffF ,  stencilSize   *sizeof(myprec), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(dcoeffS ), h_coeffS , (stencilSize+1)*sizeof(myprec), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(dcoeffVF), h_coeffVF,  stencilVisc   *sizeof(myprec), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(dcoeffVS), h_coeffVS, (stencilVisc+1)*sizeof(myprec), 0, hipMemcpyHostToDevice) );


  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_dt  ), &h_dt  ,   sizeof(myprec), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_dx  ), &h_dx  ,   sizeof(myprec), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_dy  ), &h_dy  ,   sizeof(myprec), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_dz  ), &h_dz  ,   sizeof(myprec), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_d2x ), &h_d2x ,   sizeof(myprec), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_d2y ), &h_d2y ,   sizeof(myprec), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_d2z ), &h_d2z ,   sizeof(myprec), 0, hipMemcpyHostToDevice) );

  int *h_grid, *h_block;
  h_grid  = new int[2*5];
  h_block = new int[2*5];


  // X-grid
  h_grid[0]  = my / sPencils; h_grid[1]    =  mz;
  h_block[0] = mx;            h_block[1]   =  sPencils;

  // Y-grid (2) for viscous fluxes and (4) for advective fluxes
  h_grid[2]  = mx / lPencils; h_grid[3]    =  mz;
  h_block[2] = lPencils;      h_block[3]   =  (my * sPencils) / lPencils;

  h_grid[4]  = mx / sPencils; h_grid[5]    =  mz;
  h_block[4] = my;            h_block[5]   =  sPencils;

  // Z-grid (2) for viscous fluxes and (4) for advective fluxes
  h_grid[6]  = mx / lPencils; h_grid[7]    =  my;
  h_block[6] = lPencils;      h_block[7]   =  (mz * sPencils) / lPencils;

  h_grid[8]  = mx / sPencils; h_grid[9]    =  my;
  h_block[8] = mz;            h_block[9]   =  sPencils;

  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_grid  ), h_grid  , 2*5*sizeof(int), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(d_block ), h_block , 2*5*sizeof(int), 0, hipMemcpyHostToDevice) );

  printf("Grid configuration:\n");
  printf("Grid 0: {%d, %d, %d} blocks. Blocks 0: {%d, %d, %d} threads.\n",h_grid[0], h_grid[1], 1, h_block[0], h_block[1], 1);
  printf("Grid 1: {%d, %d, %d} blocks. Blocks 1: {%d, %d, %d} threads.\n",h_grid[2], h_grid[3], 1, h_block[2], h_block[3], 1);
  printf("Grid 2: {%d, %d, %d} blocks. Blocks 2: {%d, %d, %d} threads.\n",h_grid[6], h_grid[7], 1, h_block[6], h_block[7], 1);
  printf("Grid 3: {%d, %d, %d} blocks. Blocks 1: {%d, %d, %d} threads.\n",h_grid[4], h_grid[5], 1, h_block[4], h_block[5], 1);
  printf("Grid 4: {%d, %d, %d} blocks. Blocks 2: {%d, %d, %d} threads.\n",h_grid[8], h_grid[9], 1, h_block[8], h_block[9], 1);
  printf("\n");

  hgrid  = dim3(my / sPencils, mz, 1);
  hblock = dim3(mx, sPencils, 1);

  int *h_grid0,*h_block0;
  h_grid0  = new int[2];
  h_block0 = new int[2];

  h_grid0[0]  = h_grid[0] ; h_grid0[1]  = h_grid[1];
  h_block0[0] = h_block[0]; h_block0[1] = h_block[1];
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(grid0  ), h_grid0  , 2*sizeof(int), 0, hipMemcpyHostToDevice) );
  checkCuda( hipMemcpyToSymbol(HIP_SYMBOL(block0 ), h_block0 , 2*sizeof(int), 0, hipMemcpyHostToDevice) );

  grid  = 1;
  block = 1;

  delete [] h_coeffF;
  delete [] h_coeffS;
  delete [] h_coeffVF;
  delete [] h_coeffVS;
  delete [] h_grid;
  delete [] h_block;
  delete [] h_grid0;
  delete [] h_block0;

}
#endif


void copyField(int direction) {

  myprec *fr  = new myprec[mx*my*mz];
  myprec *fu  = new myprec[mx*my*mz];
  myprec *fv  = new myprec[mx*my*mz];
  myprec *fw  = new myprec[mx*my*mz];
  myprec *fe  = new myprec[mx*my*mz];
  myprec *d_fr, *d_fu, *d_fv, *d_fw, *d_fe;
  int bytes = mx*my*mz * sizeof(myprec);
  checkCuda( hipMalloc((void**)&d_fr, bytes) );
  checkCuda( hipMalloc((void**)&d_fu, bytes) );
  checkCuda( hipMalloc((void**)&d_fv, bytes) );
  checkCuda( hipMalloc((void**)&d_fw, bytes) );
  checkCuda( hipMalloc((void**)&d_fe, bytes) );


  if(direction == 0) {

     for (int it=0; it<mx*my*mz; it++)  {
      fr[it] = (myprec) r[it];
      fu[it] = (myprec) u[it];
      fv[it] = (myprec) v[it];
      fw[it] = (myprec) w[it];
      fe[it] = (myprec) e[it];
     }

     // device arrays
     checkCuda( hipMemcpy(d_fr, fr, bytes, hipMemcpyHostToDevice) );  
     checkCuda( hipMemcpy(d_fu, fu, bytes, hipMemcpyHostToDevice) );  
     checkCuda( hipMemcpy(d_fv, fv, bytes, hipMemcpyHostToDevice) );  
     checkCuda( hipMemcpy(d_fw, fw, bytes, hipMemcpyHostToDevice) );  
     checkCuda( hipMemcpy(d_fe, fe, bytes, hipMemcpyHostToDevice) );  

     initDevice<<<hgrid, hblock>>>(d_fr,d_fu,d_fv,d_fw,d_fe);

  } else {

     checkCuda( hipMemset(d_fr, 0, bytes) );
     checkCuda( hipMemset(d_fu, 0, bytes) );
     checkCuda( hipMemset(d_fv, 0, bytes) );
     checkCuda( hipMemset(d_fw, 0, bytes) );
     checkCuda( hipMemset(d_fe, 0, bytes) );

     getResults<<<hgrid, hblock>>>(d_fr,d_fu,d_fv,d_fw,d_fe);

     checkCuda( hipMemcpy(fr, d_fr, bytes, hipMemcpyDeviceToHost) );
     checkCuda( hipMemcpy(fu, d_fu, bytes, hipMemcpyDeviceToHost) );
     checkCuda( hipMemcpy(fv, d_fv, bytes, hipMemcpyDeviceToHost) );
     checkCuda( hipMemcpy(fw, d_fw, bytes, hipMemcpyDeviceToHost) );
     checkCuda( hipMemcpy(fe, d_fe, bytes, hipMemcpyDeviceToHost) );

     for (int it=0; it<mx*my*mz; it++)  {
      r[it]   = (double) fr[it];
      u[it]   = (double) fu[it];
      v[it]   = (double) fv[it];
      w[it]   = (double) fw[it];
      e[it]   = (double) fe[it];
     }
 
  }
  
  checkCuda( hipFree(d_fr) );
  checkCuda( hipFree(d_fu) );
  checkCuda( hipFree(d_fv) );
  checkCuda( hipFree(d_fw) );
  checkCuda( hipFree(d_fe) );
  delete []  fr;  
  delete []  fu;  
  delete []  fv;  
  delete []  fw;  
  delete []  fe;  

}

__global__ void initDevice(myprec *d_fr, myprec *d_fu, myprec *d_fv, myprec *d_fw, myprec *d_fe) {

	int threadsPerBlock  = blockDim.x * blockDim.y;
	int threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
	int blockNumInGrid   = blockIdx.x  + gridDim.x  * blockIdx.y;

	int globalThreadNum = blockNumInGrid * threadsPerBlock + threadNumInBlock;

	d_r[globalThreadNum]   = d_fr[globalThreadNum];
	d_u[globalThreadNum]   = d_fu[globalThreadNum];
	d_v[globalThreadNum]   = d_fv[globalThreadNum];
	d_w[globalThreadNum]   = d_fw[globalThreadNum];
	d_e[globalThreadNum]   = d_fe[globalThreadNum];
}

__global__ void getResults(myprec *d_fr, myprec *d_fu, myprec *d_fv, myprec *d_fw, myprec *d_fe) {

	int threadsPerBlock  = blockDim.x * blockDim.y;
	int threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
	int blockNumInGrid   = blockIdx.x  + gridDim.x  * blockIdx.y;

	int globalThreadNum = blockNumInGrid * threadsPerBlock + threadNumInBlock;

	d_fr[globalThreadNum] = d_r[globalThreadNum];
	d_fu[globalThreadNum] = d_u[globalThreadNum];
	d_fv[globalThreadNum] = d_v[globalThreadNum];
	d_fw[globalThreadNum] = d_w[globalThreadNum];
	d_fe[globalThreadNum] = d_e[globalThreadNum];
}

void checkGpuMem() {

	float free_m;
	size_t free_t,total_t;

	hipMemGetInfo(&free_t,&total_t);

	free_m =(uint)free_t/1048576.0 ;

	printf ( "mem free %zu\t (%f MB mem)\n",free_t,free_m);

}

__device__ void threadBlockDeviceSynchronize(void) {
  __syncthreads();
  if(threadIdx.x == 0)
    hipDeviceSynchronize();
  __syncthreads();
}
