#include "hip/hip_runtime.h"
#include "globals.h"
#include "cuda_functions.h"
#include "cuda_globals.h"


__device__ myprec d_phi[mx*my*mz];

__device__ myprec d_rhs1[mx*my*mz];
__device__ myprec d_rhs2[mx*my*mz];
__device__ myprec d_rhs3[mx*my*mz];
__device__ myprec d_rhs4[mx*my*mz];
__device__ myprec d_temp[mx*my*mz];
__device__ myprec d_tmp[mx*my*mz];

__device__ void derDev2x(myprec *d2f, myprec *f, Indices id)
{  

  int si = id.i + stencilSize;       // local i for shared memory access + halo offset
  int sj = id.tiy;                   // local j for shared memory access

  __shared__ myprec s_f[sPencils][mx+stencilSize*2]; // 4-wide halo

  myprec d2x = 1.0/d_dx/d_dx;

  s_f[sj][si] = f[id.g];

  __syncthreads();

  // fill in periodic images in shared memory array 
  if (id.i < stencilSize) {
    s_f[sj][si-stencilSize]  = s_f[sj][si+mx-stencilSize]; // CHANGED SIMONE: s_f[sj][si+mx-stencilSize-1];
    s_f[sj][si+mx]           = s_f[sj][si];                // CHANGED SIMONE: s_f[sj][si+1];
  }

  __syncthreads();

  myprec dftemp = dcoeffS[stencilSize]*s_f[sj][si]*d2x;
  for (int it=0; it<stencilSize; it++)  { 
   dftemp += dcoeffS[it]*(s_f[sj][si+it-stencilSize]-s_f[sj][si+stencilSize-it])*d2x;
  }

  __syncthreads();
 
  d2f[id.g] = dftemp; 
}


__device__ void derDev1x(myprec *df, myprec *f, Indices id)
{  

  int si = id.i + stencilSize;       // local i for shared memory access + halo offset
  int sj = id.tiy;                   // local j for shared memory access

  __shared__ myprec s_f[sPencils][mx+stencilSize*2]; // 4-wide halo

  s_f[sj][si] = f[id.g];

  __syncthreads();

  // fill in periodic images in shared memory array 
  if (id.i < stencilSize) {
    s_f[sj][si-stencilSize]  = s_f[sj][si+mx-stencilSize]; // CHANGED SIMONE: s_f[sj][si+mx-stencilSize-1];
    s_f[sj][si+mx]           = s_f[sj][si];                // CHANGED SIMONE: s_f[sj][si+1];
  }

  __syncthreads();

  myprec dftemp = 0.0;
  for (int it=0; it<stencilSize; it++)  { 
   dftemp += dcoeffF[it]*(s_f[sj][si+it-stencilSize]-s_f[sj][si+stencilSize-it])/d_dx;
  }

  __syncthreads();
 
  df[id.g] = dftemp; 
}


__device__ void derDev1y(myprec *df, myprec *f, Indices id)
{
  __shared__ myprec s_f[my+stencilSize*2][sPencils];

  int si = id.tix;
  int sj = id.j + stencilSize;

  s_f[sj][si] = f[id.g];

  __syncthreads();

  if (id.j < stencilSize) {
    s_f[sj-stencilSize][si]  = s_f[sj+my-stencilSize][si];
    s_f[sj+my][si]           = s_f[sj][si];
  }

  __syncthreads();

  myprec dftemp = 0.0;
  for (int jt=0; jt<stencilSize; jt++)  { 
   dftemp += dcoeffF[jt]*(s_f[sj+jt-stencilSize][si]-s_f[sj+stencilSize-jt][si])/d_dy;
  }

  __syncthreads();
 
  df[id.g] = dftemp;
}


__device__ void RHSDevice(myprec *rhs, myprec *var, Indices id) {
  
  derDev1y(d_tmp,var,id);
  rhs[id.g] = -d_tmp[id.g]*U;
}


__device__ void rk4Device(Indices id) {

  RHSDevice(d_rhs1,d_phi,id); 

  d_temp[id.g] = (d_phi[id.g] + d_rhs1[id.g]*d_dt/2);
  RHSDevice(d_rhs2,d_temp,id); 
  

  d_temp[id.g] = (d_phi[id.g] + d_rhs2[id.g]*d_dt/2);
  RHSDevice(d_rhs3,d_temp,id);

  d_temp[id.g] = (d_phi[id.g] + d_rhs3[id.g]*d_dt);
  RHSDevice(d_rhs4,d_temp,id);


 
  d_phi[id.g] = d_phi[id.g] + d_dt*
                              ( d_rhs1[id.g] +
			      2*d_rhs2[id.g] + 	
			      2*d_rhs3[id.g] + 	
			        d_rhs4[id.g])/6.; 	

}


__global__ void runDevice() {

  Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);


  id.mkidY();

  for (int istep=0; istep < nsteps; istep++) {
          rk4Device(id);
  }
}
