#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "cuda_functions.h"
#include "cuda_globals.h"
#include "cuda_math.h"
#include "boundary_condition_x.h"
#include "boundary_condition_y.h"
#include "boundary_condition_z.h"
#include "comm.h"
#include "sponge.h"

__global__ void deviceAdvanceTime(myprec *dt) {
	time_on_GPU += *dt;
}

__global__ void deviceCalcPress(myprec *a, myprec *b, myprec *c) {
	*a = 0.99*(*b) - 0.5*( *a/(*c) - 1 ); //*a/(*c)-1 //a here is rw_bulk // changed from rw_bulk/r_bulk to rw_bulk
}

__global__ void derVelX(myprec *u, myprec *v, myprec *w, myprec *dudx, myprec *dvdx, myprec *dwdx) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidX();

	__shared__ myprec s_u[sPencils][mx+stencilSize*2];
	__shared__ myprec s_v[sPencils][mx+stencilSize*2];
	__shared__ myprec s_w[sPencils][mx+stencilSize*2];

	int si = id.i + stencilSize;
	int sj = id.tiy;

	s_u[sj][si] = u[id.g];
	s_v[sj][si] = v[id.g];
	s_w[sj][si] = w[id.g];
	__syncthreads();

	BCxderVel(s_u[sj],s_v[sj],s_w[sj],id,si,mx);
	__syncthreads();

	myprec wrk1;
	derDevSharedV1x(&wrk1,s_u[sj],si); dudx[id.g] = wrk1;
	derDevSharedV1x(&wrk1,s_v[sj],si); dvdx[id.g] = wrk1;
	derDevSharedV1x(&wrk1,s_w[sj],si); dwdx[id.g] = wrk1;

}

__global__ void derVelY(myprec *u, myprec *v, myprec *w, myprec *dudy, myprec *dvdy, myprec *dwdy) {

       Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
       int jNum = blockIdx.z;

       id.mkidYFlx(jNum);

       int tid = id.tix + id.bdx*id.tiy;

       int si = id.tiy + stencilSize;       // local i for shared memory access + halo offset
       int sj = id.tix;                   // local j for shared memory access
       int si1 = tid%id.bdy +  stencilSize;       // local i for shared memory access + halo offset
       int sj1 = tid/id.bdy;

       Indices id1(sj1,si1-stencilSize, blockIdx.x, blockIdx.y, blockDim.x, blockDim.y);
       id1.mkidYFlx(jNum);

       myprec wrk1=0;
       myprec wrk2=0;
       //myprec wrk3=0;

      __shared__ myprec s_u[mx/nDivX][my/nDivY+stencilSize*2];
      __shared__ myprec s_v[mx/nDivX][my/nDivY+stencilSize*2];
      //__shared__ myprec s_w[mx/nDivX][my/nDivY+stencilSize*2];
      s_u[sj][si] = u[id.g];
      s_v[sj][si] = v[id.g];
      //s_w[sj][si] = w[id.g];
      __syncthreads();
      BCyNumber2(s_u[sj],u,id,si,my,jNum);
      BCyNumber2(s_v[sj],v,id,si,my,jNum);
      //BCyNumber2(s_w[sj],w,id,si,my,jNum);
      __syncthreads();
// COMP Tile sj1 and si1
      derDevSharedV1y(&wrk1,s_u[sj1],si1);
      derDevSharedV1y(&wrk2,s_v[sj1],si1);
      //derDevSharedV1y(&wrk3,s_w[sj1],si1);
      __syncthreads();      
      s_u[sj1][si1] = wrk1;
      s_v[sj1][si1] = wrk2;
     // s_w[sj1][si1] = wrk3;
      __syncthreads();       
//MEM tile sj si      
      dudy[id.g] = s_u[sj][si]; 
      dvdy[id.g] = s_v[sj][si];
      //dwdy[id.g] = s_w[sj][si];
      __syncthreads();

/*	derDevV1yL(dudy,u,id);
	derDevV1yL(dvdy,v,id);
	//derDevV1yL(dwdy,w,id);*/
}

__global__ void derVelZ(myprec *u, myprec *v, myprec *w, myprec *dudz, myprec *dvdz, myprec *dwdz) {

       Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
       int kNum = blockIdx.z;

       id.mkidZFlx(kNum);

       int tid = id.tix + id.bdx*id.tiy;

       int si = id.tiy + stencilSize;       // local i for shared memory access + halo offset
       int sj = id.tix;                   // local j for shared memory access
       int si1 = tid%id.bdy +  stencilSize;       // local i for shared memory access + halo offset
       int sj1 = tid/id.bdy;

       Indices id1(sj1,si1-stencilSize, blockIdx.x, blockIdx.y, blockDim.x, blockDim.y);
       id1.mkidZFlx(kNum);

       myprec wrk1=0;
       myprec wrk2=0;
       myprec wrk3=0;

      __shared__ myprec s_u[mx/nDivX][mz/nDivZ+stencilSize*2];
      __shared__ myprec s_v[mx/nDivX][mz/nDivZ+stencilSize*2];
      __shared__ myprec s_w[mx/nDivX][mz/nDivZ+stencilSize*2];
      s_u[sj][si] = u[id.g];
      s_v[sj][si] = v[id.g];
      s_w[sj][si] = w[id.g];
      __syncthreads();
      BCzNumber2(s_u[sj],u,id,si,mz,kNum);
      BCzNumber2(s_v[sj],v,id,si,mz,kNum);
      BCzNumber2(s_w[sj],w,id,si,mz,kNum);
      __syncthreads();
// COMP Tile sj1 and si1
      derDevSharedV1z(&wrk1,s_u[sj1],si1);
      derDevSharedV1z(&wrk2,s_v[sj1],si1);
      derDevSharedV1z(&wrk3,s_w[sj1],si1);
      __syncthreads();      
      s_u[sj1][si1] = wrk1;
      s_v[sj1][si1] = wrk2;
      s_w[sj1][si1] = wrk3;
      __syncthreads();       
//MEM tile sj si      
      dudz[id.g] = s_u[sj][si]; 
      dvdz[id.g] = s_v[sj][si];
      dwdz[id.g] = s_w[sj][si];
      __syncthreads();
	/*derDevV1zL(dudz,u,uInit,id,kNum);
	derDevV1zL(dvdz,v,vInit,id,kNum);
	derDevV1zL(dwdz,w,wInit,id,kNum);*/
}

__global__ void derVelYBC(myprec *u, myprec *v, myprec *w, myprec *dudy, myprec *dvdy, myprec *dwdy, int direction) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);

	id.mkidYBC(direction);

	derDev1yBC(dudy,u,id,direction);
	derDev1yBC(dvdy,v,id,direction);
	derDev1yBC(dwdy,w,id,direction);
}

__global__ void derVelZBC(myprec *u, myprec *v, myprec *w, myprec *dudz, myprec *dvdz, myprec *dwdz, int direction) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);

	id.mkidZBC(direction);

	derDev1zBC(dudz,u,id,direction);
	derDev1zBC(dvdz,v,id,direction);
	derDev1zBC(dwdz,w,id,direction);
}

__global__ void calcDil(myprec *dil, myprec *dudx, myprec *dvdy, myprec *dwdz) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidX();

	//Stress goes with RHS old

	dil[id.g] = dudx[id.g] + dvdy[id.g] + dwdz[id.g];

}

void calcPressureGrad(myprec *dpdz, myprec *r, myprec *w, Communicator rk) {

    hipSetDevice(rk.nodeRank);

	myprec *workA, *dpdz_prev, *rbulk;
	dim3 gr0  = dim3(my / sPencils, mz, 1);
	dim3 bl0 = dim3(mx, sPencils, 1);

	checkCuda( hipMalloc((void**)&workA,mx*my*mz*sizeof(myprec)) );
	checkCuda( hipMalloc((void**)&dpdz_prev,     sizeof(myprec)) );
	checkCuda( hipMalloc((void**)&rbulk    ,     sizeof(myprec)) );

	deviceCpyOne<<<1,1>>>(dpdz_prev,dpdz);
	hostVolumeIntegral(rbulk,r,rk);
	deviceMul<<<gr0,bl0>>>(workA,r,w);
	hostVolumeIntegral(dpdz,workA,rk);

	deviceCalcPress<<<1,1>>>(dpdz,dpdz_prev,rbulk); //I changed here from rbulk to workA. // i CHANGED TO 1

	checkCuda( hipFree(workA) );
	checkCuda( hipFree(dpdz_prev) );
	checkCuda( hipFree(rbulk) );
}

void calcTimeStep(myprec *dt, myprec *r, myprec *u, myprec *v, myprec *w, myprec *e, myprec *mu, Communicator rk) {

	hipSetDevice(rk.nodeRank);

	myprec *workA;

	dim3 gr0 = dim3(my / sPencils, mz, 1);
	dim3 bl0 = dim3(mx, sPencils, 1);
    checkCuda( hipMalloc((void**)&workA,mx*my*mz*sizeof(myprec)) );
    deviceCalcDt<<<gr0,bl0>>>(workA,r,u,v,w,e,mu);

	hipDeviceSynchronize();
	hostReduceToMin(dt,workA,rk);

	checkCuda( hipFree(workA) );
	hipDeviceSynchronize();
}

__global__ void deviceCalcDt(myprec *wrkArray, myprec *r, myprec *u, myprec *v, myprec *w, myprec *e, myprec *mu) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);

	myprec dtConvInv = 0.0;
    myprec dtViscInv = 0.0;

    myprec ien = e[id.g]/r[id.g] - 0.5*(u[id.g]*u[id.g] + v[id.g]*v[id.g] + w[id.g]*w[id.g]);
    myprec sos = pow(gam*(gam-1)*ien,0.5);

    myprec dx,d2x;
    dx = d_dxv[id.i];
    d2x = dx*dx;

    dtConvInv =  MAX( (abs(u[id.g]) + sos)/dx, MAX( (abs(v[id.g]) + sos)*d_dy, (abs(w[id.g]) + sos)*d_dz) );
    dtViscInv =  MAX( mu[id.g]/d2x, MAX( mu[id.g]*d_d2y, mu[id.g]*d_d2z) );

    wrkArray[id.g] = CFL/MAX(dtConvInv, dtViscInv);
    __syncthreads();

}

void calcBulk(myprec *par1, myprec *par2, myprec *r, myprec *u, myprec *v, myprec *w, myprec *e, myprec *dtC, myprec *dpdz, int file, int istep, Communicator rk) {

	hipSetDevice(rk.nodeRank);

	myprec *workA, *rbulk;
	myprec *hostWork = (myprec*)malloc(sizeof(myprec));

	dim3 gr0  = dim3(my / sPencils, mz, 1);
	dim3 bl0 = dim3(mx, sPencils, 1);


	checkCuda( hipMalloc((void**)&workA,mx*my*mz*sizeof(myprec)) );
	checkCuda( hipMalloc((void**)&rbulk    ,     sizeof(myprec)) );

	if(forcing) {
		hostVolumeIntegral(rbulk,r,rk);
		checkCuda( hipMemcpy(hostWork, rbulk, sizeof(myprec), hipMemcpyDeviceToHost) );
		allReduceSum(hostWork,1);
		checkCuda( hipMemcpy(rbulk, hostWork, sizeof(myprec), hipMemcpyHostToDevice) );
		if(rk.rank==0) printf("step number %d with %le %le %3.10le\n",nsteps*(file-1) + istep ,*dtC,*dpdz, *hostWork);


		deviceMul<<<gr0,bl0>>>(workA,r,w);
		hostVolumeIntegral(par1,workA,rk);
		checkCuda( hipMemcpy(hostWork, par1, sizeof(myprec), hipMemcpyDeviceToHost) );
		allReduceSum(hostWork,1);
		checkCuda( hipMemcpy(par1, hostWork, sizeof(myprec), hipMemcpyHostToDevice) );
		deviceDivOne<<<1,1>>>(par1,par1,rbulk);

		hostVolumeIntegral(par2,e,rk);
		checkCuda( hipMemcpy(hostWork, par2, sizeof(myprec), hipMemcpyDeviceToHost) );
		allReduceSum(hostWork,1);
		checkCuda( hipMemcpy(par2, hostWork, sizeof(myprec), hipMemcpyHostToDevice) );
	} else {
		deviceSca<<<gr0,bl0>>>(workA,u,v,w,u,v,w);
		hostVolumeAverage(par1,workA,rk);
		checkCuda( hipMemcpy(hostWork, par1, sizeof(myprec), hipMemcpyDeviceToHost) );
		allReduceSum(hostWork,1);
		checkCuda( hipMemcpy(par1, hostWork, sizeof(myprec), hipMemcpyHostToDevice) );
	}
	free(hostWork);
	checkCuda( hipFree(workA) );
	checkCuda( hipFree(rbulk) );



}
