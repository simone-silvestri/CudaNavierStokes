#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "cuda_functions.h"
#include "cuda_globals.h"
#include "cuda_math.h"
#include "boundary.h"

__device__ myprec *d_workSX;
__device__ myprec *d_workSY;
__device__ myprec *d_workSZ;


__device__ void initStress() {
        checkCudaDev( hipMalloc((void**)&d_workSX,mx*my*mz*sizeof(myprec)) );
        checkCudaDev( hipMalloc((void**)&d_workSY,mx*my*mz*sizeof(myprec)) );
        checkCudaDev( hipMalloc((void**)&d_workSZ,mx*my*mz*sizeof(myprec)) );
}

__device__ void clearStress() {
        checkCudaDev( hipFree(d_workSX) );
        checkCudaDev( hipFree(d_workSY) );
        checkCudaDev( hipFree(d_workSZ) );
}

__global__ void calcStressX(myprec *u, myprec *v, myprec *w, myprec *stress[9]) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidX();

	__shared__ myprec s_u[sPencils][mx+stencilSize*2];
	__shared__ myprec s_v[sPencils][mx+stencilSize*2];
	__shared__ myprec s_w[sPencils][mx+stencilSize*2];

	int si = id.i + stencilSize;
	int sj = id.tiy;

	s_u[sj][si] = u[id.g];
	s_v[sj][si] = v[id.g];
	s_w[sj][si] = w[id.g];

	__syncthreads();

	if(id.i<stencilSize) {
#if periodicX
		perBCx(s_u[sj],si);perBCx(s_v[sj],si);perBCx(s_w[sj],si);
#else
		wallBCxVel(s_u[sj],si);wallBCxVel(s_v[sj],si);wallBCxVel(s_w[sj],si);
#endif
	}

	__syncthreads();

	myprec wrk1;
	derDevShared1x(&wrk1,s_u[sj],si); stress[0][id.g] = wrk1;
	derDevShared1x(&wrk1,s_v[sj],si); stress[1][id.g] = wrk1;
	derDevShared1x(&wrk1,s_w[sj],si); stress[2][id.g] = wrk1;

}

__global__ void calcStressY(myprec *u, myprec *v, myprec *w, myprec *stress[9]) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);

	derDev1yL(stress[3],u,id);
	derDev1yL(stress[4],v,id);
	derDev1yL(stress[5],w,id);
}

__global__ void calcStressZ(myprec *u, myprec *v, myprec *w, myprec *stress[9]) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);

	derDev1zL(stress[6],u,id);
	derDev1zL(stress[7],v,id);
	derDev1zL(stress[8],w,id);
}

__global__ void calcDil(myprec *stress[9], myprec *dil) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidX();

	//Stress goes with RHS old

	dil[id.g] = stress[0][id.g] + stress[4][id.g] + stress[8][id.g];

}

__device__ void calcPressureGrad(myprec *dpdz, myprec *r, myprec *w) {
	myprec dpdz_prev = *dpdz;
	volumeIntegral(dpdz,r);
	myprec rbulk = *dpdz;
#if (capability < capabilityMin)
	dim3 gr0,bl0;
	gr0 = dim3(grid0[0],grid0[1],1); bl0 = dim3(block0[0],block0[1],1);
	deviceMul<<<gr0,bl0>>>(d_workSX,r,w);
#else
	deviceMul<<<grid0,block0>>>(d_workSX,r,w);
#endif
	volumeIntegral(dpdz,d_workSX);
	*dpdz = *dpdz/rbulk;
	*dpdz = 0.99*dpdz_prev - 0.5*(*dpdz - 1.0);
}

__device__ void calcTimeStep(myprec *dt, myprec *r, myprec *u, myprec *v, myprec *w, myprec *e, myprec *mu) {

#if (capability < capabilityMin)
	dim3 gr0,bl0;
	gr0 = dim3(grid0[0],grid0[1],1); bl0 = dim3(block0[0],block0[1],1);
	deviceCalcDt<<<gr0,bl0>>>(d_workSX,r,u,v,w,e,mu);
#else
	deviceCalcDt<<<grid0,block0>>>(d_workSX,r,u,v,w,e,mu);
#endif
	hipDeviceSynchronize();
	reduceToMin(dt,d_workSX);
	hipDeviceSynchronize();
}

__global__ void deviceCalcDt(myprec *wrkArray, myprec *r, myprec *u, myprec *v, myprec *w, myprec *e, myprec *mu) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);

	myprec dtConvInv = 0.0;
    myprec dtViscInv = 0.0;

    myprec ien = e[id.g]/r[id.g] - 0.5*(u[id.g]*u[id.g] + v[id.g]*v[id.g] + w[id.g]*w[id.g]);
    myprec sos = pow(gamma*(gamma-1)*ien,0.5);

    myprec dx,d2x;
    dx = d_dxv[id.i];
    d2x = dx*dx;

    dtConvInv =  MAX( (abs(u[id.g]) + sos)/dx, MAX( (abs(v[id.g]) + sos)*d_dy, (abs(w[id.g]) + sos)*d_dz) );
    dtViscInv =  MAX( mu[id.g]/d2x, MAX( mu[id.g]*d_d2y, mu[id.g]*d_d2z) );

    wrkArray[id.g] = CFL/MAX(dtConvInv, dtViscInv);
    __syncthreads();

}

__device__ void calcIntegrals(myprec *r, myprec *u, myprec *v, myprec *w, myprec *stress[9], myprec *kin, myprec *enst) {

	*kin  = 0;
	*enst = 0;

	myprec dV = 1.0/d_dx/d_dy/d_dz;
#if (capability < capabilityMin)
	dim3 gr0,bl0;
	gr0 = dim3(grid0[0],grid0[1],1); bl0 = dim3(block0[0],block0[1],1);
	deviceSca<<<gr0,bl0>>>(d_workSX,u,v,w,u,v,w);
	deviceMul<<<gr0,bl0>>>(d_workSX,r,d_workSX);
#else
	deviceSca<<<grid0,block0>>>(d_workSX,u,v,w,u,v,w);
	deviceMul<<<grid0,block0>>>(d_workSX,r,d_workSX);
#endif
	hipDeviceSynchronize();
	volumeIntegral(kin,d_workSX);
	*kin *= 1.0/Lx/Ly/Lz/2.0;
#if (capability < capabilityMin)
	deviceSub<<<gr0,bl0>>>(d_workSX,stress[5],stress[7]);
	deviceSub<<<gr0,bl0>>>(d_workSY,stress[6],stress[2]);
	deviceSub<<<gr0,bl0>>>(d_workSZ,stress[1],stress[3]);

	deviceSca<<<gr0,bl0>>>(d_workSX,d_workSX,d_workSY,d_workSZ,d_workSX,d_workSY,d_workSZ);
	deviceMul<<<gr0,bl0>>>(d_workSX,r,d_workSX);
#else
	deviceSub<<<grid0,block0>>>(d_workSX,stress[5],stress[7]);
	deviceSub<<<grid0,block0>>>(d_workSY,stress[6],stress[2]);
	deviceSub<<<grid0,block0>>>(d_workSZ,stress[1],stress[3]);

	deviceSca<<<grid0,block0>>>(d_workSX,d_workSX,d_workSY,d_workSZ,d_workSX,d_workSY,d_workSZ);
	deviceMul<<<grid0,block0>>>(d_workSX,r,d_workSX);
#endif
	hipDeviceSynchronize();
	volumeIntegral(enst,d_workSX);
	*enst = *enst/Lx/Ly/Lz/Re;
}
