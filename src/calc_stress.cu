#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "cuda_functions.h"
#include "cuda_globals.h"
#include "cuda_math.h"
#include "boundary_condition_x.h"
#include "comm.h"
#include "sponge.h"

__global__ void deviceAdvanceTime(myprec *dt) {
	time_on_GPU += *dt;
}

__global__ void deviceCalcPress(myprec *a, myprec *b, myprec *c) {
	*a = 0.99*(*b) - 0.5*(*a/(*c)-1);
}

__global__ void derVelX(myprec *u, myprec *v, myprec *w, myprec *dudx, myprec *dvdx, myprec *dwdx) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidX();

	__shared__ myprec s_u[sPencils][mx+stencilSize*2];
	__shared__ myprec s_v[sPencils][mx+stencilSize*2];
	__shared__ myprec s_w[sPencils][mx+stencilSize*2];

	int si = id.i + stencilSize;
	int sj = id.tiy;

	s_u[sj][si] = u[id.g];
	s_v[sj][si] = v[id.g];
	s_w[sj][si] = w[id.g];
	__syncthreads();

	BCxderVel(s_u[sj],s_v[sj],s_w[sj],id,si,mx);
	__syncthreads();

	myprec wrk1;
	derDevSharedV1x(&wrk1,s_u[sj],si); dudx[id.g] = wrk1;
	derDevSharedV1x(&wrk1,s_v[sj],si); dvdx[id.g] = wrk1;
	derDevSharedV1x(&wrk1,s_w[sj],si); dwdx[id.g] = wrk1;

}

__global__ void derVelY(myprec *u, myprec *v, myprec *w, myprec *dudy, myprec *dvdy, myprec *dwdy) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);

	derDevV1yL(dudy,u,id);
	derDevV1yL(dvdy,v,id);
	derDevV1yL(dwdy,w,id);
}

__global__ void derVelZ(myprec *u, myprec *v, myprec *w, myprec *dudz, myprec *dvdz, myprec *dwdz, int kNum) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);

	derDevV1zL(dudz,u,uInit,id,kNum);
	derDevV1zL(dvdz,v,vInit,id,kNum);
	derDevV1zL(dwdz,w,wInit,id,kNum);
}

__global__ void derVelYBC(myprec *u, myprec *v, myprec *w, myprec *dudy, myprec *dvdy, myprec *dwdy, int direction) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);

	id.mkidYBC(direction);

	derDev1yBC(dudy,u,id,direction);
	derDev1yBC(dvdy,v,id,direction);
	derDev1yBC(dwdy,w,id,direction);
}

__global__ void derVelZBC(myprec *u, myprec *v, myprec *w, myprec *dudz, myprec *dvdz, myprec *dwdz, int direction) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);

	id.mkidZBC(direction);

	derDev1zBC(dudz,u,id,direction);
	derDev1zBC(dvdz,v,id,direction);
	derDev1zBC(dwdz,w,id,direction);
}

__global__ void calcDil(myprec *dil, myprec *dudx, myprec *dvdy, myprec *dwdz) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidX();

	//Stress goes with RHS old

	dil[id.g] = dudx[id.g] + dvdy[id.g] + dwdz[id.g];

}

void calcPressureGrad(myprec *dpdz, myprec *r, myprec *w, Communicator rk) {

    hipSetDevice(rk.nodeRank);

	myprec *workA, *dpdz_prev, *rbulk;
	dim3 gr0  = dim3(my / sPencils, mz, 1);
	dim3 bl0 = dim3(mx, sPencils, 1);

	checkCuda( hipMalloc((void**)&workA,mx*my*mz*sizeof(myprec)) );
	checkCuda( hipMalloc((void**)&dpdz_prev,     sizeof(myprec)) );
	checkCuda( hipMalloc((void**)&rbulk    ,     sizeof(myprec)) );

	deviceCpyOne<<<1,1>>>(dpdz_prev,dpdz);
	hostVolumeIntegral(rbulk,r,rk);
	deviceMul<<<gr0,bl0>>>(workA,r,w);
	hostVolumeIntegral(dpdz,workA,rk);

	deviceCalcPress<<<1,1>>>(dpdz,dpdz_prev,rbulk);

	checkCuda( hipFree(workA) );
	checkCuda( hipFree(dpdz_prev) );
	checkCuda( hipFree(rbulk) );
}

void calcTimeStep(myprec *dt, myprec *r, myprec *u, myprec *v, myprec *w, myprec *e, myprec *mu, Communicator rk) {

	hipSetDevice(rk.nodeRank);

	myprec *workA;

	dim3 gr0 = dim3(my / sPencils, mz, 1);
	dim3 bl0 = dim3(mx, sPencils, 1);
    checkCuda( hipMalloc((void**)&workA,mx*my*mz*sizeof(myprec)) );
    deviceCalcDt<<<gr0,bl0>>>(workA,r,u,v,w,e,mu);

	hipDeviceSynchronize();
	hostReduceToMin(dt,workA,rk);

	checkCuda( hipFree(workA) );
	hipDeviceSynchronize();
}

__global__ void deviceCalcDt(myprec *wrkArray, myprec *r, myprec *u, myprec *v, myprec *w, myprec *e, myprec *mu) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);

	myprec dtConvInv = 0.0;
    myprec dtViscInv = 0.0;

    myprec ien = e[id.g]/r[id.g] - 0.5*(u[id.g]*u[id.g] + v[id.g]*v[id.g] + w[id.g]*w[id.g]);
    myprec sos = pow(gam*(gam-1)*ien,0.5);

    myprec dx,d2x;
    dx = d_dxv[id.i];
    d2x = dx*dx;

    dtConvInv =  MAX( (abs(u[id.g]) + sos)/dx, MAX( (abs(v[id.g]) + sos)*d_dy, (abs(w[id.g]) + sos)*d_dz) );
    dtViscInv =  MAX( mu[id.g]/d2x, MAX( mu[id.g]*d_d2y, mu[id.g]*d_d2z) );

    wrkArray[id.g] = CFL/MAX(dtConvInv, dtViscInv);
    __syncthreads();

}

void calcBulk(myprec *par1, myprec *par2, myprec *r, myprec *u, myprec *v, myprec *w, myprec *e, Communicator rk) {

	hipSetDevice(rk.nodeRank);

	myprec *workA, *rbulk;
	myprec *hostWork = (myprec*)malloc(sizeof(myprec));
	dim3 gr0  = dim3(my / sPencils, mz, 1);
	dim3 bl0 = dim3(mx, sPencils, 1);

	checkCuda( hipMalloc((void**)&workA,mx*my*mz*sizeof(myprec)) );
	checkCuda( hipMalloc((void**)&rbulk    ,     sizeof(myprec)) );

	if(forcing) {
		hostVolumeIntegral(rbulk,r,rk);
		checkCuda( hipMemcpy(hostWork, rbulk, sizeof(myprec), hipMemcpyDeviceToHost) );
		allReduceSum(hostWork,1);
		checkCuda( hipMemcpy(rbulk, hostWork, sizeof(myprec), hipMemcpyHostToDevice) );

		deviceMul<<<gr0,bl0>>>(workA,r,w);
		hostVolumeIntegral(par1,workA,rk);
		checkCuda( hipMemcpy(hostWork, par1, sizeof(myprec), hipMemcpyDeviceToHost) );
		allReduceSum(hostWork,1);
		checkCuda( hipMemcpy(par1, hostWork, sizeof(myprec), hipMemcpyHostToDevice) );
		deviceDivOne<<<1,1>>>(par1,par1,rbulk);

		hostVolumeIntegral(par2,e,rk);
		checkCuda( hipMemcpy(hostWork, par2, sizeof(myprec), hipMemcpyDeviceToHost) );
		allReduceSum(hostWork,1);
		checkCuda( hipMemcpy(par2, hostWork, sizeof(myprec), hipMemcpyHostToDevice) );
	} else {
		deviceSca<<<gr0,bl0>>>(workA,u,v,w,u,v,w);
		hostVolumeAverage(par1,workA,rk);
		checkCuda( hipMemcpy(hostWork, par1, sizeof(myprec), hipMemcpyDeviceToHost) );
		allReduceSum(hostWork,1);
		checkCuda( hipMemcpy(par1, hostWork, sizeof(myprec), hipMemcpyHostToDevice) );
	}
	free(hostWork);
	checkCuda( hipFree(workA) );
	checkCuda( hipFree(rbulk) );
}
