#include "hip/hip_runtime.h"
#include "globals.h"
#include "cuda_globals.h"
#include "cuda_functions.h"
#include "comm.h"
#include "main.h"
#include "sponge.h"

__device__ myprec spongeX[mx];
__device__ myprec spongeZ[mz];
__device__ myprec rref[mx*mz];
__device__ myprec uref[mx*mz];
__device__ myprec vref[mx*mz];
__device__ myprec wref[mx*mz];
__device__ myprec eref[mx*mz];

__device__ myprec rInit[mx];
__device__ myprec uInit[mx];
__device__ myprec vInit[mx];
__device__ myprec wInit[mx];
__device__ myprec eInit[mx];
__device__ myprec pInit[mx];
__device__ myprec tInit[mx];
__device__ myprec hInit[mx];
__device__ myprec mInit[mx];
__device__ myprec lInit[mx];


void spline(myprec x[], myprec y[], int n, myprec yp1, myprec ypn, myprec y2[]);
myprec splint(myprec xa[], myprec ya[], myprec y2a[], int n, myprec x);

__global__ void addSponge(myprec *rhsr, myprec *rhsu, myprec *rhsv, myprec *rhsw, myprec *rhse,
						  myprec *r, myprec *u, myprec *v, myprec *w, myprec *e) {

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);

	rhsr[id.g] += (spongeX[id.i] + spongeZ[id.k]) * (rref[idx2(id.i,id.k)] - r[id.g]);
	rhsu[id.g] += (spongeX[id.i] + spongeZ[id.k]) * (uref[idx2(id.i,id.k)] - r[id.g]*u[id.g]);
	rhsv[id.g] += (spongeX[id.i] + spongeZ[id.k]) * (vref[idx2(id.i,id.k)] - r[id.g]*v[id.g]);
	rhsw[id.g] += (spongeX[id.i] + spongeZ[id.k]) * (wref[idx2(id.i,id.k)] - r[id.g]*w[id.g]);
	rhse[id.g] += (spongeX[id.i] + spongeZ[id.k]) * (eref[idx2(id.i,id.k)] - e[id.g]);
}

__global__ void copySpongeToDevice(myprec *d_spongeX, myprec *d_spongeZ, myprec *d_rref, myprec *d_uref, myprec *d_wref, myprec *d_eref,
																		 myprec *d_rIn, myprec *d_uIn, myprec *d_wIn, myprec *d_eIn) {

	int bdx = blockDim.x ;
	int tix = threadIdx.x;
	int bix = blockIdx.x ;

	int gl = tix + bdx * bix;

	if(bix==0)	spongeX[tix] = d_spongeX[tix];
	__syncthreads();
	if(tix==0)	spongeZ[bix] = d_spongeZ[bix];
	__syncthreads();

	rref[gl] = d_rref[gl];
	uref[gl] = d_uref[gl]*d_rref[gl];
	vref[gl] = 0.0;
	wref[gl] = d_wref[gl]*d_rref[gl];
	eref[gl] = d_eref[gl];

	if(bix==0) {
		int idx = tix;
		rInit[idx] = d_rIn[idx];
		uInit[idx] = d_uIn[idx];
		vInit[idx] = 0.0;
		wInit[idx] = d_wIn[idx];
		eInit[idx] = d_eIn[idx];

		pInit[idx] = Rgas;
		tInit[idx] = pInit[idx]/(Rgas*rInit[idx]);

		hInit[idx]   = (eInit[idx] + pInit[idx])/rInit[idx];

		myprec suth = pow(tInit[idx],viscexp);
		mInit[idx]    = suth/Re;
		lInit[idx]    = suth/Re/Pr/Ec;
	}
    __syncthreads();
}

void calculateSponge(Communicator rk) {

	myprec *h_spongeX = new myprec[mx];
	myprec *h_spongeZ = new myprec[mz];
	myprec *h_rref = new myprec[mx*mz];
	myprec *h_uref = new myprec[mx*mz];
	myprec *h_wref = new myprec[mx*mz];
	myprec *h_eref = new myprec[mx*mz];
	myprec *h_rIn  = new myprec[mx];
	myprec *h_uIn  = new myprec[mx];
	myprec *h_wIn  = new myprec[mx];
	myprec *h_eIn  = new myprec[mx];
	myprec *d_spongeX, *d_spongeZ;
	myprec *d_rref;
	myprec *d_uref;
	myprec *d_wref;
	myprec *d_eref;
	myprec *d_rIn,*d_uIn,*d_wIn,*d_eIn;

	checkCuda( hipMalloc((void**)&d_spongeX, mx*sizeof(myprec)) );
	checkCuda( hipMalloc((void**)&d_spongeZ, mz*sizeof(myprec)) );

	checkCuda( hipMalloc((void**)&d_rref, mz*mx*sizeof(myprec)) );
	checkCuda( hipMalloc((void**)&d_uref, mz*mx*sizeof(myprec)) );
	checkCuda( hipMalloc((void**)&d_wref, mz*mx*sizeof(myprec)) );
	checkCuda( hipMalloc((void**)&d_eref, mz*mx*sizeof(myprec)) );

	checkCuda( hipMalloc((void**)&d_rIn, mx*sizeof(myprec)) );
	checkCuda( hipMalloc((void**)&d_uIn, mx*sizeof(myprec)) );
	checkCuda( hipMalloc((void**)&d_wIn, mx*sizeof(myprec)) );
	checkCuda( hipMalloc((void**)&d_eIn, mx*sizeof(myprec)) );

	for(int i=0; i<mx; i++) {
		h_spongeX[i] = 0.0;
		if ((spTopLen > 0.0) && (x[i] >= Lx - spTopLen))
			h_spongeX[i] = spTopStr*pow((x[i] - (Lx-spTopLen))/spTopLen , spTopExp);
	}

	for(int k=0; k<mz; k++) {
		h_spongeZ[k] = 0.0;
		myprec fz = z[k+rk.kstart];
		if ((spInlLen > 0.0) && (fz <= spInlLen))
			h_spongeZ[k] = spInlStr*pow( (spInlLen-fz)/spInlLen , spInlExp);

		if ((spOutLen > 0.0) && (fz >= (Lz-spOutLen)))
			h_spongeZ[k] = spOutStr*pow((fz - (Lz-spOutLen))/spOutLen,spOutExp);
	}

	FILE *fp = fopen("blasius1D/xProf.bin","rb");

	fseek(fp, 0, SEEK_END); 		  // seek to end of file
	int size = ftell(fp) / sizeof(double); // get current file pointer
	fseek(fp, 0, SEEK_SET); 		  // seek back to beginning of file

	myprec xIn[size] , rIn[size] , uIn[size] , wIn[size] , eIn[size];
	myprec             r2In[size], u2In[size], w2In[size], e2In[size];
	size_t result;

	result = fread(xIn, sizeof(double), size, fp);
	if (result != size) {fputs ("Reading error",stderr); exit (3);}; fclose(fp);
	fp = fopen("blasius1D/rProf.bin","rb");
	result = fread(rIn, sizeof(double), size, fp);
	if (result != size) {fputs ("Reading error",stderr); exit (3);}; fclose(fp);
	fp = fopen("blasius1D/uProf.bin","rb");
	result = fread(uIn, sizeof(double), size, fp);
	if (result != size) {fputs ("Reading error",stderr); exit (3);}; fclose(fp);
	fp = fopen("blasius1D/wProf.bin","rb");
	result = fread(wIn, sizeof(double), size, fp);
	if (result != size) {fputs ("Reading error",stderr); exit (3);}; fclose(fp);
	fp = fopen("blasius1D/eProf.bin","rb");
	result = fread(eIn, sizeof(double), size, fp);
	if (result != size) {fputs ("Reading error",stderr); exit (3);}; fclose(fp);

	spline(xIn, rIn, size, 1e30, 1e30, r2In);
	spline(xIn, uIn, size, 1e30, 1e30, u2In);
	spline(xIn, wIn, size, 1e30, 1e30, w2In);
	spline(xIn, eIn, size, 1e30, 1e30, e2In);
	for (int k=0; k<mz; k++)
		for (int i=0; i<mx; i++) {
			myprec scale = pow( 1 + z[k+rk.kstart]/Re, 0.5 );
			h_rref[idx2(i,k)] = splint(xIn,rIn,r2In,size,x[i]/scale);
			h_uref[idx2(i,k)] = splint(xIn,uIn,u2In,size,x[i]/scale);
			h_uref[idx2(i,k)]/= (scale*Re);
			h_wref[idx2(i,k)] = splint(xIn,wIn,w2In,size,x[i]/scale);
			myprec pconst    = Rgas;
			h_eref[idx2(i,k)] = pconst/(gam-1.0) + h_rref[idx2(i,k)]*0.5*(h_uref[idx2(i,k)]*h_uref[idx2(i,k)]+h_wref[idx2(i,k)]*h_wref[idx2(i,k)]);
		}

	for (int i=0; i<mx; i++) {
		h_rIn[i] = splint(xIn,rIn,r2In,size,x[i]);
		h_uIn[i] = splint(xIn,uIn,u2In,size,x[i]);
		h_uIn[i]/= (Re);
		h_wIn[i] = splint(xIn,wIn,w2In,size,x[i]);
		h_eIn[i] = Rgas/(gam-1.0) + h_rIn[i]*0.5*(h_uIn[i]*h_uIn[i]+h_wIn[i]*h_wIn[i]);
	}

	FILE *fw = fopen("inProf.txt","w+");
	for(int i=0; i<size; i++) {
		fprintf(fw,"%le %le %le %le %le\n",xIn[i],rIn[i],uIn[i],wIn[i],eIn[i]);
	}
	fclose(fw);

	if(restartFile<0) {
		for (int k=0; k<mz; k++)
			for (int j=0; j<my; j++)
				for (int i=0; i<mx; i++) {
					r[idx(i,j,k)] = h_rref[idx2(i,k)];
					u[idx(i,j,k)] = h_uref[idx2(i,k)];
					v[idx(i,j,k)] = 0.0;
					w[idx(i,j,k)] = h_wref[idx2(i,k)];
					e[idx(i,j,k)] = h_eref[idx2(i,k)];
				}
	}

	fw = fopen("inRef.txt","w+");
	for(int i=0; i<mx; i++) {
		fprintf(fw,"%le %le %le %le %le\n",x[i],r[idx(i,0,0)],u[idx(i,0,0)],w[idx(i,0,0)],e[idx(i,0,0)]);
	}
	fclose(fw);


    checkCuda( hipMemcpy(d_spongeX, h_spongeX, mx*sizeof(myprec), hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(d_spongeZ, h_spongeZ, mz*sizeof(myprec), hipMemcpyHostToDevice) );

    checkCuda( hipMemcpy(d_rref, h_rref, mz*mx*sizeof(myprec), hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(d_uref, h_uref, mz*mx*sizeof(myprec), hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(d_wref, h_wref, mz*mx*sizeof(myprec), hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(d_eref, h_eref, mz*mx*sizeof(myprec), hipMemcpyHostToDevice) );

    checkCuda( hipMemcpy(d_rIn, h_rIn, mx*sizeof(myprec), hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(d_uIn, h_uIn, mx*sizeof(myprec), hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(d_wIn, h_wIn, mx*sizeof(myprec), hipMemcpyHostToDevice) );
    checkCuda( hipMemcpy(d_eIn, h_eIn, mx*sizeof(myprec), hipMemcpyHostToDevice) );

	delete [] h_spongeX;
    delete [] h_spongeZ;
    delete [] h_rref;
    delete [] h_uref;
    delete [] h_wref;
    delete [] h_eref;
    delete [] h_rIn;
    delete [] h_uIn;
    delete [] h_wIn;
    delete [] h_eIn;

    copySpongeToDevice<<< mz, mx >>>(d_spongeX,d_spongeZ,d_rref,d_uref,d_wref,d_eref,d_rIn,d_uIn,d_wIn,d_eIn);

    checkCuda( hipFree(d_spongeX) );
    checkCuda( hipFree(d_spongeZ) );
    checkCuda( hipFree(d_rref) );
    checkCuda( hipFree(d_uref) );
    checkCuda( hipFree(d_wref) );
    checkCuda( hipFree(d_eref) );
    checkCuda( hipFree(d_rIn) );
    checkCuda( hipFree(d_uIn) );
    checkCuda( hipFree(d_wIn) );
    checkCuda( hipFree(d_eIn) );
}

myprec *vector12(long nl, long nh)
/* allocate a float vector with subscript range v[nl..nh] */
{
	myprec *v;

	v=(myprec *)malloc((size_t) ((nh-nl+1+1)*sizeof(myprec)));
	if (!v){
		fprintf(stderr,"allocation failure in vector()");
		fprintf(stderr,"...now exiting to system...\n");
		exit(1);
	}
	return v-nl+1;
}

void free_vector(myprec *v, long nl, long nh)
/* free a float vector allocated with vector() */
{
	free((char*) (v+nl-1));
}

void spline(myprec x[], myprec y[], int n, myprec yp1, myprec ypn, myprec y2[])
{
	int i,k;
	myprec p,qn,sig,un,*u;
	u=vector12(1,n-1);
	if (yp1 > 0.99e30)
		y2[1]=u[1]=0.0;
	else {
		y2[1] = -0.5;
		u[1]=(3.0/(x[2]-x[1]))*((y[2]-y[1])/(x[2]-x[1])-yp1);
	}
	for (i=2;i<=n-1;i++) {
		sig=(x[i]-x[i-1])/(x[i+1]-x[i-1]);
		p=sig*y2[i-1]+2.0;
		y2[i]=(sig-1.0)/p;
		u[i]=(y[i+1]-y[i])/(x[i+1]-x[i]) - (y[i]-y[i-1])/(x[i]-x[i-1]);
		u[i]=(6.0*u[i]/(x[i+1]-x[i-1])-sig*u[i-1])/p;
	}
	if (ypn > 0.99e30)
		qn=un=0.0;
	else {
		qn=0.5;
		un=(3.0/(x[n]-x[n-1]))*(ypn-(y[n]-y[n-1])/(x[n]-x[n-1]));
	}
	y2[n]=(un-qn*u[n-1])/(qn*y2[n-1]+1.0);
	for (k=n-1;k>=1;k--)
		y2[k]=y2[k]*y2[k+1]+u[k];
	free_vector(u,1,n-1);
}

myprec splint(myprec xa[], myprec ya[], myprec y2a[], int n, myprec x)
{
	int klo,khi,k;
	myprec h,b,a,y;
	klo=1;
	khi=n;
	while (khi-klo > 1) {
		k=(khi+klo) >> 1;
		if (xa[k] > x) khi=k;
		else klo=k;
	}
	h=xa[khi]-xa[klo];
	if (h == 0.0){
		fprintf(stderr,"Bad xa input to routine splint");
		fprintf(stderr,"...now exiting to system...\n");
		exit(1);
	}
	a=(xa[khi]-x)/h;
	b=(x-xa[klo])/h;
	y=a*ya[klo]+b*ya[khi]+((a*a*a-a)*y2a[klo]+(b*b*b-b)*y2a[khi])*(h*h)/6.0;
	return y;
}
