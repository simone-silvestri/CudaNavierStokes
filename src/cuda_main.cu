#include "hip/hip_runtime.h"
#include "globals.h"
#include "cuda_functions.h"
#include "cuda_globals.h"
#include "cuda_main.h"
#include "cuda_math.h"
#include "comm.h"

hipStream_t s[9];

void runSimulation(myprec *par1, myprec *par2, myprec *time, Communicator rk) {

	hipSetDevice(rk.nodeRank);

	myprec h_dt,h_dpdz;

	/* allocating temporary arrays and streams */
	void (*RHSDeviceDir[3])(myprec*, myprec*, myprec*, myprec*, myprec*, myprec*, myprec*, myprec*,
							myprec*, myprec*, myprec*, myprec*, myprec*, myprec*, myprec*, myprec*);

	RHSDeviceDir[0] = RHSDeviceSharedFlxX;
	RHSDeviceDir[1] = RHSDeviceSharedFlxY;
	RHSDeviceDir[2] = RHSDeviceSharedFlxZ;

    for (int istep = 0; istep < nsteps; istep++) {
    	if(istep%checkCFLcondition==0) calcTimeStepPressGrad(istep,dtC,dpdz,&h_dt,&h_dpdz,rk);
    	if(istep>0)  deviceSumOne<<<1,1>>>(&time[istep],&time[istep-1] ,dtC);
    	if(istep==0) deviceSumOne<<<1,1>>>(&time[istep],&time[nsteps-1],dtC);
    	if(istep%checkBulk==0) calcBulk(&par1[istep],&par2[istep],d_r,d_w,d_e,rk);

    	deviceMul<<<grid0,block0,0,s[0]>>>(d_uO,d_r,d_u);
    	deviceMul<<<grid0,block0,0,s[1]>>>(d_vO,d_r,d_v);
    	deviceMul<<<grid0,block0,0,s[2]>>>(d_wO,d_r,d_w);
    	deviceCpy<<<grid0,block0,0,s[3]>>>(d_rO,d_r);
    	deviceCpy<<<grid0,block0,0,s[4]>>>(d_eO,d_e);

    	//Starting the Runge-Kutta Steps

    	//runge kutta step 1
		calcState<<<grid0,block0,0,s[0]>>>(d_r,d_u,d_v,d_w,d_e,d_h,d_t,d_p,d_m,d_l,0);
		calcStressX<<<d_grid[0],d_block[0],0,s[1]>>>(d_u,d_v,d_w);
		calcStressY<<<d_grid[3],d_block[3],0,s[2]>>>(d_u,d_v,d_w);
		calcStressZ<<<d_grid[4],d_block[4],0,s[3]>>>(d_u,d_v,d_w);
		if(multiGPU) {
			updateHaloFive(d_r,d_u,d_v,d_w,d_e,rk); hipDeviceSynchronize();
			calcState<<<gridHalo,blockHalo,0,s[4]>>>(d_r,d_u,d_v,d_w,d_e,d_h,d_t,d_p,d_m,d_l,1);
			calcStressYBC<<<gridHaloY,blockHaloY,0,s[0]>>>(d_u,d_v,d_w,0);
			calcStressZBC<<<gridHaloZ,blockHaloZ,0,s[1]>>>(d_u,d_v,d_w,0);
			calcStressYBC<<<gridHaloY,blockHaloY,0,s[2]>>>(d_u,d_v,d_w,1);
			calcStressZBC<<<gridHaloZ,blockHaloZ,0,s[3]>>>(d_u,d_v,d_w,1);
		}
		hipDeviceSynchronize();
    	calcDil<<<grid0,block0>>>(d_dil);
    	hipDeviceSynchronize();
#if useStreams
    	if(multiGPU) updateHalo(d_dil,rk); hipDeviceSynchronize();
    	for (int d = 0; d < 3; d++)
    		RHSDeviceDir[d]<<<d_grid[d],d_block[d],0,s[d]>>>(d_rhsr1[d],d_rhsu1[d],d_rhsv1[d],d_rhsw1[d],d_rhse1[d],d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,d_dil,dpdz);
#else
		if(multiGPU) deviceBlocker<<<grid0,block0,0,s[0]>>>();
		RHSDeviceDir[0]<<<d_grid[0],d_block[0],0,s[0]>>>(d_rhsr1[0],d_rhsu1[0],d_rhsv1[0],d_rhsw1[0],d_rhse1[0],d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,d_dil,dpdz);
    	if(multiGPU) updateHalo(d_dil,rk); hipDeviceSynchronize();
		RHSDeviceDir[1]<<<d_grid[1],d_block[1]>>>(d_rhsr1[0],d_rhsu1[0],d_rhsv1[0],d_rhsw1[0],d_rhse1[0],d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,d_dil,dpdz);
		RHSDeviceDir[2]<<<d_grid[2],d_block[2]>>>(d_rhsr1[0],d_rhsu1[0],d_rhsv1[0],d_rhsw1[0],d_rhse1[0],d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,d_dil,dpdz);
#endif
    	for (int d=0; d<fin; d++) {
    		eulerSum<<<grid0,block0>>>(d_r,d_rO,d_rhsr1[d],dtC,d);
    		eulerSum<<<grid0,block0>>>(d_e,d_eO,d_rhse1[d],dtC,d);    	}
		hipDeviceSynchronize();
		for (int d=0; d<fin; d++) {
    		eulerSumR<<<grid0,block0,0,s[0]>>>(d_u,d_uO,d_rhsu1[d],d_r,dtC,d);
    		eulerSumR<<<grid0,block0,0,s[1]>>>(d_v,d_vO,d_rhsv1[d],d_r,dtC,d);
    		eulerSumR<<<grid0,block0,0,s[2]>>>(d_w,d_wO,d_rhsw1[d],d_r,dtC,d);    	}
		hipDeviceSynchronize();

		if(multiGPU) {  //To initiate slowly the routines so that we have time to initiate the memory transfer
			deviceCpy<<<grid0,block0,0,s[0]>>>(d_r,d_r);
			deviceCpy<<<grid0,block0,0,s[1]>>>(d_u,d_u);
			deviceCpy<<<grid0,block0,0,s[2]>>>(d_v,d_v);
			deviceCpy<<<grid0,block0,0,s[3]>>>(d_w,d_w); }

		//runge kutta step 2
		calcState<<<grid0,block0,0,s[0]>>>(d_r,d_u,d_v,d_w,d_e,d_h,d_t,d_p,d_m,d_l,0);
		calcStressX<<<d_grid[0],d_block[0],0,s[1]>>>(d_u,d_v,d_w);
		calcStressY<<<d_grid[3],d_block[3],0,s[2]>>>(d_u,d_v,d_w);
		calcStressZ<<<d_grid[4],d_block[4],0,s[3]>>>(d_u,d_v,d_w);
		if(multiGPU) {
			updateHaloFive(d_r,d_u,d_v,d_w,d_e,rk); hipDeviceSynchronize();
			calcState<<<gridHalo,blockHalo,0,s[4]>>>(d_r,d_u,d_v,d_w,d_e,d_h,d_t,d_p,d_m,d_l,1);
			calcStressYBC<<<gridHaloY,blockHaloY,0,s[0]>>>(d_u,d_v,d_w,0);
			calcStressZBC<<<gridHaloZ,blockHaloZ,0,s[1]>>>(d_u,d_v,d_w,0);
			calcStressYBC<<<gridHaloY,blockHaloY,0,s[2]>>>(d_u,d_v,d_w,1);
			calcStressZBC<<<gridHaloZ,blockHaloZ,0,s[3]>>>(d_u,d_v,d_w,1);
		}
		hipDeviceSynchronize();
    	calcDil<<<grid0,block0>>>(d_dil);
    	hipDeviceSynchronize();
#if useStreams
    	if(multiGPU) updateHalo(d_dil,rk); hipDeviceSynchronize();
    	for (int d = 0; d < 3; d++)
    		RHSDeviceDir[d]<<<d_grid[d],d_block[d],0,s[d]>>>(d_rhsr2[d],d_rhsu2[d],d_rhsv2[d],d_rhsw2[d],d_rhse2[d],d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,d_dil,dpdz);
#else
		if(multiGPU) deviceBlocker<<<grid0,block0,0,s[0]>>>();
		RHSDeviceDir[0]<<<d_grid[0],d_block[0],0,s[0]>>>(d_rhsr2[0],d_rhsu2[0],d_rhsv2[0],d_rhsw2[0],d_rhse2[0],d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,d_dil,dpdz);
    	if(multiGPU) updateHalo(d_dil,rk); hipDeviceSynchronize();
		RHSDeviceDir[1]<<<d_grid[1],d_block[1]>>>(d_rhsr2[0],d_rhsu2[0],d_rhsv2[0],d_rhsw2[0],d_rhse2[0],d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,d_dil,dpdz);
		RHSDeviceDir[2]<<<d_grid[2],d_block[2]>>>(d_rhsr2[0],d_rhsu2[0],d_rhsv2[0],d_rhsw2[0],d_rhse2[0],d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,d_dil,dpdz);
#endif
		for (int d=0; d<fin; d++) {
			eulerSum3<<<grid0,block0>>>(d_r,d_rO,d_rhsr1[d],d_rhsr2[d],dtC,d);
			eulerSum3<<<grid0,block0>>>(d_e,d_eO,d_rhse1[d],d_rhse2[d],dtC,d);   	}
		hipDeviceSynchronize();
		for (int d=0; d<fin; d++) {
			eulerSum3R<<<grid0,block0,0,s[0]>>>(d_u,d_uO,d_rhsu1[d],d_rhsu2[d],d_r,dtC,d);
			eulerSum3R<<<grid0,block0,0,s[1]>>>(d_v,d_vO,d_rhsv1[d],d_rhsv2[d],d_r,dtC,d);
			eulerSum3R<<<grid0,block0,0,s[2]>>>(d_w,d_wO,d_rhsw1[d],d_rhsw2[d],d_r,dtC,d); }
    	hipDeviceSynchronize();

		if(multiGPU) {  //To initiate slowly the routines so that we have time to initiate the memory transfer
			deviceCpy<<<grid0,block0,0,s[0]>>>(d_r,d_r);
			deviceCpy<<<grid0,block0,0,s[1]>>>(d_u,d_u);
			deviceCpy<<<grid0,block0,0,s[2]>>>(d_v,d_v);
			deviceCpy<<<grid0,block0,0,s[3]>>>(d_w,d_w); }

    	//runge kutta step 3
		calcState<<<grid0,block0,0,s[0]>>>(d_r,d_u,d_v,d_w,d_e,d_h,d_t,d_p,d_m,d_l,0);
		calcStressX<<<d_grid[0],d_block[0],0,s[1]>>>(d_u,d_v,d_w);
		calcStressY<<<d_grid[3],d_block[3],0,s[2]>>>(d_u,d_v,d_w);
		calcStressZ<<<d_grid[4],d_block[4],0,s[3]>>>(d_u,d_v,d_w);
		if(multiGPU) {
			updateHaloFive(d_r,d_u,d_v,d_w,d_e,rk); hipDeviceSynchronize();
			calcState<<<gridHalo,blockHalo,0,s[4]>>>(d_r,d_u,d_v,d_w,d_e,d_h,d_t,d_p,d_m,d_l,1);
			calcStressYBC<<<gridHaloY,blockHaloY,0,s[0]>>>(d_u,d_v,d_w,0);
			calcStressZBC<<<gridHaloZ,blockHaloZ,0,s[1]>>>(d_u,d_v,d_w,0);
			calcStressYBC<<<gridHaloY,blockHaloY,0,s[2]>>>(d_u,d_v,d_w,1);
			calcStressZBC<<<gridHaloZ,blockHaloZ,0,s[3]>>>(d_u,d_v,d_w,1);
		}
		hipDeviceSynchronize();
    	calcDil<<<grid0,block0>>>(d_dil);
    	hipDeviceSynchronize();
#if useStreams
    	if(multiGPU) updateHalo(d_dil,rk); hipDeviceSynchronize();
    	for (int d = 0; d < 3; d++)
    		RHSDeviceDir[d]<<<d_grid[d],d_block[d],0,s[d]>>>(d_rhsr3[d],d_rhsu3[d],d_rhsv3[d],d_rhsw3[d],d_rhse3[d],d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,d_dil,dpdz);
#else
		if(multiGPU) deviceBlocker<<<grid0,block0,0,s[0]>>>();
		RHSDeviceDir[0]<<<d_grid[0],d_block[0],0,s[0]>>>(d_rhsr3[0],d_rhsu3[0],d_rhsv3[0],d_rhsw3[0],d_rhse3[0],d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,d_dil,dpdz);
    	if(multiGPU) updateHalo(d_dil,rk); hipDeviceSynchronize();
		RHSDeviceDir[1]<<<d_grid[1],d_block[1]>>>(d_rhsr3[0],d_rhsu3[0],d_rhsv3[0],d_rhsw3[0],d_rhse3[0],d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,d_dil,dpdz);
		RHSDeviceDir[2]<<<d_grid[2],d_block[2]>>>(d_rhsr3[0],d_rhsu3[0],d_rhsv3[0],d_rhsw3[0],d_rhse3[0],d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,d_dil,dpdz);
#endif
    	for (int d=0; d<fin; d++) {
    		rk3final<<<grid0,block0>>>(d_r,d_rO,d_rhsr1[d],d_rhsr2[d],d_rhsr3[d],dtC,d);
    		rk3final<<<grid0,block0>>>(d_e,d_eO,d_rhse1[d],d_rhse2[d],d_rhse3[d],dtC,d); 	}
		hipDeviceSynchronize();
		for (int d=0; d<fin; d++) {
    		rk3finalR<<<grid0,block0,0,s[0]>>>(d_u,d_uO,d_rhsu1[d],d_rhsu2[d],d_rhsu3[d],d_r,dtC,d);
    		rk3finalR<<<grid0,block0,0,s[1]>>>(d_v,d_vO,d_rhsv1[d],d_rhsv2[d],d_rhsv3[d],d_r,dtC,d);
    		rk3finalR<<<grid0,block0,0,s[2]>>>(d_w,d_wO,d_rhsw1[d],d_rhsw2[d],d_rhsw3[d],d_r,dtC,d); }
    	hipDeviceSynchronize();
	}
}

__global__ void eulerSum(myprec *a, myprec *b, myprec *c, myprec *dt, int i) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	if(i==0) {
	    a[id.g] = (b[id.g] + c[id.g]*(*dt)/2.0);
	} else {
	    a[id.g] += ( c[id.g]*(*dt)/2.0 );
	}
}

__global__ void eulerSumR(myprec *a, myprec *b, myprec *c, myprec *r, myprec *dt, int i) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	if(i==0) {
	    a[id.g] = (b[id.g] + c[id.g]*(*dt)/2.0)/r[id.g];
	} else {
	    a[id.g] += ( c[id.g]*(*dt)/2.0 )/r[id.g];
	}
}

__global__ void eulerSum3(myprec *a, myprec *b, myprec *c1, myprec *c2, myprec *dt, int i) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	if(i==0) {
		a[id.g] = b[id.g] + (2*c2[id.g] - c1[id.g])*(*dt);
	} else {
		a[id.g] +=  ( 2*c2[id.g] - c1[id.g] )*(*dt);
	}
}

__global__ void eulerSum3R(myprec *a, myprec *b, myprec *c1, myprec *c2, myprec *r, myprec *dt, int i) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	if(i==0 ) {
		a[id.g] = ( b[id.g] + (2*c2[id.g] - c1[id.g])*(*dt) )/r[id.g];
	} else {
		a[id.g] +=  ( 2*c2[id.g] - c1[id.g] )*(*dt) / r[id.g];
	}
}

__global__ void rk3final(myprec *a1, myprec *a2, myprec *b, myprec *c, myprec *d, myprec *dt, int i) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	if(i==0) {
		a1[id.g] = a2[id.g] + (*dt)*( b[id.g] + 4*c[id.g] + d[id.g])/6.;
	} else {
		a1[id.g] +=  (*dt)*( b[id.g] + 4*c[id.g] + d[id.g] )/6. ;
	}
}

__global__ void rk3finalR(myprec *a1, myprec *a2, myprec *b, myprec *c, myprec *d, myprec *r, myprec *dt, int i) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	if(i==0) {
		a1[id.g] = ( a2[id.g] + (*dt)*( b[id.g] + 4*c[id.g] + d[id.g] )/6. )/ r[id.g];
	} else {
		a1[id.g] += ( (*dt)*( b[id.g] + 4*c[id.g] + d[id.g] )/6. )/ r[id.g];
	}
}

__global__ void calcState(myprec *rho, myprec *uvel, myprec *vvel, myprec *wvel, myprec *ret, myprec *ht, myprec *tem, myprec *pre, myprec *mu, myprec *lam, int bc) {

	int threadsPerBlock  = blockDim.x * blockDim.y;
	int threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
	int blockNumInGrid   = blockIdx.x  + gridDim.x  * blockIdx.y;

	int gl = blockNumInGrid * threadsPerBlock + threadNumInBlock;

	if(bc==1) gl += mx*my*mz;

    myprec cvInv = (gamma - 1.0)/Rgas;

    myprec invrho = 1.0/rho[gl];

    myprec en = ret[gl]*invrho - 0.5*(uvel[gl]*uvel[gl] + vvel[gl]*vvel[gl] + wvel[gl]*wvel[gl]);
    tem[gl]   = cvInv*en;
    pre[gl]   = rho[gl]*Rgas*tem[gl];
    ht[gl]    = (ret[gl] + pre[gl])*invrho;

    myprec suth = pow(tem[gl],viscexp);
    mu[gl]      = suth/Re;
    lam[gl]     = suth/Re/Pr/Ec;
    __syncthreads();

}

void calcTimeStepPressGrad(int istep, myprec *dtC, myprec *dpdz, myprec *h_dt, myprec *h_dpdz, Communicator rk) {

	hipSetDevice(rk.nodeRank);
	calcTimeStep(dtC,d_r,d_u,d_v,d_w,d_e,d_m,rk);
	hipMemcpy(h_dt  , dtC , sizeof(myprec), hipMemcpyDeviceToHost);
	allReduceToMin(h_dt,1);
	mpiBarrier();
	hipMemcpy(dtC , h_dt  , sizeof(myprec), hipMemcpyHostToDevice);
	if(forcing) {
		calcPressureGrad(dpdz,d_r,d_w,rk);
		hipMemcpy(h_dpdz, dpdz, sizeof(myprec), hipMemcpyDeviceToHost);
		allReduceArray(h_dpdz,1);
		mpiBarrier();
		hipMemcpy(dpdz, h_dpdz, sizeof(myprec), hipMemcpyHostToDevice);
	}
	if(rk.rank==0) printf("step number %d with %le %le\n",istep,*h_dt,*h_dpdz);
}

void solverWrapper(Communicator rk) {

	hipSetDevice(rk.nodeRank);

	int start;
    myprec *dpar1, *dpar2, *dtime;
    myprec *hpar1 = new myprec[nsteps];
    myprec *hpar2 = new myprec[nsteps];
    myprec *htime = new myprec[nsteps];

    checkCuda( hipMalloc((void**)&dpar1, nsteps*sizeof(myprec)) );
    checkCuda( hipMalloc((void**)&dpar2, nsteps*sizeof(myprec)) );
    checkCuda( hipMalloc((void**)&dtime, nsteps*sizeof(myprec)) );

    FILE *fp;

    //check the memory usage of the GPU
    checkGpuMem(rk);

    if(restartFile<0) {
    	start=0;
    } else {
    	start=restartFile;
    }

//  checkCuda( hipFuncSetCacheConfig(reinterpret_cast<const void*>( RHSDeviceSharedFlxX), hipFuncCachePreferShared ) );
//	checkCuda( hipFuncSetCacheConfig(reinterpret_cast<const void*>( RHSDeviceSharedFlxY), hipFuncCachePreferShared ) );
//	checkCuda( hipFuncSetCacheConfig(reinterpret_cast<const void*>( RHSDeviceSharedFlxZ), hipFuncCachePreferShared ) );

    if(rk.rank==0) fp = fopen("solution.txt","w+");
    for(int file = start+1; file<nfiles+start+1; file++) {

    	runSimulation(dpar1,dpar2,dtime,rk);  //running the simulation on the GPU
    	copyField(1,rk);					  //copying back partial results to CPU

    	writeField(file,rk);

    	hipDeviceSynchronize();

    	checkCuda( hipMemcpy(htime, dtime, nsteps*sizeof(myprec) , hipMemcpyDeviceToHost) );
    	checkCuda( hipMemcpy(hpar1, dpar1, nsteps*sizeof(myprec) , hipMemcpyDeviceToHost) );
    	checkCuda( hipMemcpy(hpar2, dpar2, nsteps*sizeof(myprec) , hipMemcpyDeviceToHost) );

    	calcAvgChan(rk);

    	if(rk.rank==0) {
    		printf("file number: %d  \t step: %d  \t time: %lf  \t kin: %le  \t energy: %le\n",file,file*nsteps,htime[nsteps-1],hpar1[nsteps-1],hpar2[nsteps-1]);
    		for(int t=0; t<nsteps-1; t+=checkCFLcondition)
    			fprintf(fp,"%d %lf %lf %lf %lf\n",file*(t+1),htime[t],hpar1[t],hpar2[t],htime[t+1]-htime[t]);
    	}
    	mpiBarrier();
    }
    if(rk.rank==0) fclose(fp);

    clearSolver(rk);
    hipDeviceReset();
}




