#include "hip/hip_runtime.h"
#include "globals.h"
#include "cuda_functions.h"
#include "cuda_globals.h"
#include "cuda_main.h"
#include "cuda_math.h"
#include "comm.h"
#include "sponge.h"



__device__ __constant__ myprec alpha[] = {0.    , -17./60., -5./12.};
__device__ __constant__ myprec beta[]  = {8./15.,   5./12.,  3./4. };

hipStream_t s[8+nDivZ];

inline void calcRHS(myprec *rhsr, myprec *rhsu, myprec *rhsv, myprec *rhsw, myprec *rhse, Communicator rk) {
	calcState<<<grid0,block0,0,s[0]>>>(d_r,d_u,d_v,d_w,d_e,d_h,d_t,d_p,d_m,d_l,0); //here 0 means interior points
	//derVelX<<<d_grid[0],d_block[0],0,s[1]>>>(d_u,d_v,d_w,gij[0],gij[1],gij[2]);
	derVelY<<<d_grid[3],d_block[3],0,s[2]>>>(d_u,d_v,d_w,gij[3],gij[4],gij[5]);
	//for (int kNum=0; kNum<1; kNum++) //CHANGED!!!!!
	derVelZ<<<d_grid[4],d_block[4],0,s[8]>>>(d_u,d_v,d_w,gij[6],gij[7],gij[8]);
	if(multiGPU) {
		updateHaloFive(d_r,d_u,d_v,d_w,d_e,rk); hipDeviceSynchronize();
		calcState<<<gridHalo,blockHalo,0,s[4]>>>(d_r,d_u,d_v,d_w,d_e,d_h,d_t,d_p,d_m,d_l,1); //here 1 means halo points
		derVelYBC<<<gridHaloY,blockHaloY,0,s[0]>>>(d_u,d_v,d_w,gij[3],gij[4],gij[5],0);  //here 0 means lower boundary (0-index)
		derVelZBC<<<gridHaloZ,blockHaloZ,0,s[1]>>>(d_u,d_v,d_w,gij[6],gij[7],gij[8],0);	//here 0 means lower boundary (0-index)
		derVelYBC<<<gridHaloY,blockHaloY,0,s[2]>>>(d_u,d_v,d_w,gij[3],gij[4],gij[5],1);	//here 1 means upper boundary (my-index)
		derVelZBC<<<gridHaloZ,blockHaloZ,0,s[3]>>>(d_u,d_v,d_w,gij[6],gij[7],gij[8],1);	//here 1 means upper boundary (mz-index)
	}
	hipDeviceSynchronize();
	//calcDil<<<grid0,block0>>>(d_dil,gij[0],gij[4],gij[8]);
	hipDeviceSynchronize();
	if(multiGPU) deviceBlocker<<<grid0,block0,0,s[0]>>>();   //in order to hide the halo update with deviceRHSX (on stream s[0])
	deviceRHSX<<<d_grid[0],d_block[0],0,s[0]>>>(rhsr,rhsu,rhsv,rhsw,rhse,d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,gij[0],gij[1],gij[2],gij[3],gij[6],gij[4],gij[8],d_dil,dpdz,0);
	if(multiGPU) updateHalo(d_dil,rk);
	hipDeviceSynchronize();
	deviceRHSY<<<d_grid[1],d_block[1]>>>(rhsr,rhsu,rhsv,rhsw,rhse,d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,gij[1],gij[3],gij[4],gij[5],gij[7],d_dil,dpdz);
        hipDeviceSynchronize();	
        deviceRHSZ<<<d_grid[2],d_block[2],0,s[8]>>>(rhsr,rhsu,rhsv,rhsw,rhse,d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,gij[2],gij[5],gij[6],gij[7],gij[8],d_dil,dpdz);
	hipDeviceSynchronize();
	if(boundaryLayer) addSponge<<<d_grid[0],d_block[0]>>>(rhsr,rhsu,rhsv,rhsw,rhse,d_r,d_u,d_v,d_w,d_e);
	hipDeviceSynchronize();
}

void runSimulation(int file , myprec *par1, myprec *par2, myprec *time, Communicator rk) {

          hipSetDevice(rk.nodeRank);
          myprec h_dt,h_dpdz;
  
          for (int istep = 0; istep < nsteps; istep++) {
  
          	if(istep%checkCFLcondition==0) calcTimeStepPressGrad(istep,dtC,dpdz,&h_dt,&h_dpdz,rk);
          	if(istep>0)  deviceSumOne<<<1,1>>>(&time[istep],&time[istep-1] ,dtC);
          	if(istep==0) deviceSumOne<<<1,1>>>(&time[istep],&time[nsteps-1],dtC);
          	deviceAdvanceTime<<<1,1>>>(dtC);
          	if(istep%checkBulk==0) calcBulk(&par1[istep],&par2[istep],d_r,d_u,d_v,d_w,d_e,&h_dt,&h_dpdz,file, istep, rk);
		
		
		deviceMul<<<grid0,block0,0,s[0]>>>(d_uO,d_r,d_u);
		deviceMul<<<grid0,block0,0,s[1]>>>(d_vO,d_r,d_v);
		deviceMul<<<grid0,block0,0,s[2]>>>(d_wO,d_r,d_w);
		deviceCpy<<<grid0,block0,0,s[3]>>>(d_rO,d_r);
		deviceCpy<<<grid0,block0,0,s[4]>>>(d_eO,d_e);

		//Starting the Runge-Kutta Steps

		//runge kutta step 1
		calcRHS(d_rhsr1,d_rhsu1,d_rhsv1,d_rhsw1,d_rhse1,rk);
		//eulerSum<<<grid0,block0>>>(d_r,d_rO,d_rhsr1,dtC);
		//eulerSum<<<grid0,block0>>>(d_e,d_eO,d_rhse1,dtC);

                eulerSumAll<<<grid0,block0>>>(d_r, d_rO, d_u, d_uO, d_v, d_vO, d_w, d_wO, d_e, d_eO, 
                            d_rhsr1, d_rhsu1, d_rhsv1, d_rhsw1, d_rhse1,dtC);
		//hipDeviceSynchronize();
		//eulerSumR<<<grid0,block0,0,s[0]>>>(d_u,d_uO,d_rhsu1,d_r,dtC);
		//eulerSumR<<<grid0,block0,0,s[1]>>>(d_v,d_vO,d_rhsv1,d_r,dtC);
		//eulerSumR<<<grid0,block0,0,s[2]>>>(d_w,d_wO,d_rhsw1,d_r,dtC);
		hipDeviceSynchronize();

		if(multiGPU) {  //To initiate slowly the routines so that we have time to initiate the memory transfer
			deviceCpy<<<grid0,block0,0,s[0]>>>(d_r,d_r);
			deviceCpy<<<grid0,block0,0,s[1]>>>(d_u,d_u);
			deviceCpy<<<grid0,block0,0,s[2]>>>(d_v,d_v);
			deviceCpy<<<grid0,block0,0,s[3]>>>(d_w,d_w); }

		//runge kutta step 2
		calcRHS(d_rhsr2,d_rhsu2,d_rhsv2,d_rhsw2,d_rhse2,rk);
		//eulerSum3<<<grid0,block0>>>(d_r,d_rO,d_rhsr1,d_rhsr2,dtC);
		//eulerSum3<<<grid0,block0>>>(d_e,d_eO,d_rhse1,d_rhse2,dtC);
		eulerSumAll2<<<grid0,block0>>>(d_r, d_rO, d_u, d_uO, d_v, d_vO, d_w, d_wO, d_e, d_eO, 
                            d_rhsr1, d_rhsu1, d_rhsv1, d_rhsw1, d_rhse1,d_rhsr2, d_rhsu2, d_rhsv2, d_rhsw2, d_rhse2, dtC);
                //hipDeviceSynchronize();
		//eulerSum3R<<<grid0,block0,0,s[0]>>>(d_u,d_uO,d_rhsu1,d_rhsu2,d_r,dtC);
		//eulerSum3R<<<grid0,block0,0,s[1]>>>(d_v,d_vO,d_rhsv1,d_rhsv2,d_r,dtC);
		//eulerSum3R<<<grid0,block0,0,s[2]>>>(d_w,d_wO,d_rhsw1,d_rhsw2,d_r,dtC);
		hipDeviceSynchronize();

		if(multiGPU) {  //To initiate slowly the routines so that we have time to initiate the memory transfer
			deviceCpy<<<grid0,block0,0,s[0]>>>(d_r,d_r);
			deviceCpy<<<grid0,block0,0,s[1]>>>(d_u,d_u);
			deviceCpy<<<grid0,block0,0,s[2]>>>(d_v,d_v);
			deviceCpy<<<grid0,block0,0,s[3]>>>(d_w,d_w); }

		//runge kutta step 3
		calcRHS(d_rhsr3,d_rhsu3,d_rhsv3,d_rhsw3,d_rhse3,rk);
//		rk3final<<<grid0,block0>>>(d_r,d_rO,d_rhsr1,d_rhsr2,d_rhsr3,dtC);
//		rk3final<<<grid0,block0>>>(d_e,d_eO,d_rhse1,d_rhse2,d_rhse3,dtC);
		eulerSumAll3<<<grid0,block0>>>(d_r, d_rO, d_u, d_uO, d_v, d_vO, d_w, d_wO, d_e, d_eO, 
                            d_rhsr1, d_rhsu1, d_rhsv1, d_rhsw1, d_rhse1,d_rhsr2, d_rhsu2, d_rhsv2, d_rhsw2, d_rhse2,
                            d_rhsr3, d_rhsu3, d_rhsv3, d_rhsw3, d_rhse3,dtC);

//		hipDeviceSynchronize();
//		rk3finalR<<<grid0,block0,0,s[0]>>>(d_u,d_uO,d_rhsu1,d_rhsu2,d_rhsu3,d_r,dtC);
//		rk3finalR<<<grid0,block0,0,s[1]>>>(d_v,d_vO,d_rhsv1,d_rhsv2,d_rhsv3,d_r,dtC);
//		rk3finalR<<<grid0,block0,0,s[2]>>>(d_w,d_wO,d_rhsw1,d_rhsw2,d_rhsw3,d_r,dtC);
		hipDeviceSynchronize();
	}
}

void runSimulationLowStorage(myprec *par1, myprec *par2, myprec *time, Communicator rk) {

	hipSetDevice(rk.nodeRank);

	myprec h_dt,h_dpdz;

	for (int istep = 0; istep < nsteps; istep++) {
/*		if(istep%checkCFLcondition==0) calcTimeStepPressGrad(istep,dtC,dpdz,&h_dt,&h_dpdz,rk);
		if(istep>0)  deviceSumOne<<<1,1>>>(&time[istep],&time[istep-1] ,dtC);
		if(istep==0) deviceSumOne<<<1,1>>>(&time[istep],&time[nsteps-1],dtC);
		deviceAdvanceTime<<<1,1>>>(dtC);
		if(istep%checkBulk==0) calcBulk(&par1[istep],&par2[istep],d_r,d_u,d_v,d_w,d_e,rk);*/

		//Starting the Runge-Kutta Steps

		//runge kutta step 1
		calcRHS(d_rhsr1,d_rhsu1,d_rhsv1,d_rhsw1,d_rhse1,rk);
		deviceMul<<<grid0,block0,0,s[1]>>>(d_u,d_u,d_r);
		deviceMul<<<grid0,block0,0,s[2]>>>(d_v,d_v,d_r);
		deviceMul<<<grid0,block0,0,s[3]>>>(d_w,d_w,d_r);
		sumLowStorageRK3<<<grid0,block0,0,s[0]>>>(d_r, d_rhsr1, d_rhsr1, dtC, 0);
		sumLowStorageRK3<<<grid0,block0,0,s[1]>>>(d_u, d_rhsu1, d_rhsu1, dtC, 0);
		sumLowStorageRK3<<<grid0,block0,0,s[2]>>>(d_v, d_rhsv1, d_rhsv1, dtC, 0);
		sumLowStorageRK3<<<grid0,block0,0,s[3]>>>(d_w, d_rhsw1, d_rhsw1, dtC, 0);
		sumLowStorageRK3<<<grid0,block0,0,s[4]>>>(d_e, d_rhse1, d_rhse1, dtC, 0);
		hipStreamSynchronize(s[0]);
		deviceDiv<<<grid0,block0,0,s[1]>>>(d_u,d_u,d_r);
		deviceDiv<<<grid0,block0,0,s[2]>>>(d_v,d_v,d_r);
		deviceDiv<<<grid0,block0,0,s[3]>>>(d_w,d_w,d_r);
		hipDeviceSynchronize();

		if(multiGPU) {  //To initiate slowly the routines so that we have time to initiate the memory transfer
			deviceCpy<<<grid0,block0,0,s[0]>>>(d_r,d_r);
			deviceCpy<<<grid0,block0,0,s[1]>>>(d_u,d_u);
			deviceCpy<<<grid0,block0,0,s[2]>>>(d_v,d_v);
			deviceCpy<<<grid0,block0,0,s[3]>>>(d_w,d_w); }


		//runge kutta step 2
		calcRHS(d_rhsr2,d_rhsu2,d_rhsv2,d_rhsw2,d_rhse2,rk);
		deviceMul<<<grid0,block0,0,s[1]>>>(d_u,d_u,d_r);
		deviceMul<<<grid0,block0,0,s[2]>>>(d_v,d_v,d_r);
		deviceMul<<<grid0,block0,0,s[3]>>>(d_w,d_w,d_r);
		sumLowStorageRK3<<<grid0,block0,0,s[0]>>>(d_r, d_rhsr1, d_rhsr2, dtC, 1);
		sumLowStorageRK3<<<grid0,block0,0,s[1]>>>(d_u, d_rhsu1, d_rhsu2, dtC, 1);
		sumLowStorageRK3<<<grid0,block0,0,s[2]>>>(d_v, d_rhsv1, d_rhsv2, dtC, 1);
		sumLowStorageRK3<<<grid0,block0,0,s[3]>>>(d_w, d_rhsw1, d_rhsw2, dtC, 1);
		sumLowStorageRK3<<<grid0,block0,0,s[4]>>>(d_e, d_rhse1, d_rhse2, dtC, 1);
		hipStreamSynchronize(s[0]);
		deviceDiv<<<grid0,block0,0,s[1]>>>(d_u,d_u,d_r);
		deviceDiv<<<grid0,block0,0,s[2]>>>(d_v,d_v,d_r);
		deviceDiv<<<grid0,block0,0,s[3]>>>(d_w,d_w,d_r);
		hipDeviceSynchronize();

		if(multiGPU) {  //To initiate slowly the routines so that we have time to initiate the memory transfer
			deviceCpy<<<grid0,block0,0,s[0]>>>(d_r,d_r);
			deviceCpy<<<grid0,block0,0,s[1]>>>(d_u,d_u);
			deviceCpy<<<grid0,block0,0,s[2]>>>(d_v,d_v);
			deviceCpy<<<grid0,block0,0,s[3]>>>(d_w,d_w); }

		//runge kutta step 3
		calcRHS(d_rhsr1,d_rhsu1,d_rhsv1,d_rhsw1,d_rhse1,rk);
		deviceMul<<<grid0,block0,0,s[1]>>>(d_u,d_u,d_r);
		deviceMul<<<grid0,block0,0,s[2]>>>(d_v,d_v,d_r);
		deviceMul<<<grid0,block0,0,s[3]>>>(d_w,d_w,d_r);
		sumLowStorageRK3<<<grid0,block0,0,s[0]>>>(d_r, d_rhsr2, d_rhsr1, dtC, 2);
		sumLowStorageRK3<<<grid0,block0,0,s[1]>>>(d_u, d_rhsu2, d_rhsu1, dtC, 2);
		sumLowStorageRK3<<<grid0,block0,0,s[2]>>>(d_v, d_rhsv2, d_rhsv1, dtC, 2);
		sumLowStorageRK3<<<grid0,block0,0,s[3]>>>(d_w, d_rhsw2, d_rhsw1, dtC, 2);
		sumLowStorageRK3<<<grid0,block0,0,s[4]>>>(d_e, d_rhse2, d_rhse1, dtC, 2);
		hipStreamSynchronize(s[0]);
		deviceDiv<<<grid0,block0,0,s[1]>>>(d_u,d_u,d_r);
		deviceDiv<<<grid0,block0,0,s[2]>>>(d_v,d_v,d_r);
		deviceDiv<<<grid0,block0,0,s[3]>>>(d_w,d_w,d_r);
		hipDeviceSynchronize();
	}
}

__global__ void eulerSumAll(myprec *r, myprec *r0,myprec *u, myprec *u0, myprec *v, myprec *v0,myprec *w, 
                            myprec *w0, myprec *e, myprec *e0, myprec *rhsr1, myprec *rhsu1, myprec *rhsv1, myprec *rhsw1, myprec *rhse1
                            ,myprec *dt) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
        id.mkidX();
        myprec tmp1 = 0;
	tmp1 = (r0[id.g] + rhsr1[id.g]*(*dt)/2.0);
        r[id.g] = tmp1;
	u[id.g] = (u0[id.g] + rhsu1[id.g]*(*dt)/2.0)/tmp1;
	v[id.g] = (v0[id.g] + rhsv1[id.g]*(*dt)/2.0)/tmp1;
	w[id.g] = (w0[id.g] + rhsw1[id.g]*(*dt)/2.0)/tmp1;
        e[id.g] = (e0[id.g] + rhse1[id.g]*(*dt)/2.0);

}


__global__ void eulerSumAll2(myprec *r, myprec *r0,myprec *u, myprec *u0, myprec *v, myprec *v0,myprec *w, 
                            myprec *w0, myprec *e, myprec *e0, myprec *rhsr1, myprec *rhsu1, myprec *rhsv1, myprec *rhsw1, myprec *rhse1,
                            myprec *rhsr2, myprec *rhsu2, myprec *rhsv2, myprec *rhsw2, myprec *rhse2, myprec *dt) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
        id.mkidX();
        myprec tmp1 = 0;
	tmp1 =    ( r0[id.g] + (2*rhsr2[id.g] - rhsr1[id.g])*(*dt) );
        r[id.g] = tmp1;
	u[id.g] = ( u0[id.g] + (2*rhsu2[id.g] - rhsu1[id.g])*(*dt) )/tmp1;
	v[id.g] = ( v0[id.g] + (2*rhsv2[id.g] - rhsv1[id.g])*(*dt) )/tmp1;
	w[id.g] = ( w0[id.g] + (2*rhsw2[id.g] - rhsw1[id.g])*(*dt) )/tmp1;
        e[id.g] = ( e0[id.g] + (2*rhse2[id.g] - rhse1[id.g])*(*dt) );

}


__global__ void eulerSumAll3(myprec *r, myprec *r0,myprec *u, myprec *u0, myprec *v, myprec *v0,myprec *w, 
                            myprec *w0, myprec *e, myprec *e0, myprec *rhsr1, myprec *rhsu1, myprec *rhsv1, myprec *rhsw1, myprec *rhse1,
                            myprec *rhsr2, myprec *rhsu2, myprec *rhsv2, myprec *rhsw2, myprec *rhse2,
                            myprec *rhsr3, myprec *rhsu3, myprec *rhsv3, myprec *rhsw3, myprec *rhse3, myprec *dt) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
        id.mkidX();
        myprec tmp1 = 0;
	tmp1    =    r0[id.g] + (*dt)*( rhsr1[id.g] + 4*rhsr2[id.g] + rhsr3[id.g])/6.;
        r[id.g] = tmp1;
	u[id.g] = (  u0[id.g] + (*dt)*( rhsu1[id.g] + 4*rhsu2[id.g] + rhsu3[id.g])/6.)/tmp1;
	v[id.g] = (  v0[id.g] + (*dt)*( rhsv1[id.g] + 4*rhsv2[id.g] + rhsv3[id.g])/6.)/tmp1;
	w[id.g] = (  w0[id.g] + (*dt)*( rhsw1[id.g] + 4*rhsw2[id.g] + rhsw3[id.g])/6.)/tmp1;
        e[id.g] = (  e0[id.g] + (*dt)*( rhse1[id.g] + 4*rhse2[id.g] + rhse3[id.g])/6.);

}


__global__ void eulerSum(myprec *a, myprec *b, myprec *c, myprec *dt) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	a[id.g] = (b[id.g] + c[id.g]*(*dt)/2.0);
}

__global__ void eulerSumR(myprec *a, myprec *b, myprec *c, myprec *r, myprec *dt) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	a[id.g] = (b[id.g] + c[id.g]*(*dt)/2.0)/r[id.g];
}

__global__ void eulerSum3(myprec *a, myprec *b, myprec *c1, myprec *c2, myprec *dt) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	a[id.g] = b[id.g] + (2*c2[id.g] - c1[id.g])*(*dt);
}

__global__ void eulerSum3R(myprec *a, myprec *b, myprec *c1, myprec *c2, myprec *r, myprec *dt) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	a[id.g] = ( b[id.g] + (2*c2[id.g] - c1[id.g])*(*dt) )/r[id.g];
}

__global__ void rk3final(myprec *a1, myprec *a2, myprec *b, myprec *c, myprec *d, myprec *dt) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	a1[id.g] = a2[id.g] + (*dt)*( b[id.g] + 4*c[id.g] + d[id.g])/6.;
}

__global__ void rk3finalR(myprec *a1, myprec *a2, myprec *b, myprec *c, myprec *d, myprec *r, myprec *dt) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	a1[id.g] = ( a2[id.g] + (*dt)*( b[id.g] + 4*c[id.g] + d[id.g] )/6. )/ r[id.g];
}

__global__ void calcState(myprec *rho, myprec *uvel, myprec *vvel, myprec *wvel, myprec *ret, myprec *ht, myprec *tem, myprec *pre, myprec *mu, myprec *lam, int bc) {

/*	int threadsPerBlock  = blockDim.x * blockDim.y;
	int threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
	int blockNumInGrid   = blockIdx.x  + gridDim.x  * blockIdx.y;

	int gl = blockNumInGrid * threadsPerBlock + threadNumInBlock;

	if(bc==1) gl += mx*my*mz;

    myprec cvInv = (gam - 1.0)/Rgas;

    myprec invrho = 1.0/rho[gl];

    myprec en = ret[gl]*invrho - 0.5*(uvel[gl]*uvel[gl] + vvel[gl]*vvel[gl] + wvel[gl]*wvel[gl]);
    tem[gl]   = cvInv*en;
    pre[gl]   = rho[gl]*Rgas*tem[gl];
    ht[gl]    = (ret[gl] + pre[gl])*invrho;

    myprec suth = pow(tem[gl],viscexp);
    mu[gl]      = suth/Re;
    lam[gl]     = suth/Re/Pr/Ec;
    __syncthreads();*/

	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidX();
    int gl = id.g;
    if(bc==1) gl += mx*my*mz;

    myprec cvInv = (gam - 1.0)/Rgas;
    myprec r = rho[gl];
    myprec u = uvel[gl];
    myprec v = vvel[gl];
    myprec w = wvel[gl];
    myprec etot = ret[gl];

    myprec invrho = 1.0/r;
    myprec en = etot*invrho - 0.5*(u*u + v*v + w*w);

    myprec  t   = cvInv*en;
    myprec  p   = r*Rgas*t; //tmp1 is temperature // tmp2 is pressure    
    tem[gl]     = t;
    pre[gl]     = p;

    ht[gl]      = (etot + p)*invrho;

    myprec suth = pow(t,viscexp);
    mu[gl]      = suth/Re;
    lam[gl]     = suth/Re/Pr/Ec;

    __syncthreads();

}

__global__ void sumLowStorageRK3(myprec *var, myprec *rhs1, myprec *rhs2, myprec *dt, int step) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	var[id.g] = var[id.g] + (*dt)*(alpha[step]*rhs1[id.g] + beta[step]*rhs2[id.g]);
}

void calcTimeStepPressGrad(int istep, myprec *dtC, myprec *dpdz, myprec *h_dt, myprec *h_dpdz, Communicator rk) {

	hipSetDevice(rk.nodeRank);
	calcTimeStep(dtC,d_r,d_u,d_v,d_w,d_e,d_m,rk);
	hipMemcpy(h_dt  , dtC , sizeof(myprec), hipMemcpyDeviceToHost);
	allReduceToMin(h_dt,1);
	mpiBarrier();
	hipMemcpy(dtC , h_dt  , sizeof(myprec), hipMemcpyHostToDevice);
	if(forcing) {
		calcPressureGrad(dpdz,d_r,d_w,rk); //Changed here dpdz_new = dpdz_old - 0.5*(rw_bulk - 1 )
		hipMemcpy(h_dpdz, dpdz, sizeof(myprec), hipMemcpyDeviceToHost);
		allReduceArray(h_dpdz,1);
		mpiBarrier();
		hipMemcpy(dpdz, h_dpdz, sizeof(myprec), hipMemcpyHostToDevice);
	}
	//if(rk.rank==0) printf("step number %d with %le %le\n",nsteps*(file) + istep ,*h_dt,*h_dpdz);
}

void solverWrapper(Communicator rk) {

	hipSetDevice(rk.nodeRank);

	int start;
    myprec *dpar1, *dpar2, *dtime;
    myprec *hpar1 = new myprec[nsteps];
    myprec *hpar2 = new myprec[nsteps];
    myprec *htime = new myprec[nsteps];

    checkCuda( hipMalloc((void**)&dpar1, nsteps*sizeof(myprec)) );
    checkCuda( hipMalloc((void**)&dpar2, nsteps*sizeof(myprec)) );
    checkCuda( hipMalloc((void**)&dtime, nsteps*sizeof(myprec)) );

    FILE *fp;

    //check the memory usage of the GPU
    checkGpuMem(rk);


	/*derVelX<<<d_grid[0],d_block[0],0,s[1]>>>(d_u,d_v,d_w,gij[0],gij[1],gij[2]);
	derVelY<<<d_grid[3],d_block[3],0,s[2]>>>(d_u,d_v,d_w,gij[3],gij[4],gij[5]);
	derVelZ<<<d_grid[4],d_block[4],0,s[8]>>>(d_u,d_v,d_w,gij[6],gij[7],gij[8]);
	calcDil<<<grid0,block0>>>(d_dil,gij[0],gij[4],gij[8]);*/ // In Y and Z derivatives are turned off. so need to fix that or create a new kernel.



    if(restartFile<0) {
    	start=0;
    } else {
    	start=restartFile;
    }
    for(int file = start+1; file<nfiles+start+1; file++) {  
        
        if(rk.rank==0) fp = fopen("solution.txt","w+");

        /////Time----------------------------------------------------------------------------------
	float tm;
	hipEvent_t start1, stop; checkCuda( hipEventCreate(&start1) ); checkCuda( hipEventCreate(&stop) ); checkCuda( hipEventRecord(start1, 0) );

	if(lowStorage) {
        	runSimulationLowStorage(dpar1,dpar2,dtime,rk);  //running the simulation on the GPU
    	} else {
    		runSimulation(file, dpar1,dpar2,dtime,rk);  //running the simulation on the GPU
          	}
       
	checkCuda( hipEventRecord(stop, 0) );	checkCuda( hipEventSynchronize(stop) );	checkCuda( hipEventElapsedTime(&tm, start1, stop) );
	printf("Time for runSimulation  %3.4f s \n", tm/nsteps/1000);
        // EndTime---------------------------------------------------------------------------------

    	copyField(1,rk);					  //copying back partial results to CPU

    	writeField(file,rk);

    	hipDeviceSynchronize();

    	checkCuda( hipMemcpy(htime, dtime, nsteps*sizeof(myprec) , hipMemcpyDeviceToHost) );
    	checkCuda( hipMemcpy(hpar1, dpar1, nsteps*sizeof(myprec) , hipMemcpyDeviceToHost) );
    	checkCuda( hipMemcpy(hpar2, dpar2, nsteps*sizeof(myprec) , hipMemcpyDeviceToHost) );

    	calcAvgChan(rk);

    	if(rk.rank==0) {
    		printf("file number: %d  \t step: %d  \t time: %lf  \t kin: %le  \t energy: %le\n",file,file*nsteps,htime[nsteps-1],hpar1[nsteps-1],hpar2[nsteps-1]);
    		for(int t=0; t<nsteps-1; t+=checkCFLcondition)
    			fprintf(fp,"%d %lf %lf %lf %lf\n",file*(t+1),htime[t],hpar1[t],hpar2[t],htime[t+1]-htime[t]);
    	}
    	mpiBarrier();
    }
    if(rk.rank==0) fclose(fp);

    clearSolver(rk);
    hipDeviceReset();
}




