#include "hip/hip_runtime.h"
#include "globals.h"
#include "cuda_functions.h"
#include "cuda_globals.h"
#include "cuda_main.h"
#include "cuda_math.h"

#if (capability>capabilityMin)
__global__ void runDevice(myprec *kin, myprec *enst, myprec *time) {

	dtC = d_dt;

	/* allocating temporary arrays and streams */
	void (*RHSDeviceDir[3])(myprec*, myprec*, myprec*, myprec*, myprec*, myprec*, myprec*, myprec*,
							myprec*, myprec*, myprec*, myprec*, myprec*, myprec*, myprec**, myprec*, myprec);

	RHSDeviceDir[0] = RHSDeviceSharedFlxX;
	RHSDeviceDir[1] = RHSDeviceSharedFlxY;
	RHSDeviceDir[2] = RHSDeviceSharedFlxZ;

	__syncthreads();

	hipStream_t s[3];
    for (int i=0; i<3; i++) {
    	checkCudaDev( hipStreamCreateWithFlags(&s[i], hipStreamNonBlocking) );
    }

	initSolver();
	initStress();

    for (int istep = 0; istep < nsteps; istep++) {

    	calcState<<<grid0,block0>>>(d_r,d_u,d_v,d_w,d_e,d_h,d_t,d_p,d_m,d_l);
    	hipDeviceSynchronize();

    	if(istep%checkCFLcondition==0) {
    		calcTimeStep(&dtC,d_r,d_u,d_v,d_w,d_e,d_m);
    		if(forcing)  calcPressureGrad(&dpdz,d_r,d_w);
    	}

    	dt2 = dtC/2.;
    	if(istep==0) {
    		time[istep] = time[nsteps-1] + dtC;
    	} else{
    		time[istep] = time[istep-1] + dtC; }

    	deviceMul<<<grid0,block0>>>(d_uO,d_r,d_u);
    	deviceMul<<<grid0,block0>>>(d_vO,d_r,d_v);
    	deviceMul<<<grid0,block0>>>(d_wO,d_r,d_w);
    	deviceCpy<<<grid0,block0>>>(d_rO,d_r);
    	deviceCpy<<<grid0,block0>>>(d_eO,d_e);

    	/* rk step 1 */
    	hipDeviceSynchronize();
		calcStressX<<<d_grid[0],d_block[0],0,s[0]>>>(d_u,d_v,d_w,sij);
		calcStressY<<<d_grid[3],d_block[3],0,s[1]>>>(d_u,d_v,d_w,sij);
		calcStressZ<<<d_grid[4],d_block[4],0,s[2]>>>(d_u,d_v,d_w,sij);
    	hipDeviceSynchronize();

    	if(istep%checkCFLcondition==0) {
    		calcIntegrals(d_r,d_u,d_v,d_w,sij,&kin[istep],&enst[istep]);
    		enst[istep] = dpdz;
    	}
    	hipDeviceSynchronize();

    	calcDil<<<grid0,block0>>>(sij,d_dil);
    	hipDeviceSynchronize();

    	for (int d = 0; d < 3; d++)
    		RHSDeviceDir[d]<<<d_grid[d],d_block[d],0,s[d]>>>(d_rhsr1[d],d_rhsu1[d],d_rhsv1[d],d_rhsw1[d],d_rhse1[d],d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,sij,d_dil,dpdz);
    	hipDeviceSynchronize();
    	eulerSum<<<grid0,block0>>>(d_r,d_rO,d_rhsr1,&dt2);
    	hipDeviceSynchronize();
    	eulerSum<<<grid0,block0>>>(d_e,d_eO,d_rhse1,&dt2);
    	eulerSumR<<<grid0,block0>>>(d_u,d_uO,d_rhsu1,d_r,&dt2);
    	eulerSumR<<<grid0,block0>>>(d_v,d_vO,d_rhsv1,d_r,&dt2);
    	eulerSumR<<<grid0,block0>>>(d_w,d_wO,d_rhsw1,d_r,&dt2);
    	hipDeviceSynchronize();

    	//rk step 2
    	calcState<<<grid0,block0>>>(d_r,d_u,d_v,d_w,d_e,d_h,d_t,d_p,d_m,d_l);
		calcStressX<<<d_grid[0],d_block[0],0,s[0]>>>(d_u,d_v,d_w,sij);
		calcStressY<<<d_grid[3],d_block[3],0,s[1]>>>(d_u,d_v,d_w,sij);
		calcStressZ<<<d_grid[4],d_block[4],0,s[2]>>>(d_u,d_v,d_w,sij);
		hipDeviceSynchronize();
    	calcDil<<<grid0,block0>>>(sij,d_dil);
    	hipDeviceSynchronize();
    	for (int d = 0; d < 3; d++)
    		RHSDeviceDir[d]<<<d_grid[d],d_block[d],0,s[d]>>>(d_rhsr2[d],d_rhsu2[d],d_rhsv2[d],d_rhsw2[d],d_rhse2[d],d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,sij,d_dil,dpdz);
    	hipDeviceSynchronize();
#if rk==4
    	eulerSum<<<grid0,block0>>>(d_r,d_rO,d_rhsr2,&dt2);
    	hipDeviceSynchronize();
    	eulerSum<<<grid0,block0>>>(d_e,d_eO,d_rhse2,&dt2);
    	eulerSumR<<<grid0,block0>>>(d_u,d_uO,d_rhsu2,d_r,&dt2);
    	eulerSumR<<<grid0,block0>>>(d_v,d_vO,d_rhsv2,d_r,&dt2);
    	eulerSumR<<<grid0,block0>>>(d_w,d_wO,d_rhsw2,d_r,&dt2);
#elif rk==3
    	eulerSum3<<<grid0,block0>>>(d_r,d_rO,d_rhsr1,d_rhsr2,&dtC);
    	hipDeviceSynchronize();
    	eulerSum3<<<grid0,block0>>>(d_e,d_eO,d_rhse1,d_rhse2,&dtC);
    	eulerSum3R<<<grid0,block0>>>(d_u,d_uO,d_rhsu1,d_rhsu2,d_r,&dtC);
    	eulerSum3R<<<grid0,block0>>>(d_v,d_vO,d_rhsv1,d_rhsv2,d_r,&dtC);
    	eulerSum3R<<<grid0,block0>>>(d_w,d_wO,d_rhsw1,d_rhsw2,d_r,&dtC);
#endif
    	hipDeviceSynchronize();

    	//rk step 3
    	calcState<<<grid0,block0>>>(d_r,d_u,d_v,d_w,d_e,d_h,d_t,d_p,d_m,d_l);
		calcStressX<<<d_grid[0],d_block[0],0,s[0]>>>(d_u,d_v,d_w,sij);
		calcStressY<<<d_grid[3],d_block[3],0,s[1]>>>(d_u,d_v,d_w,sij);
		calcStressZ<<<d_grid[4],d_block[4],0,s[2]>>>(d_u,d_v,d_w,sij);
    	hipDeviceSynchronize();
    	calcDil<<<grid0,block0>>>(sij,d_dil);
    	hipDeviceSynchronize();
    	for (int d = 0; d < 3; d++)
    		RHSDeviceDir[d]<<<d_grid[d],d_block[d],0,s[d]>>>(d_rhsr3[d],d_rhsu3[d],d_rhsv3[d],d_rhsw3[d],d_rhse3[d],d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,sij,d_dil,dpdz);
    	hipDeviceSynchronize();
#if rk==4
    	eulerSum<<<grid0,block0>>>(d_r,d_rO,d_rhsr3,&dtC);
    	hipDeviceSynchronize();
    	eulerSum<<<grid0,block0>>>(d_e,d_eO,d_rhse3,&dtC);
    	eulerSumR<<<grid0,block0>>>(d_u,d_uO,d_rhsu3,d_r,&dtC);
    	eulerSumR<<<grid0,block0>>>(d_v,d_vO,d_rhsv3,d_r,&dtC);
    	eulerSumR<<<grid0,block0>>>(d_w,d_wO,d_rhsw3,d_r,&dtC);
    	hipDeviceSynchronize();

    	//rk step 4
    	calcState<<<grid0,block0>>>(d_r,d_u,d_v,d_w,d_e,d_h,d_t,d_p,d_m,d_l);
		calcStressX<<<d_grid[0],d_block[0],0,s[0]>>>(d_u,d_v,d_w,sij);
		calcStressY<<<d_grid[3],d_block[3],0,s[1]>>>(d_u,d_v,d_w,sij);
		calcStressZ<<<d_grid[4],d_block[4],0,s[2]>>>(d_u,d_v,d_w,sij);
    	hipDeviceSynchronize();
    	calcDil<<<grid0,block0>>>(sij,d_dil);
    	hipDeviceSynchronize();
    	for (int d = 0; d < 3; d++)
    		RHSDeviceDir[d]<<<d_grid[d],d_block[d],0,s[d]>>>(d_rhsr4[d],d_rhsu4[d],d_rhsv4[d],d_rhsw4[d],d_rhse4[d],d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,sij,d_dil,dpdz);
    	hipDeviceSynchronize();
    	rk4final<<<grid0,block0>>>(d_r,d_rO,d_rhsr1,d_rhsr2,d_rhsr3,d_rhsr4,&dtC);
    	hipDeviceSynchronize();
    	rk4final<<<grid0,block0>>>(d_e,d_eO,d_rhse1,d_rhse2,d_rhse3,d_rhse4,&dtC);
    	rk4finalR<<<grid0,block0>>>(d_u,d_uO,d_rhsu1,d_rhsu2,d_rhsu3,d_rhsu4,d_r,&dtC);
    	rk4finalR<<<grid0,block0>>>(d_v,d_vO,d_rhsv1,d_rhsv2,d_rhsv3,d_rhsv4,d_r,&dtC);
    	rk4finalR<<<grid0,block0>>>(d_w,d_wO,d_rhsw1,d_rhsw2,d_rhsw3,d_rhsw4,d_r,&dtC);
#elif rk==3
    	rk3final<<<grid0,block0>>>(d_r,d_rO,d_rhsr1,d_rhsr2,d_rhsr3,&dtC);
    	hipDeviceSynchronize();
    	rk3final<<<grid0,block0>>>(d_e,d_eO,d_rhse1,d_rhse2,d_rhse3,&dtC);
    	rk3finalR<<<grid0,block0>>>(d_u,d_uO,d_rhsu1,d_rhsu2,d_rhsu3,d_r,&dtC);
    	rk3finalR<<<grid0,block0>>>(d_v,d_vO,d_rhsv1,d_rhsv2,d_rhsv3,d_r,&dtC);
    	rk3finalR<<<grid0,block0>>>(d_w,d_wO,d_rhsw1,d_rhsw2,d_rhsw3,d_r,&dtC);
#endif
    	hipDeviceSynchronize();

	}
    __syncthreads();

	for (int i=0; i<3; i++) {
		checkCudaDev( hipStreamDestroy(s[i]) );
	}

	clearSolver();
	clearStress();
}
#else
__global__ void runDevice(myprec *kin, myprec *enst, myprec *time) {

	dtC = d_dt;

	dim3 gr[5],bl[5],gr0,bl0;

	gr[0] = dim3(d_grid[0],d_grid[1],1);
	gr[1] = dim3(d_grid[4],d_grid[5],1);
	gr[2] = dim3(d_grid[8],d_grid[9],1);
	gr[3] = dim3(d_grid[2],d_grid[3],1);
	gr[4] = dim3(d_grid[6],d_grid[7],1);

	bl[0] = dim3(d_block[0],d_block[1],1);
	bl[1] = dim3(d_block[4],d_block[5],1);
	bl[2] = dim3(d_block[8],d_block[9],1);
	bl[3] = dim3(d_block[2],d_block[3],1);
	bl[4] = dim3(d_block[6],d_block[7],1);

	gr0 = dim3(grid0[0],grid0[1],1); bl0 = dim3(block0[0],block0[1],1);

	/* allocating temporary arrays and streams */
	void (*RHSDeviceDir[3])(myprec*, myprec*, myprec*, myprec*, myprec*, myprec*, myprec*, myprec*,
							myprec*, myprec*, myprec*, myprec*, myprec*, myprec*, myprec**, myprec*, myprec*);


	RHSDeviceDir[0] = RHSDeviceSharedFlxX;
	RHSDeviceDir[1] = RHSDeviceSharedFlxY;
	RHSDeviceDir[2] = RHSDeviceSharedFlxZ;

	__syncthreads();

	hipStream_t s[3];
    for (int i=0; i<3; i++) {
    	checkCudaDev( hipStreamCreateWithFlags(&s[i], hipStreamNonBlocking) );
    }

    initSolver();
    initStress();

    for (int istep = 0; istep < nsteps; istep++) {

    	calcState<<<gr0,bl0>>>(d_r,d_u,d_v,d_w,d_e,d_h,d_t,d_p,d_m,d_l);
    	hipDeviceSynchronize();

    	if(istep%checkCFLcondition==0) {
    		calcTimeStep(&dtC,d_r,d_u,d_v,d_w,d_e,d_m);
    		if(forcing)  calcPressureGrad(&dpdz,d_r,d_w);
    	}
    	dt2 = dtC/2.;
    	if(istep==0) {
    		time[istep] = time[nsteps-1] + dtC;
    	} else{
    		time[istep] = time[istep-1] + dtC; }

    	deviceMul<<<gr0,bl0>>>(d_uO,d_r,d_u);
    	deviceMul<<<gr0,bl0>>>(d_vO,d_r,d_v);
    	deviceMul<<<gr0,bl0>>>(d_wO,d_r,d_w);
    	deviceCpy<<<gr0,bl0>>>(d_rO,d_r);
    	deviceCpy<<<gr0,bl0>>>(d_eO,d_e);

    	/* rk step 1 */
    	hipDeviceSynchronize();
    	calcStressX<<<gr[0],bl[0],0,s[0]>>>(d_u,d_v,d_w,sij);
    	calcStressY<<<gr[3],bl[3],0,s[1]>>>(d_u,d_v,d_w,sij);
    	calcStressZ<<<gr[4],bl[4],0,s[2]>>>(d_u,d_v,d_w,sij);
    	hipDeviceSynchronize();

    	if(istep%checkCFLcondition==0) {
    		calcIntegrals(d_r,d_u,d_v,d_w,sij,&kin[istep],&enst[istep]);
    		enst[istep] = dpdz;
    	}
    	hipDeviceSynchronize();

    	calcDil<<<gr0,bl0>>>(sij,d_dil);
    	hipDeviceSynchronize();

    	for (int d = 0; d < 3; d++)
    		RHSDeviceDir[d]<<<gr[d],bl[d],0,s[d]>>>(d_rhsr1[d],d_rhsu1[d],d_rhsv1[d],d_rhsw1[d],d_rhse1[d],d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,sij,d_dil,dpdz);
    	hipDeviceSynchronize();
    	eulerSum<<<gr0,bl0>>>(d_r,d_rO,d_rhsr1,&dt2);
    	hipDeviceSynchronize();
    	eulerSum<<<gr0,bl0>>>(d_e,d_eO,d_rhse1,&dt2);
    	eulerSumR<<<gr0,bl0>>>(d_u,d_uO,d_rhsu1,d_r,&dt2);
    	eulerSumR<<<gr0,bl0>>>(d_v,d_vO,d_rhsv1,d_r,&dt2);
    	eulerSumR<<<gr0,bl0>>>(d_w,d_wO,d_rhsw1,d_r,&dt2);
    	hipDeviceSynchronize();

    	//rk step 2
    	calcState<<<gr0,bl0>>>(d_r,d_u,d_v,d_w,d_e,d_h,d_t,d_p,d_m,d_l);
    	calcStressX<<<gr[0],bl[0],0,s[0]>>>(d_u,d_v,d_w,sij);
    	calcStressY<<<gr[3],bl[3],0,s[1]>>>(d_u,d_v,d_w,sij);
    	calcStressZ<<<gr[4],bl[4],0,s[2]>>>(d_u,d_v,d_w,sij);
    	hipDeviceSynchronize();
    	calcDil<<<gr0,bl0>>>(sij,d_dil);
    	hipDeviceSynchronize();
    	for (int d = 0; d < 3; d++)
    		RHSDeviceDir[d]<<<gr[d],bl[d],0,s[d]>>>(d_rhsr2[d],d_rhsu2[d],d_rhsv2[d],d_rhsw2[d],d_rhse2[d],d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,sij,d_dil,dpdz);
    	hipDeviceSynchronize();
#if rk==4
    	eulerSum<<<gr0,bl0>>>(d_r,d_rO,d_rhsr2,&dt2);
    	hipDeviceSynchronize();
    	eulerSum<<<gr0,bl0>>>(d_e,d_eO,d_rhse2,&dt2);
    	eulerSumR<<<gr0,bl0>>>(d_u,d_uO,d_rhsu2,d_r,&dt2);
    	eulerSumR<<<gr0,bl0>>>(d_v,d_vO,d_rhsv2,d_r,&dt2);
    	eulerSumR<<<gr0,bl0>>>(d_w,d_wO,d_rhsw2,d_r,&dt2);
#elif rk==3
    	eulerSum3<<<gr0,bl0>>>(d_r,d_rO,d_rhsr1,d_rhsr2,&dtC);
    	hipDeviceSynchronize();
    	eulerSum3<<<gr0,bl0>>>(d_e,d_eO,d_rhse1,d_rhse2,&dtC);
    	eulerSum3R<<<gr0,bl0>>>(d_u,d_uO,d_rhsu1,d_rhsu2,d_r,&dtC);
    	eulerSum3R<<<gr0,bl0>>>(d_v,d_vO,d_rhsv1,d_rhsv2,d_r,&dtC);
    	eulerSum3R<<<gr0,bl0>>>(d_w,d_wO,d_rhsw1,d_rhsw2,d_r,&dtC);
#endif
    	hipDeviceSynchronize();


    	//rk step 3
    	calcState<<<gr0,bl0>>>(d_r,d_u,d_v,d_w,d_e,d_h,d_t,d_p,d_m,d_l);
    	calcStressX<<<gr[0],bl[0],0,s[0]>>>(d_u,d_v,d_w,sij);
    	calcStressY<<<gr[3],bl[3],0,s[1]>>>(d_u,d_v,d_w,sij);
    	calcStressZ<<<gr[4],bl[4],0,s[2]>>>(d_u,d_v,d_w,sij);
    	hipDeviceSynchronize();
    	calcDil<<<gr0,bl0>>>(sij,d_dil);
    	hipDeviceSynchronize();
    	for (int d = 0; d < 3; d++)
    		RHSDeviceDir[d]<<<gr[d],bl[d],0,s[d]>>>(d_rhsr3[d],d_rhsu3[d],d_rhsv3[d],d_rhsw3[d],d_rhse3[d],d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,sij,d_dil,dpdz);
    	hipDeviceSynchronize();
#if rk==4
    	eulerSum<<<gr0,bl0>>>(d_r,d_rO,d_rhsr3,&dtC);
    	hipDeviceSynchronize();
    	eulerSum<<<gr0,bl0>>>(d_e,d_eO,d_rhse3,&dtC);
    	eulerSumR<<<gr0,bl0>>>(d_u,d_uO,d_rhsu3,d_r,&dtC);
    	eulerSumR<<<gr0,bl0>>>(d_v,d_vO,d_rhsv3,d_r,&dtC);
    	eulerSumR<<<gr0,bl0>>>(d_w,d_wO,d_rhsw3,d_r,&dtC);
    	hipDeviceSynchronize();

    	//rk step 4
    	calcState<<<gr0,bl0>>>(d_r,d_u,d_v,d_w,d_e,d_h,d_t,d_p,d_m,d_l);
    	calcStressX<<<gr[0],bl[0],0,s[0]>>>(d_u,d_v,d_w,sij);
    	calcStressY<<<gr[3],bl[3],0,s[1]>>>(d_u,d_v,d_w,sij);
    	calcStressZ<<<gr[4],bl[4],0,s[2]>>>(d_u,d_v,d_w,sij);
    	hipDeviceSynchronize();
    	calcDil<<<gr0,bl0>>>(sij,d_dil);
    	hipDeviceSynchronize();
    	for (int d = 0; d < 3; d++)
    		RHSDeviceDir[d]<<<gr[d],bl[d],0,s[d]>>>(d_rhsr4[d],d_rhsu4[d],d_rhsv4[d],d_rhsw4[d],d_rhse4[d],d_r,d_u,d_v,d_w,d_h,d_t,d_p,d_m,d_l,sij,d_dil,dpdz);
    	hipDeviceSynchronize();
    	rk4final<<<gr0,bl0>>>(d_r,d_rO,d_rhsr1,d_rhsr2,d_rhsr3,d_rhsr4,&dtC);
    	hipDeviceSynchronize();
    	rk4final<<<gr0,bl0>>>(d_e,d_eO,d_rhse1,d_rhse2,d_rhse3,d_rhse4,&dtC);
    	rk4finalR<<<gr0,bl0>>>(d_u,d_uO,d_rhsu1,d_rhsu2,d_rhsu3,d_rhsu4,d_r,&dtC);
    	rk4finalR<<<gr0,bl0>>>(d_v,d_vO,d_rhsv1,d_rhsv2,d_rhsv3,d_rhsv4,d_r,&dtC);
    	rk4finalR<<<gr0,bl0>>>(d_w,d_wO,d_rhsw1,d_rhsw2,d_rhsw3,d_rhsw4,d_r,&dtC);
#elif rk==3
    	rk3final<<<gr0,bl0>>>(d_r,d_rO,d_rhsr1,d_rhsr2,d_rhsr3,&dtC);
    	hipDeviceSynchronize();
    	rk3final<<<gr0,bl0>>>(d_e,d_eO,d_rhse1,d_rhse2,d_rhse3,&dtC);
    	rk3finalR<<<gr0,bl0>>>(d_u,d_uO,d_rhsu1,d_rhsu2,d_rhsu3,d_r,&dtC);
    	rk3finalR<<<gr0,bl0>>>(d_v,d_vO,d_rhsv1,d_rhsv2,d_rhsv3,d_r,&dtC);
    	rk3finalR<<<gr0,bl0>>>(d_w,d_wO,d_rhsw1,d_rhsw2,d_rhsw3,d_r,&dtC);
#endif
    	hipDeviceSynchronize();

	}
    __syncthreads();

	for (int i=0; i<3; i++) {
		checkCudaDev( hipStreamDestroy(s[i]) );
	}

	clearSolver();
	clearStress();
}
#endif

__global__ void eulerSum(myprec *a, myprec *b, myprec *c[3], myprec *dt) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidX();
	a[id.g] = b[id.g] + ( c[0][id.g] + c[1][id.g] + c[2][id.g] )*(*dt);
}

__global__ void eulerSumR(myprec *a, myprec *b, myprec *c[3], myprec *r, myprec *dt) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidX();
	a[id.g] =  ( b[id.g] +  ( c[0][id.g] + c[1][id.g] + c[2][id.g] ) *(*dt) ) /r[id.g];
}

__global__ void eulerSum3(myprec *a, myprec *b, myprec *c1[3], myprec *c2[3], myprec *dt) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidX();
	a[id.g] = b[id.g] -     ( c1[0][id.g] + c1[1][id.g] + c1[2][id.g] )*(*dt)
					  + 2 * ( c2[0][id.g] + c2[1][id.g] + c2[2][id.g] )*(*dt);
}

__global__ void eulerSum3R(myprec *a, myprec *b, myprec *c1[3], myprec *c2[3], myprec *r, myprec *dt) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidX();
	a[id.g] = ( b[id.g] -     ( c1[0][id.g] + c1[1][id.g] + c1[2][id.g] )*(*dt)
					    + 2 * ( c2[0][id.g] + c2[1][id.g] + c2[2][id.g] )*(*dt) )/ r[id.g];
}

__global__ void rk4final(myprec *a1, myprec *a2, myprec *b[3], myprec *c[3], myprec *d[3], myprec *e[3], myprec *dt) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidX();
	a1[id.g] = a2[id.g];
	for (int it=0; it<3; it++)
		a1[id.g] = a1[id.g] + (*dt)*( b[it][id.g] + 2*c[it][id.g] + 2*d[it][id.g] + e[it][id.g])/6.;
}

__global__ void rk4finalR(myprec *a1, myprec *a2, myprec *b[3], myprec *c[3], myprec *d[3], myprec *e[3], myprec *r, myprec *dt) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidX();
	a1[id.g] =  a2[id.g]/r[id.g];
	for (int it=0; it<3; it++)
		a1[id.g] +=  (*dt)*( b[it][id.g] + 2*c[it][id.g] + 2*d[it][id.g] + e[it][id.g])/6./ r[id.g];
}

__global__ void rk3final(myprec *a1, myprec *a2, myprec *b[3], myprec *c[3], myprec *d[3], myprec *dt) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidX();
	a1[id.g] = a2[id.g];
	for (int it=0; it<3; it++)
		a1[id.g] = a1[id.g] + (*dt)*( b[it][id.g] + 4*c[it][id.g] + d[it][id.g])/6.;
}

__global__ void rk3finalR(myprec *a1, myprec *a2, myprec *b[3], myprec *c[3], myprec *d[3], myprec *r, myprec *dt) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	id.mkidX();
	a1[id.g] = a2[id.g]/r[id.g];
	for (int it=0; it<3; it++)
		a1[id.g] += (*dt)*( b[it][id.g] + 4*c[it][id.g] + d[it][id.g])/6. / r[id.g];
}

__global__ void calcState(myprec *rho, myprec *uvel, myprec *vvel, myprec *wvel, myprec *ret, myprec *ht, myprec *tem, myprec *pre, myprec *mu, myprec *lam) {

	int threadsPerBlock  = blockDim.x * blockDim.y;
	int threadNumInBlock = threadIdx.x + blockDim.x * threadIdx.y;
	int blockNumInGrid   = blockIdx.x  + gridDim.x  * blockIdx.y;

	int gt = blockNumInGrid * threadsPerBlock + threadNumInBlock;

    myprec cvInv = (gamma - 1.0)/Rgas;

    myprec invrho = 1.0/rho[gt];

    myprec en = ret[gt]*invrho - 0.5*(uvel[gt]*uvel[gt] + vvel[gt]*vvel[gt] + wvel[gt]*wvel[gt]);
    tem[gt]   = cvInv*en;
    pre[gt]   = rho[gt]*Rgas*tem[gt];
    ht[gt]    = (ret[gt] + pre[gt])*invrho;

    myprec suth = pow(tem[gt],viscexp);
    mu[gt]      = suth/Re;
    lam[gt]     = suth/Re/Pr/Ec;
    __syncthreads();

}

__device__ void initSolver() {

    for (int i=0; i<3; i++) {
    	checkCudaDev( hipMalloc((void**)&d_rhsr1[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhsu1[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhsv1[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhsw1[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhse1[i],mx*my*mz*sizeof(myprec)) );

    	checkCudaDev( hipMalloc((void**)&d_rhsr2[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhsu2[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhsv2[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhsw2[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhse2[i],mx*my*mz*sizeof(myprec)) );

    	checkCudaDev( hipMalloc((void**)&d_rhsr3[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhsu3[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhsv3[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhsw3[i],mx*my*mz*sizeof(myprec)) );
    	checkCudaDev( hipMalloc((void**)&d_rhse3[i],mx*my*mz*sizeof(myprec)) );
#if rk == 4
    	checkCudaDev( hipMalloc((void**)&d_rhsr4[i],mx*my*mz*sizeof(myprec)) );
		checkCudaDev( hipMalloc((void**)&d_rhsu4[i],mx*my*mz*sizeof(myprec)) );
		checkCudaDev( hipMalloc((void**)&d_rhsv4[i],mx*my*mz*sizeof(myprec)) );
		checkCudaDev( hipMalloc((void**)&d_rhsw4[i],mx*my*mz*sizeof(myprec)) );
		checkCudaDev( hipMalloc((void**)&d_rhse4[i],mx*my*mz*sizeof(myprec)) );
#endif
    }

	checkCudaDev( hipMalloc((void**)&d_h,mx*my*mz*sizeof(myprec)) );
	checkCudaDev( hipMalloc((void**)&d_t,mx*my*mz*sizeof(myprec)) );
	checkCudaDev( hipMalloc((void**)&d_p,mx*my*mz*sizeof(myprec)) );
	checkCudaDev( hipMalloc((void**)&d_m,mx*my*mz*sizeof(myprec)) );
	checkCudaDev( hipMalloc((void**)&d_l,mx*my*mz*sizeof(myprec)) );

	checkCudaDev( hipMalloc((void**)&d_rO,mx*my*mz*sizeof(myprec)) );
	checkCudaDev( hipMalloc((void**)&d_eO,mx*my*mz*sizeof(myprec)) );
	checkCudaDev( hipMalloc((void**)&d_uO,mx*my*mz*sizeof(myprec)) );
	checkCudaDev( hipMalloc((void**)&d_vO,mx*my*mz*sizeof(myprec)) );
	checkCudaDev( hipMalloc((void**)&d_wO,mx*my*mz*sizeof(myprec)) );

	checkCudaDev( hipMalloc((void**)&d_dil,mx*my*mz*sizeof(myprec)) );
	for (int i=0; i<9; i++)
    	checkCudaDev( hipMalloc((void**)&sij[i],mx*my*mz*sizeof(myprec)) );

}

__device__ void clearSolver() {

	for (int i=0; i<3; i++) {
		checkCudaDev( hipFree(d_rhsr1[i]) );
		checkCudaDev( hipFree(d_rhsu1[i]) );
		checkCudaDev( hipFree(d_rhsv1[i]) );
		checkCudaDev( hipFree(d_rhsw1[i]) );
		checkCudaDev( hipFree(d_rhse1[i]) );

		checkCudaDev( hipFree(d_rhsr2[i]) );
		checkCudaDev( hipFree(d_rhsu2[i]) );
		checkCudaDev( hipFree(d_rhsv2[i]) );
		checkCudaDev( hipFree(d_rhsw2[i]) );
		checkCudaDev( hipFree(d_rhse2[i]) );

		checkCudaDev( hipFree(d_rhsr3[i]) );
		checkCudaDev( hipFree(d_rhsu3[i]) );
		checkCudaDev( hipFree(d_rhsv3[i]) );
		checkCudaDev( hipFree(d_rhsw3[i]) );
		checkCudaDev( hipFree(d_rhse3[i]) );
#if rk==4
		checkCudaDev( hipFree(d_rhsr4[i]) );
		checkCudaDev( hipFree(d_rhsu4[i]) );
		checkCudaDev( hipFree(d_rhsv4[i]) );
		checkCudaDev( hipFree(d_rhsw4[i]) );
		checkCudaDev( hipFree(d_rhse4[i]) );
#endif
	}
	checkCudaDev( hipFree(d_h) );
	checkCudaDev( hipFree(d_t) );
	checkCudaDev( hipFree(d_p) );
	checkCudaDev( hipFree(d_m) );
	checkCudaDev( hipFree(d_l) );

	checkCudaDev( hipFree(d_rO) );
	checkCudaDev( hipFree(d_eO) );
	checkCudaDev( hipFree(d_uO) );
	checkCudaDev( hipFree(d_vO) );
	checkCudaDev( hipFree(d_wO) );

	checkCudaDev( hipFree(d_dil) );
	for (int i=0; i<9; i++)
    	checkCudaDev( hipFree(sij[i]) );

}
