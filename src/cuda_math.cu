#include "hip/hip_runtime.h"
#include "globals.h"
#include "cuda_functions.h"
#include "cuda_globals.h"
#include "cuda_math.h"

__device__ myprec *wrkM;
__device__ unsigned int block, grid, total;

__global__ void deviceBlocker() {
}

__global__ void deviceSum(myprec *a, myprec *b, myprec *c) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	a[id.g] = b[id.g] + c[id.g];
}

__global__ void deviceSumOne(myprec *a, myprec *b, myprec *c) {
	*a = *b + *c;
}

__global__ void deviceSub(myprec *a, myprec *b, myprec *c) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	a[id.g] = b[id.g] - c[id.g];
}

__global__ void deviceSca(myprec *a, myprec *bx, myprec *by, myprec *bz, myprec *cx, myprec *cy, myprec *cz) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	a[id.g] = bx[id.g]*cx[id.g] + by[id.g]*cy[id.g] + bz[id.g]*cz[id.g];
}

__global__ void deviceMul(myprec *a, myprec *b, myprec *c) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
		a[id.g] = b[id.g]*c[id.g];
}

__global__ void deviceDiv(myprec *a, myprec *b, myprec *c) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	a[id.g] = b[id.g]/c[id.g];
}

__global__ void deviceDivOne(myprec *a, myprec *b, myprec *c) {
	*a = *b/(*c);
}

__global__ void deviceCpy(myprec *a, myprec *b) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	a[id.g] = b[id.g];
}

__global__ void deviceCpyOne(myprec *a, myprec *b) {
	*a = *b;
}

__device__ void reduceToMax(myprec *gOut, myprec *var) {

	total = mx*my*mz;

	grid  = my / sPencils *  mz;
	block = mx * sPencils;

	checkCudaDev( hipMalloc((void**)&wrkM ,grid*sizeof(myprec)) );
	hipDeviceSynchronize();

	block = findPreviousPowerOf2(block);

	maxOfThreads<<<grid, block, block*sizeof(myprec)>>>(wrkM , var,  total);
	hipDeviceSynchronize();
	maxOfThreads<<<   1, block, block*sizeof(myprec)>>>(wrkM, wrkM, grid);
	hipDeviceSynchronize();

	*gOut = wrkM[0];

	checkCudaDev( hipFree(wrkM  ) );
}

__device__ void reduceToMin(myprec *gOut, myprec *var) {


	total = mx*my*mz;

	grid  = my / sPencils *  mz;
	block = mx * sPencils;

	checkCudaDev( hipMalloc((void**)&wrkM ,grid*sizeof(myprec)) );
	hipDeviceSynchronize();

	block = findPreviousPowerOf2(block);

	minOfThreads<<<grid, block, block*sizeof(myprec)>>>(wrkM, var,  total);
	hipDeviceSynchronize();
	minOfThreads<<<   1, block, block*sizeof(myprec)>>>(wrkM, wrkM, grid);
	hipDeviceSynchronize();

	*gOut = wrkM[0];

	checkCudaDev( hipFree(wrkM  ) );

}

__device__ void volumeIntegral(myprec *gOut, myprec *var) {

	grid  = my*mz;
	block = mx;
	total = mx*my*mz;

	checkCudaDev( hipMalloc((void**)&wrkM ,grid*sizeof(myprec)) );
	hipDeviceSynchronize();

	block = findPreviousPowerOf2(block);

	integrateThreads<<<grid, block, block*sizeof(myprec)>>>(wrkM, var , total);
	hipDeviceSynchronize();
	reduceThreads<<<   1, block, block*sizeof(myprec)>>>(wrkM, wrkM, grid);
	hipDeviceSynchronize();

	*gOut = wrkM[0];
	checkCudaDev( hipFree( wrkM ) );
}

__device__ void reduceToOne(myprec *gOut, myprec *var) {

	total = mx*my*mz;

	grid  = my / sPencils *  mz;
	block = mx * sPencils;


	checkCudaDev( hipMalloc((void**)&wrkM ,grid*sizeof(myprec)) );
	hipDeviceSynchronize();

	block = findPreviousPowerOf2(block);

	reduceThreads<<<grid, block, block*sizeof(myprec)>>>(wrkM, var , total);
	hipDeviceSynchronize();
	reduceThreads<<<   1, block, block*sizeof(myprec)>>>(wrkM, wrkM, grid);
	hipDeviceSynchronize();

	*gOut = wrkM[0];
	checkCudaDev( hipFree(wrkM  ) );

}

__global__ void integrateThreads(myprec *gOut, myprec *gArr, int arraySize) {

	int bdim  = blockDim.x;
	int tix   = threadIdx.x;
	int bix   = blockIdx.x;
	int gdim  = gridDim.x*blockDim.x;

	int glb   = tix + bix * bdim;

	myprec sum = 0;
	for (int it = glb; it < arraySize; it += gdim) {
		int i = it % mx;
		sum += gArr[it]*d_dxv[i]/d_dy/d_dz /(Lx*Ly*Lz);
	}
	extern __shared__ myprec shArr[];
	shArr[tix] = sum;
	__syncthreads();
	for (int size = bdim/2; size>0; size/=2) {
		if (tix<size)
			shArr[tix] += shArr[tix+size];
		__syncthreads();
	}
	if (tix == 0) {
			gOut[bix] = shArr[0];
	}

	__syncthreads();

}

__global__ void AverageThreads(myprec *gOut, myprec *gArr, int arraySize) {

	int bdim  = blockDim.x;
	int tix   = threadIdx.x;
	int bix   = blockIdx.x;
	int gdim  = gridDim.x*blockDim.x;

	int glb   = tix + bix * bdim;

	myprec sum = 0;
	for (int it = glb; it < arraySize; it += gdim) {
		int i = it % mx;
		sum += gArr[it]*d_dxv[i]/d_dy/d_dz/Lx/Ly/Lz;
	}
	extern __shared__ myprec shArr[];
	shArr[tix] = sum;
	__syncthreads();
	for (int size = bdim/2; size>0; size/=2) {
		if (tix<size)
			shArr[tix] += shArr[tix+size];
		__syncthreads();
	}
	if (tix == 0) {
			gOut[bix] = shArr[0];
	}

	__syncthreads();

}

__global__ void reduceThreads(myprec *gOut, myprec *gArr, int arraySize) {

	int bdim  = blockDim.x;
	int tix   = threadIdx.x;
	int bix   = blockIdx.x;
	int gdim  = gridDim.x*blockDim.x;

	int glb   = tix + bix * bdim;

	myprec sum = 0;
    for (int i = glb; i < arraySize; i += gdim)
        sum += gArr[i];

	extern __shared__ myprec shArr[];
	shArr[tix] = sum;
	__syncthreads();
	for (int size = bdim/2; size>0; size/=2) {
		if (tix<size)
			shArr[tix] += shArr[tix+size];
		__syncthreads();
	}
	if (tix == 0) {
			gOut[bix] = shArr[0];
	}

	__syncthreads();

}

__global__ void minOfThreads(myprec *gOut, myprec *gArr, int arraySize) {

	int bdim  = blockDim.x;
	int tix   = threadIdx.x;
	int bix   = blockIdx.x;
	int gdim  = gridDim.x*blockDim.x;

	int glb   = tix + bix * bdim;

	myprec sum = 100000;
    for (int i = glb; i < arraySize; i += gdim)
        sum = MIN(sum,gArr[i]);

	extern __shared__ myprec shArr[];
	shArr[tix] = sum;
	__syncthreads();
	for (int size = bdim/2; size>0; size/=2) {
		if (tix<size)
			shArr[tix] = MIN(shArr[tix],shArr[tix+size]);
		__syncthreads();
	}
	if (tix == 0) {
			gOut[bix] = shArr[0];
	}

	__syncthreads();

}

__global__ void maxOfThreads(myprec *gOut, myprec *gArr, int arraySize) {

	int bdim  = blockDim.x;
	int tix   = threadIdx.x;
	int bix   = blockIdx.x;
	int gdim  = gridDim.x*blockDim.x;

	int glb   = tix + bix * bdim;

	myprec sum = -100000;
    for (int i = glb; i < arraySize; i += gdim)
        sum = MAX(sum,gArr[i]);

	extern __shared__ myprec shArr[];
	shArr[tix] = sum;
	__syncthreads();
	for (int size = bdim/2; size>0; size/=2) {
		if (tix<size)
			shArr[tix] = MAX(shArr[tix],shArr[tix+size]);
		__syncthreads();
	}
	if (tix == 0) {
			gOut[bix] = shArr[0];
	}

	__syncthreads();

}

__device__ unsigned int findPreviousPowerOf2(unsigned int n)
{
    while (n & n - 1) {
        n = n & n - 1;        // unset rightmost bit
    }

    return n;
}

__device__ unsigned int findNextPowerOf2(unsigned int n)
{
    unsigned count = 0;
    while(n != 0)
    {
        n >>= 1;
        count += 1;
    }
    
    n = 1 << count;
    return n;
}

unsigned int hostFindPreviousPowerOf2(unsigned int n)
{
    while (n & n - 1) {
        n = n & n - 1;        // unset rightmost bit
    }

    return n;
}

unsigned int hostfindNextPowerOf2(unsigned int n)
{
    unsigned count = 0;
    while(n != 0)
    {
        n >>= 1;
        count += 1;
    }

    n = 1 << count;
    return n;
}


void hostReduceToMin(myprec *gOut, myprec *var, Communicator rk) {

	hipSetDevice(rk.nodeRank);

	myprec *dwrkM;

	int tot = mx*my*mz;

	int gr  = my / sPencils *  mz;
	int bl = mx * sPencils;

	checkCuda( hipMalloc((void**)&dwrkM ,gr*sizeof(myprec)) );
	hipDeviceSynchronize();

	bl = hostFindPreviousPowerOf2(bl);

	minOfThreads<<< gr, bl, bl*sizeof(myprec)>>>(dwrkM, var,  tot);
	hipDeviceSynchronize();
	minOfThreads<<< 1 , bl, bl*sizeof(myprec)>>>(dwrkM, dwrkM, gr);
	hipDeviceSynchronize();

	deviceCpyOne<<<1,1>>>(gOut,&dwrkM[0]);

	checkCuda( hipFree(dwrkM  ) );

}

void hostVolumeIntegral(myprec *gOut, myprec *var, Communicator rk) {

	hipSetDevice(rk.nodeRank);

	myprec *dwrkM;

	int tot = mx*my*mz;

	int gr  = my *  mz;
	int bl  = mx ;

	bl = hostFindPreviousPowerOf2(bl);

	checkCuda( hipMalloc((void**)&dwrkM ,gr*sizeof(myprec)) );
	hipDeviceSynchronize();
	integrateThreads<<<gr, bl, bl*sizeof(myprec)>>>(dwrkM, var , tot);
	hipDeviceSynchronize();
	reduceThreads<<<   1 , bl, bl*sizeof(myprec)>>>(dwrkM, dwrkM, gr);
	hipDeviceSynchronize();

	deviceCpyOne<<<1,1>>>(gOut,&dwrkM[0]);

	checkCuda( hipFree(dwrkM  ) );
}

void hostVolumeAverage(myprec *gOut, myprec *var, Communicator rk) {

	hipSetDevice(rk.nodeRank);

	myprec *dwrkM;

	int tot = mx*my*mz;

	int gr  = my *  mz;
	int bl = mx ;

	checkCuda( hipMalloc((void**)&dwrkM ,gr*sizeof(myprec)) );
	hipDeviceSynchronize();

	bl = hostFindPreviousPowerOf2(bl);

	AverageThreads<<<gr, bl, bl*sizeof(myprec)>>>(dwrkM, var , tot);
	hipDeviceSynchronize();
	reduceThreads<<<   1 , bl, bl*sizeof(myprec)>>>(dwrkM, dwrkM, gr);
	hipDeviceSynchronize();

	deviceCpyOne<<<1,1>>>(gOut,&dwrkM[0]);

	checkCuda( hipFree(dwrkM  ) );
}


