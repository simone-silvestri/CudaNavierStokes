#include "hip/hip_runtime.h"
#include "globals.h"
#include "cuda_functions.h"
#include "cuda_globals.h"
#include "cuda_math.h"

__device__ myprec *wrkM;
__device__ int block, grid, total;

__global__ void deviceSum(myprec *a, myprec *b, myprec *c) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	a[id.g] = b[id.g] + c[id.g];
}

__global__ void deviceSub(myprec *a, myprec *b, myprec *c) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	a[id.g] = b[id.g] - c[id.g];
}

__global__ void deviceSca(myprec *a, myprec *bx, myprec *by, myprec *bz, myprec *cx, myprec *cy, myprec *cz) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	a[id.g] = bx[id.g]*cx[id.g] + by[id.g]*cy[id.g] + bz[id.g]*cz[id.g];
}

__global__ void deviceMul(myprec *a, myprec *b, myprec *c) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	a[id.g] = b[id.g]*c[id.g];
}

__global__ void deviceDiv(myprec *a, myprec *b, myprec *c) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	a[id.g] = b[id.g]/c[id.g];
}

__global__ void deviceCpy(myprec *a, myprec *b) {
	Indices id(threadIdx.x,threadIdx.y,blockIdx.x,blockIdx.y,blockDim.x,blockDim.y);
	a[id.g] = b[id.g];
}

__global__ void reduceToMax(myprec *gOut, myprec *var) {

	checkCudaDev( hipMalloc((void**)&wrkM ,grid0.x*grid0.y*sizeof(myprec)) );

	total = mx*my*mz;
	grid  = grid0.x*grid0.y;
	block = block0.x*block0.y;

	maxOfThreads<<<grid, block, block*sizeof(myprec)>>>(wrkM , var,  total);
	hipDeviceSynchronize();
	maxOfThreads<<<   1, block, block*sizeof(myprec)>>>(wrkM, wrkM, grid);
	hipDeviceSynchronize();

	*gOut = wrkM[0];

	checkCudaDev( hipFree(wrkM  ) );
}

__global__ void reduceToMin(myprec *gOut, myprec *var) {

	checkCudaDev( hipMalloc((void**)&wrkM ,grid0.x*grid0.y*sizeof(myprec)) );

	total = mx*my*mz;
	grid  = grid0.x*grid0.y;
	block = block0.x*block0.y;

	minOfThreads<<<grid, block, block*sizeof(myprec)>>>(wrkM, var,  total);
	hipDeviceSynchronize();
	minOfThreads<<<   1, block, block*sizeof(myprec)>>>(wrkM, wrkM, grid);
	hipDeviceSynchronize();

	*gOut = wrkM[0];

	checkCudaDev( hipFree(wrkM  ) );

}

__global__ void reduceToOne(myprec *gOut, myprec *var) {

	checkCudaDev( hipMalloc((void**)&wrkM ,grid0.x*grid0.y*sizeof(myprec)) );

	total = mx*my*mz;
	grid  = grid0.x*grid0.y;
	block = block0.x*block0.y;

	reduceThreads<<<grid, block, block*sizeof(myprec)>>>(wrkM, var , total);
	hipDeviceSynchronize();
	reduceThreads<<<   1, block, block*sizeof(myprec)>>>(wrkM, wrkM, grid);
	hipDeviceSynchronize();

	*gOut = wrkM[0];

	checkCudaDev( hipFree(wrkM  ) );

}

__global__ void reduceThreads(myprec *gOut, myprec *gArr, int arraySize) {

	int bdim  = blockDim.x;
	int tix   = threadIdx.x;
	int bix   = blockIdx.x;
	int gdim  = gridDim.x*blockDim.x;

	int glb   = tix + bix * bdim;

	myprec sum = 0;
    for (int i = glb; i < arraySize; i += gdim)
        sum += gArr[i];

	extern __shared__ myprec shArr[];
	shArr[tix] = sum;
	__syncthreads();
	for (int size = bdim/2; size>0; size/=2) {
		if (tix<size)
			shArr[tix] += shArr[tix+size];
		__syncthreads();
	}
	if (tix == 0) {
			gOut[bix] = shArr[0];
	}

	__syncthreads();

}

__global__ void minOfThreads(myprec *gOut, myprec *gArr, int arraySize) {

	int bdim  = blockDim.x;
	int tix   = threadIdx.x;
	int bix   = blockIdx.x;
	int gdim  = gridDim.x*blockDim.x;

	int glb   = tix + bix * bdim;

	myprec sum = 100000;
    for (int i = glb; i < arraySize; i += gdim)
        sum = MIN(sum,gArr[i]);

	extern __shared__ myprec shArr[];
	shArr[tix] = sum;
	__syncthreads();
	for (int size = bdim/2; size>0; size/=2) {
		if (tix<size)
			shArr[tix] = MIN(shArr[tix],shArr[tix+size]);
		__syncthreads();
	}
	if (tix == 0) {
			gOut[bix] = shArr[0];
	}

	__syncthreads();

}

__global__ void maxOfThreads(myprec *gOut, myprec *gArr, int arraySize) {

	int bdim  = blockDim.x;
	int tix   = threadIdx.x;
	int bix   = blockIdx.x;
	int gdim  = gridDim.x*blockDim.x;

	int glb   = tix + bix * bdim;

	myprec sum = -100000;
    for (int i = glb; i < arraySize; i += gdim)
        sum = MAX(sum,gArr[i]);

	extern __shared__ myprec shArr[];
	shArr[tix] = sum;
	__syncthreads();
	for (int size = bdim/2; size>0; size/=2) {
		if (tix<size)
			shArr[tix] = MAX(shArr[tix],shArr[tix+size]);
		__syncthreads();
	}
	if (tix == 0) {
			gOut[bix] = shArr[0];
	}

	__syncthreads();

}





